```c
/* 
Aim of the program:
Check if the device supports host-mapped pageable memory access (`pageableMemoryAccess`).

Thinking:
To determine whether the CUDA-capable device allows host-mapped pageable memory access, we need to inspect the device's properties exposed by the CUDA Runtime API. The `hipDeviceProp_t` structure contains a boolean flag `pageableMemoryAccess` that indicates this capability. The steps are:
1. Retrieve the current device ID with `hipGetDevice()`.
2. Allocate a `hipDeviceProp_t` structure and fill it using `hipGetDeviceProperties()`.
3. Read the `pageableMemoryAccess` field.
4. Print a friendly message indicating whether the feature is supported.
This program compiles with `nvcc` and prints the result to the console.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int device = 0;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get current device: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device properties: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Device %d: %s\n", device, prop.name);
    if (prop.pageableMemoryAccess) {
        printf("Host-mapped pageable memory access (pageableMemoryAccess) is supported.\n");
    } else {
        printf("Host-mapped pageable memory access (pageableMemoryAccess) is NOT supported.\n");
    }

    return 0;
}
```