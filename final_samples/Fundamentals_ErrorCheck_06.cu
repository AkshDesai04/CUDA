#include "hip/hip_runtime.h"
```cpp
/* 
Aim: Create a C++ helper function `void checkKernelLaunch()` that calls `hipDeviceSynchronize()` and checks for errors. Call this function after every kernel launch in your programs to ensure the kernel executed without an asynchronous error.

Thinking:
- The goal is to provide a reusable helper that can be inserted immediately after any CUDA kernel launch.
- We need to capture both the synchronous error that might occur during the kernel launch itself (`hipGetLastError`) and any asynchronous errors that surface during kernel execution (`hipDeviceSynchronize`).
- The helper will check the return status of both calls, print a descriptive error message, and terminate the program if an error is detected.
- For demonstration, a simple vector addition kernel is implemented. After launching the kernel, `checkKernelLaunch()` is invoked to validate the launch.
- This file is self‑contained and can be compiled with `nvcc -o vector_add vector_add.cu`.
*/

#include <iostream>
#include <hip/hip_runtime.h>

// Helper function to check kernel launch success
void checkKernelLaunch()
{
    // Check for errors during kernel launch (synchronous)
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    // Wait for the device to finish and check for asynchronous errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        std::cerr << "Kernel execution error: " << hipGetErrorString(err) << std::endl;
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

// Simple vector addition kernel
__global__ void vectorAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main()
{
    const int N = 1 << 20; // 1M elements
    const size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize host data
    for (int i = 0; i < N; ++i)
    {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Check kernel launch and execution
    checkKernelLaunch();

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify results (simple check)
    bool success = true;
    for (int i = 0; i < N; ++i)
    {
        if (h_C[i] != h_A[i] + h_B[i])
        {
            std::cerr << "Verification failed at index " << i << std::endl;
            success = false;
            break;
        }
    }

    if (success)
        std::cout << "Vector addition successful!" << std::endl;

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    // Reset device
    hipDeviceReset();

    return 0;
}
```