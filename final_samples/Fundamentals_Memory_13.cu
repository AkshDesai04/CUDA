```c
/*
Aim: Demonstrate a `hipErrorOutOfMemory` error by attempting to allocate an impossibly large amount of memory on the device (e.g., 100 GB). Catch and report the error gracefully.

Thinking:
1. We want to trigger `hipErrorOutOfMemory` by asking for more memory than the GPU can provide. 100 GB is far beyond typical device memory.
2. We'll query the device memory using `hipMemGetInfo` to display the available memory before the allocation attempt.
3. We'll use `hipMalloc` to request 100 GB of device memory and check the returned `hipError_t` code.
4. If the allocation fails, we'll print the error string using `hipGetErrorString` and exit gracefully.
5. For clarity and reusability, we wrap error checking in a simple macro `checkCudaErrors`.
6. The program is written in C (using CUDA runtime API) and compiles as a .cu file. No external libraries or frameworks are required.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Helper macro for error checking */
#define checkCudaErrors(call)                                    \
    do {                                                         \
        hipError_t err = call;                                  \
        if (err != hipSuccess) {                                \
            fprintf(stderr, "CUDA error at %s:%d - %s\n",        \
                    __FILE__, __LINE__, hipGetErrorString(err));\
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    } while (0)

int main(void)
{
    /* Print device information */
    int deviceCount = 0;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable devices found.\n");
        return EXIT_FAILURE;
    }

    int dev = 0;
    checkCudaErrors(hipSetDevice(dev));

    /* Get free and total memory */
    size_t freeMem, totalMem;
    checkCudaErrors(hipMemGetInfo(&freeMem, &totalMem));

    printf("Device %d: Total memory = %.2f GB, Free memory = %.2f GB\n",
           dev, (double)totalMem / (1024.0 * 1024.0 * 1024.0),
           (double)freeMem  / (1024.0 * 1024.0 * 1024.0));

    /* Attempt to allocate 100 GB */
    const size_t hugeSize = 100ULL * 1024ULL * 1024ULL * 1024ULL; // 100 GiB
    void *d_ptr = NULL;
    printf("Attempting to allocate %.2f GB on device...\n",
           (double)hugeSize / (1024.0 * 1024.0 * 1024.0));

    hipError_t status = hipMalloc(&d_ptr, hugeSize);
    if (status != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(status));
        if (status == hipErrorOutOfMemory) {
            fprintf(stderr, "Error: Not enough device memory available.\n");
        } else {
            fprintf(stderr, "Error: An unexpected CUDA error occurred.\n");
        }
        return EXIT_FAILURE;
    }

    /* If allocation somehow succeeded (unlikely) */
    printf("Allocation succeeded (unexpected). Cleaning up.\n");
    checkCudaErrors(hipFree(d_ptr));

    return EXIT_SUCCESS;
}
```