#include "hip/hip_runtime.h"
```c
/*
If you are using CUDA events, wrap `hipEventCreate`, `hipEventRecord`, and `hipEventDestroy`.

The goal of this program is to illustrate how to encapsulate CUDA event functions
within custom wrapper functions that perform error checking and provide a clean
interface for creating, recording, and destroying events.  By wrapping these
functions, we can centralize error handling and simplify the main program
logic.  The wrapper functions (`safeEventCreate`, `safeEventRecord`, and
`safeEventDestroy`) each return a `hipError_t` status, allowing the caller
to decide how to handle failures.

In addition to the wrappers, the program demonstrates a simple vector addition
kernel.  Two events (`start` and `stop`) are created, recorded before and after
the kernel launch, and then used to compute the elapsed time.  After timing,
the events are destroyed.  All CUDA API calls are checked using the wrapper
functions, ensuring consistent error handling.

The program is self‑contained and can be compiled with `nvcc`:
    nvcc -o cuda_event_wrappers cuda_event_wrappers.cu
It will output the elapsed time for the kernel execution on the GPU.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

/* Wrapper for hipEventCreate */
hipError_t safeEventCreate(hipEvent_t* event) {
    hipError_t err = hipEventCreate(event);
    if (err != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed: %s\n", hipGetErrorString(err));
    }
    return err;
}

/* Wrapper for hipEventRecord */
hipError_t safeEventRecord(hipEvent_t event, hipStream_t stream) {
    hipError_t err = hipEventRecord(event, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "hipEventRecord failed: %s\n", hipGetErrorString(err));
    }
    return err;
}

/* Wrapper for hipEventDestroy */
hipError_t safeEventDestroy(hipEvent_t event) {
    hipError_t err = hipEventDestroy(event);
    if (err != hipSuccess) {
        fprintf(stderr, "hipEventDestroy failed: %s\n", hipGetErrorString(err));
    }
    return err;
}

/* Simple vector addition kernel */
__global__ void vecAdd(const float *A, const float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) C[idx] = A[idx] + B[idx];
}

int main(void) {
    const int N = 1 << 20;          // 1M elements
    const size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = i * 1.0f;
        h_B[i] = (N - i) * 1.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    safeEventCreate(&start);
    safeEventCreate(&stop);

    /* Record start event */
    safeEventRecord(start, 0);

    /* Launch kernel */
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    /* Record stop event */
    safeEventRecord(stop, 0);

    /* Wait for kernel to finish */
    hipDeviceSynchronize();

    /* Compute elapsed time */
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time: %f ms\n", milliseconds);

    /* Copy result back to host */
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    /* Cleanup */
    safeEventDestroy(start);
    safeEventDestroy(stop);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
```