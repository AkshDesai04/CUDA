```cuda
/*
Aim of the program: Check if the device is part of a TCC (Tesla Compute Cluster) (`isTCCDriver`).

Thinking:
The task is to query each CUDA device to determine whether it is operating in Tesla Compute Cluster (TCC) mode. The CUDA Driver API provides the `hipDeviceGetAttribute` function, which can be used with the attribute `hipDeviceAttributeTccDriver` to check this status. We need to:
1. Initialize the CUDA driver with `hipInit(0)`.
2. Retrieve the number of devices with `hipGetDeviceCount`.
3. Iterate over each device:
   a. Get the device handle with `hipDeviceGet`.
   b. Query the TCC attribute via `hipDeviceGetAttribute`.
   c. Print whether the device is in TCC mode.
4. Handle any CUDA errors gracefully by printing messages and exiting.
The code will be written in C with CUDA and compiled as a `.cu` file. No external files or libraries beyond the CUDA Driver API are required.*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERR(err, msg)                                            \
    do {                                                                     \
        if (err != hipSuccess) {                                          \
            const char *errStr = nullptr;                                   \
            hipDrvGetErrorString(err, &errStr);                                 \
            fprintf(stderr, "CUDA Error: %s: %s\n", msg,                    \
                    errStr ? errStr : "Unknown error");                     \
            return -1;                                                      \
        }                                                                    \
    } while (0)

int main(void) {
    hipError_t res;

    /* Initialize the CUDA driver API */
    res = hipInit(0);
    CHECK_CUDA_ERR(res, "hipInit failed");

    /* Get the number of CUDA devices */
    int deviceCount = 0;
    res = hipGetDeviceCount(&deviceCount);
    CHECK_CUDA_ERR(res, "hipGetDeviceCount failed");

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return 0;
    }

    printf("Detected %d CUDA device(s).\n", deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDevice_t device;
        res = hipDeviceGet(&device, dev);
        CHECK_CUDA_ERR(res, "hipDeviceGet failed");

        /* Retrieve device name */
        char name[256] = {0};
        res = hipDeviceGetName(name, sizeof(name), device);
        CHECK_CUDA_ERR(res, "hipDeviceGetName failed");

        /* Query TCC driver attribute */
        int isTCC = 0;
        res = hipDeviceGetAttribute(&isTCC,
                                   hipDeviceAttributeTccDriver,
                                   device);
        /* The attribute may not be supported on all GPUs; handle that case */
        if (res == hipErrorInvalidValue) {
            printf("Device %d (%s): TCC attribute not supported.\n", dev, name);
            continue;
        }
        CHECK_CUDA_ERR(res, "hipDeviceGetAttribute failed");

        printf("Device %d (%s): TCC mode is %s.\n",
               dev, name, isTCC ? "ENABLED" : "DISABLED");
    }

    return 0;
}
```