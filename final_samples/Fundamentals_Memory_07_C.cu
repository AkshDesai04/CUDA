#include "hip/hip_runtime.h"
```cuda
/*
Allocate memory with hipMallocManaged. Now try to access this pointer from the host. Does it work? Why?

Thinking:
- hipMallocManaged allocates unified memory accessible from both the host and the device.
- When accessed on the host, the pointer is valid; the runtime will migrate the page if it is not resident in the host memory.
- However, if the device writes to that memory, the host must call hipDeviceSynchronize() or hipMemcpy to ensure data is visible.
- The code below demonstrates allocation, a kernel that writes to the unified memory, synchronizing, and then reading from the host.
- The program prints the results, showing that host access works.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void writeKernel(int *data, int val) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Simple write: each thread writes its index plus val
    data[idx] = idx + val;
}

int main(void) {
    const int N = 10;
    int *unifiedArray;

    // Allocate unified memory
    hipError_t err = hipMallocManaged(&unifiedArray, N * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Launch kernel to write values
    writeKernel<<<1, N>>>(unifiedArray, 100);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(unifiedArray);
        return -1;
    }

    // Wait for device to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        hipFree(unifiedArray);
        return -1;
    }

    // Access from host
    printf("Unified memory contents accessed from host:\n");
    for (int i = 0; i < N; ++i) {
        printf("unifiedArray[%d] = %d\n", i, unifiedArray[i]);
    }

    // Clean up
    hipFree(unifiedArray);
    return 0;
}
```