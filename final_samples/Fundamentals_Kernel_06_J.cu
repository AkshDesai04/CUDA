#include "hip/hip_runtime.h"
```cuda
/*
Aim: Add robust error checking around every CUDA API call (`hipMalloc`, `hipMemcpy`, `hipFree`).

Thinking:
The goal is to wrap each CUDA runtime API call with error checking so that any failure is reported
with a clear message and the program exits gracefully.  A convenient approach is to define a
macro that executes the call, captures the returned hipError_t, and if it is not hipSuccess,
prints the file, line, and error string and terminates the program.  We also need to check for
kernel launch errors using hipGetLastError() and hipDeviceSynchronize() after the kernel
launch.  All memory allocations (host and device), copies, and frees will use this macro.  The
example program below allocates a small array, copies it to the device, runs a simple kernel
that increments each element, copies the result back, prints it, and finally frees the
resources, with robust error handling at each step.  This demonstrates the requested robust
error checking in a self‑contained CUDA C program.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/* Macro for checking CUDA runtime API calls */
#define CUDA_CHECK(call)                                                     \
    do {                                                                     \
        hipError_t err = call;                                              \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                     \
                    __FILE__, __LINE__, hipGetErrorString(err));           \
            exit(EXIT_FAILURE);                                             \
        }                                                                    \
    } while (0)

/* Simple kernel that adds 1 to each element of the array */
__global__ void add_one(int *d_arr, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_arr[idx] += 1;
    }
}

int main(void)
{
    const int N = 10;
    size_t size = N * sizeof(int);
    int *h_arr = (int *)malloc(size);
    if (!h_arr) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    /* Initialize host array */
    for (int i = 0; i < N; ++i) {
        h_arr[i] = i;
    }

    /* Device memory allocation with error checking */
    int *d_arr = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_arr, size));

    /* Copy data from host to device with error checking */
    CUDA_CHECK(hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice));

    /* Launch kernel and check for launch errors */
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add_one<<<blocksPerGrid, threadsPerBlock>>>(d_arr, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    /* Copy result back to host with error checking */
    CUDA_CHECK(hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost));

    /* Print the result */
    printf("Result after adding 1 to each element:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    /* Free device memory with error checking */
    CUDA_CHECK(hipFree(d_arr));

    /* Free host memory */
    free(h_arr);

    return EXIT_SUCCESS;
}
```