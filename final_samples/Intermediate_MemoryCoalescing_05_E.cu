/*
Aim of the program: Profile and time both versions and report the speedup of SoA over AoS.

The idea is to show how struct‑of‑arrays (SoA) can provide better memory coalescing and hence speed up
memory‑bound kernels compared to array‑of‑structs (AoS).  The program does the following:

1.  Defines a simple data structure for an AoS: a struct containing two floats.
2.  Allocates large arrays (16 million elements) for both AoS and SoA representations.
3.  Initializes the data on the host, copies it to the device.
4.  Implements two identical kernels: one that operates on an AoS array and one that operates
    on a SoA (two separate float arrays).  Each kernel performs a trivial computation
    (increments the two fields by constants) to keep the focus on memory access patterns.
5.  Times each kernel execution using CUDA events, ensuring proper synchronization.
6.  Computes and prints the elapsed times and the speedup factor of the SoA version over the AoS
    version.
7.  Cleans up all allocated resources.

By keeping the computation simple, the main performance difference comes from how the GPU
accesses memory: AoS requires fetching two floats that are interleaved, whereas SoA allows
coalesced access to one field at a time.  The program demonstrates this with measurable
timing outputs.

The code below is a complete .cu file that can be compiled with nvcc.
*/

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                              \
    do {                                                              \
        hipError_t err = call;                                       \
        if (err != hipSuccess) {                                     \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",              \
                    __FILE__, __LINE__, hipGetErrorString(err));     \
            exit(EXIT_FAILURE);                                       \
        }                                                             \
    } while (0)

// Number of elements (adjustable)
const size_t N = 1 << 24; // 16,777,216 elements

// AoS data structure
struct AoS {
    float x;
    float y;
};

// Kernel operating on AoS
__global__ void kernel_aos(AoS* data, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Simple operation: add constants
        data[idx].x += 1.0f;
        data[idx].y += 2.0f;
    }
}

// Kernel operating on SoA
__global__ void kernel_soa(float* x, float* y, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        x[idx] += 1.0f;
        y[idx] += 2.0f;
    }
}

int main(void) {
    // Allocate host memory
    AoS* h_aos = (AoS*)malloc(N * sizeof(AoS));
    float* h_x = (float*)malloc(N * sizeof(float));
    float* h_y = (float*)malloc(N * sizeof(float));

    // Initialize host data
    for (size_t i = 0; i < N; ++i) {
        h_aos[i].x = static_cast<float>(i);
        h_aos[i].y = static_cast<float>(i) * 0.5f;
        h_x[i] = static_cast<float>(i);
        h_y[i] = static_cast<float>(i) * 0.5f;
    }

    // Allocate device memory
    AoS* d_aos;
    float *d_x, *d_y;
    CHECK_CUDA(hipMalloc((void**)&d_aos, N * sizeof(AoS)));
    CHECK_CUDA(hipMalloc((void**)&d_x, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_y, N * sizeof(float)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_aos, h_aos, N * sizeof(AoS), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));

    // Define grid and block dimensions
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Timing variables
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // 1. Run AoS kernel
    CHECK_CUDA(hipEventRecord(start));
    kernel_aos<<<blocksPerGrid, threadsPerBlock>>>(d_aos, N);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float elapsed_aos;
    CHECK_CUDA(hipEventElapsedTime(&elapsed_aos, start, stop));

    // 2. Run SoA kernel
    CHECK_CUDA(hipEventRecord(start));
    kernel_soa<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, N);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float elapsed_soa;
    CHECK_CUDA(hipEventElapsedTime(&elapsed_soa, start, stop));

    // Print results
    printf("AoS kernel time:  %.3f ms\n", elapsed_aos);
    printf("SoA kernel time:  %.3f ms\n", elapsed_soa);
    if (elapsed_soa > 0.0f) {
        printf("Speedup (SoA / AoS): %.2fx\n", elapsed_aos / elapsed_soa);
    }

    // Clean up
    CHECK_CUDA(hipFree(d_aos));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
    free(h_aos);
    free(h_x);
    free(h_y);
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return 0;
}
