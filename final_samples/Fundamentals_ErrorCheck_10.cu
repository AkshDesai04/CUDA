#include "hip/hip_runtime.h"
```cpp
/*
Create a `GpuTimer` class in C++. The constructor should call `hipEventCreate` for start and stop events. A `start()` method calls `hipEventRecord`. A `stop()` method calls `hipEventRecord` and `hipEventSynchronize`. An `elapsed_ms()` method calls `hipEventElapsedTime`. Add error checking to all CUDA calls.

Thinking:
- The class will hold two `hipEvent_t` members: `start_` and `stop_`.
- The constructor will create these events via `hipEventCreate`, checking errors after each call.
- The destructor will destroy the events via `hipEventDestroy`.
- `start()` will record the start event on the default stream.
- `stop()` will record the stop event and then synchronize the event to ensure completion.
- `elapsed_ms()` will compute the elapsed time between `start_` and `stop_` using `hipEventElapsedTime`.
- A helper inline function `checkCuda` will wrap CUDA calls, printing the error string and aborting if the call fails.
- We'll include a small example kernel and a `main` function to demonstrate the timer usage.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// Helper to check CUDA errors
inline void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error (" << msg << "): " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

class GpuTimer {
public:
    GpuTimer() {
        checkCuda(hipEventCreate(&start_), "hipEventCreate start");
        checkCuda(hipEventCreate(&stop_), "hipEventCreate stop");
    }

    ~GpuTimer() {
        checkCuda(hipEventDestroy(start_), "hipEventDestroy start");
        checkCuda(hipEventDestroy(stop_), "hipEventDestroy stop");
    }

    void start() {
        checkCuda(hipEventRecord(start_, 0), "hipEventRecord start");
    }

    void stop() {
        checkCuda(hipEventRecord(stop_, 0), "hipEventRecord stop");
        checkCuda(hipEventSynchronize(stop_), "hipEventSynchronize stop");
    }

    float elapsed_ms() const {
        float ms = 0.0f;
        checkCuda(hipEventElapsedTime(&ms, start_, stop_), "hipEventElapsedTime");
        return ms;
    }

private:
    hipEvent_t start_;
    hipEvent_t stop_;
};

// Example kernel
__global__ void dummyKernel(int *d_out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_out[idx] = idx * 2;
    }
}

int main() {
    const int N = 1 << 20; // 1M elements
    int *d_data = nullptr;
    size_t size = N * sizeof(int);

    // Allocate device memory
    checkCuda(hipMalloc(&d_data, size), "hipMalloc d_data");

    // Launch kernel
    GpuTimer timer;
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

    timer.start();
    dummyKernel<<<gridSize, blockSize>>>(d_data, N);
    timer.stop();

    // Check for kernel launch errors
    checkCuda(hipGetLastError(), "Kernel launch");

    std::cout << "Kernel execution time: " << timer.elapsed_ms() << " ms" << std::endl;

    // Clean up
    checkCuda(hipFree(d_data), "hipFree d_data");

    return 0;
}
```