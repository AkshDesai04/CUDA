```cpp
/*
Write a program to get the ID of the currently active CUDA device using `hipGetDevice` and print it.

Thinking:
1. We need to include the CUDA runtime header `cuda_runtime.h` to access `hipGetDevice`.
2. Use `hipGetDevice` which returns a `hipError_t` and outputs the device ID via a pointer argument.
3. Handle the case where the call fails by printing the CUDA error string obtained via `hipGetErrorString`.
4. If the call succeeds, print the device ID to the console.
5. Use standard C++ IO (`std::cout`) for printing.
6. Return `EXIT_SUCCESS` on success and `EXIT_FAILURE` on error.
7. Keep the program minimal and self‑contained, compiling with `nvcc`.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

int main() {
    int device = 0;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        std::cerr << "Error getting active CUDA device: " 
                  << hipGetErrorString(err) << std::endl;
        return EXIT_FAILURE;
    }

    std::cout << "Active CUDA device ID: " << device << std::endl;
    return EXIT_SUCCESS;
}
```