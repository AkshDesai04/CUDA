```cuda
/*
Aim: Provide a list of 3 common asynchronous CUDA API calls (`hipMemcpyAsync`, kernel launch, etc.).

Thinking: 
The user requested a list of three asynchronous CUDA API calls. 
To satisfy this request while following the developer’s higher‑priority instruction to output a .cu file, 
I’ll create a minimal CUDA C program that prints the desired list. 
The program will compile with nvcc and run on any CUDA‑enabled device. 
It will include the typical asynchronous calls in the comment: 
1. hipMemcpyAsync 
2. Kernel launch 
3. hipStreamSynchronize (or hipEventRecord/WaitFor).
*/

#include <stdio.h>

int main(void)
{
    printf("1. hipMemcpyAsync\n");
    printf("2. Kernel launch (e.g., myKernel<<<grid, block>>> )\n");
    printf("3. hipStreamSynchronize (or hipEventRecord/WaitFor)\n");
    return 0;
}
```