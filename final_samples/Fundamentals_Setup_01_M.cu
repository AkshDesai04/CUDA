```cuda
/*
 * Aim of the program:
 * Query and print the major and minor compute capability numbers for the primary CUDA device separately.
 *
 * Entire thinking:
 * 1. The CUDA runtime provides `hipGetDevice` to get the currently active device
 *    (the primary device when the program starts, unless changed explicitly).
 * 2. Once we have the device ID, we can query device properties using
 *    `hipGetDeviceProperties`.  The `hipDeviceProp_t` struct contains `major`
 *    and `minor` fields that represent the compute capability version.
 * 3. For safety, we check the return codes of CUDA API calls and report
 *    errors if they occur.  The program will exit with a non‑zero status
 *    in case of failure.
 * 4. The output should display the major and minor numbers on separate lines
 *    for clarity, following the requirement.
 *
 * Implementation details:
 * - Include <hip/hip_runtime.h> and <stdio.h>.
 * - Use `hipGetDevice(&device)` to obtain the current device.
 * - Allocate a `hipDeviceProp_t` struct and call `hipGetDeviceProperties(&prop, device)`.
 * - Print `prop.major` and `prop.minor` using `printf`.
 * - Return 0 on success, non‑zero on failure.
 *
 * The program is self‑contained in a single .cu file and can be compiled
 * with `nvcc -o compute_capability compute_capability.cu`.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
    int device = 0;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: hipGetDevice failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: hipGetDeviceProperties failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Compute Capability Major: %d\n", prop.major);
    printf("Compute Capability Minor: %d\n", prop.minor);

    return 0;
}
```