```cpp
/*
Aim: Try to call `hipFree` on the same pointer twice. What happens? Use error checking to see the result of the second call.

Thinking:
The goal is to demonstrate that freeing the same device pointer twice results in an error. 
In CUDA, `hipFree` returns a `hipError_t`. After the first call, the memory block is deallocated and the pointer is no longer valid. 
Calling `hipFree` again on the same address should produce an error such as `hipErrorInvalidDevicePointer` or a similar error code. 
We will allocate a small integer on the device, free it once, and then attempt to free it again. 
After each call we will check the return status and print both the numeric error code and the human‑readable error string provided by `hipGetErrorString`. 
This will illustrate the error handling mechanism for repeated frees. 
The program is written in C/C++ using CUDA runtime API, compiled with `nvcc` as a `.cu` file.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Macro for checking CUDA API call errors
#define CUDA_CHECK(call)                                         \
    do {                                                         \
        hipError_t err = call;                                  \
        if (err != hipSuccess) {                                \
            fprintf(stderr, "CUDA error in file '%s' at line %d: %s (%d)\n", \
                    __FILE__, __LINE__, hipGetErrorString(err), err); \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    } while (0)

int main(void)
{
    int *d_ptr = NULL;
    size_t size = sizeof(int);

    // Allocate device memory
    CUDA_CHECK(hipMalloc((void**)&d_ptr, size));
    printf("Allocated %zu bytes on device at %p\n", size, (void*)d_ptr);

    // First hipFree call
    hipError_t err1 = hipFree(d_ptr);
    if (err1 == hipSuccess) {
        printf("First hipFree succeeded.\n");
    } else {
        printf("First hipFree failed with error: %s (%d)\n",
               hipGetErrorString(err1), err1);
    }

    // Second hipFree call on the same pointer
    hipError_t err2 = hipFree(d_ptr);
    if (err2 == hipSuccess) {
        printf("Second hipFree succeeded (unexpected).\n");
    } else {
        printf("Second hipFree failed with error: %s (%d)\n",
               hipGetErrorString(err2), err2);
    }

    // Reset the device (optional but clean)
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
```