/*
Implement `C[i] = (A[i] == B[i]) ? 1 : 0`.

Thinking: 
To demonstrate a simple element‑wise comparison in CUDA, we implement a kernel that
takes two input integer arrays `A` and `B` and writes to a third array `C` such that
`C[i]` is 1 when `A[i]` equals `B[i]`, otherwise 0.  
Each thread handles one array index.  
The host code allocates sample data, copies it to the device, launches the kernel,
copies the result back, and prints it.  
Error checking is performed via a helper macro.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(call)                                            \
    do {                                                                  \
        hipError_t err = call;                                           \
        if (err != hipSuccess) {                                         \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",             \
                    __FUNCTION__, __FILE__, __LINE__,                    \
                    hipGetErrorString(err));                            \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    } while (0)

__global__ void compareKernel(const int *A, const int *B, int *C, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        C[idx] = (A[idx] == B[idx]) ? 1 : 0;
    }
}

int main(void)
{
    const size_t N = 10;
    size_t bytes = N * sizeof(int);

    int hA[N], hB[N], hC[N];

    // Initialize host arrays
    for (size_t i = 0; i < N; ++i)
    {
        hA[i] = i;
        hB[i] = (i % 2 == 0) ? i : i + 1; // some elements will differ
    }

    // Device pointers
    int *dA = NULL, *dB = NULL, *dC = NULL;

    CHECK_CUDA_ERROR(hipMalloc((void **)&dA, bytes));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dB, bytes));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dC, bytes));

    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice));

    // Launch kernel
    const int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    compareKernel<<<blocks, threadsPerBlock>>>(dA, dB, dC, N);
    CHECK_CUDA_ERROR(hipGetLastError());

    // Copy result back to host
    CHECK_CUDA_ERROR(hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost));

    // Print results
    printf("Index\tA\tB\tC\n");
    for (size_t i = 0; i < N; ++i)
    {
        printf("%zu\t%d\t%d\t%d\n", i, hA[i], hB[i], hC[i]);
    }

    // Clean up
    CHECK_CUDA_ERROR(hipFree(dA));
    CHECK_CUDA_ERROR(hipFree(dB));
    CHECK_CUDA_ERROR(hipFree(dC));

    return 0;
}
