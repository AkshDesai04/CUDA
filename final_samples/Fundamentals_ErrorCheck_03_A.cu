/*
Explain the difference between `cudaPeekAtLastError` and `cudaGetLastError` in a comment. (Peek does not reset the error state).

The aim of this program is to illustrate how `cudaPeekAtLastError()` and `cudaGetLastError()` behave differently.  
`cudaPeekAtLastError()` simply returns the last CUDA error that occurred, but it does *not* clear the error state.  
`cudaGetLastError()` also returns the last CUDA error, but it *does* reset the error state so that subsequent checks see no error.

In the program we intentionally launch a kernel with an invalid configuration (`<<<0,1>>>`), which generates an error (`cudaErrorInvalidConfiguration`).  
We then:

1. Call `cudaPeekAtLastError()` – this should return the error.
2. Call `cudaGetLastError()` – this should also return the error *and* reset the error state.
3. Call `cudaPeekAtLastError()` again – this should now return `cudaSuccess` because the state has been cleared.

This demonstrates the key difference: peek does not reset, get does.
*/
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void dummyKernel() {
    // Empty kernel
}

int main() {
    // Intentionally launch with 0 blocks to generate an error
    dummyKernel<<<0, 1>>>();

    // Check error with peek (does not reset)
    hipError_t errPeek1 = hipPeekAtLastError();
    printf("After invalid launch, cudaPeekAtLastError() returned: %s\n",
           hipGetErrorString(errPeek1));

    // Check error with get (resets)
    hipError_t errGet = hipGetLastError();
    printf("cudaGetLastError() returned: %s\n", hipGetErrorString(errGet));

    // Now peek again – should be cudaSuccess because get reset the state
    hipError_t errPeek2 = hipPeekAtLastError();
    printf("After cudaGetLastError(), cudaPeekAtLastError() returned: %s\n",
           hipGetErrorString(errPeek2));

    // Also peek again to show that state remains cleared
    hipError_t errPeek3 = hipPeekAtLastError();
    printf("Second peek, still cleared: %s\n", hipGetErrorString(errPeek3));

    return 0;
}
