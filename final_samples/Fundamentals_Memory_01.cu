```cpp
/* 
Aim of the program: Write a program that allocates an array of 256 integers on the host and initializes it with values from 0 to 255. Then, allocate memory for 256 integers on the GPU device using `hipMalloc`. Finally, copy the host array to the device array using `hipMemcpy` with the `hipMemcpyHostToDevice` kind.

Thinking:
1. Include the necessary headers: <cstdio> for printf, <hip/hip_runtime.h> for CUDA API.
2. Define a macro `CUDA_CHECK` to wrap CUDA calls and report errors.
3. In main:
   a. Allocate an array of 256 integers on the host (`int *h_array = new int[256];`).
   b. Initialize it with values 0..255 using a loop.
   c. Allocate memory on the device (`int *d_array;` with `hipMalloc((void**)&d_array, 256 * sizeof(int));`).
   d. Copy the data from host to device using `hipMemcpy(d_array, h_array, 256 * sizeof(int), hipMemcpyHostToDevice);`.
   e. (Optional) Verify by copying back to a second host array and printing a few values to confirm.
   f. Clean up: delete host array, hipFree device memory.
4. Return 0.
5. Add error handling after each CUDA call using `CUDA_CHECK`.
6. Ensure the code compiles as a .cu file and uses standard CUDA runtime API.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                          \
    do {                                                          \
        hipError_t err = call;                                   \
        if (err != hipSuccess) {                                 \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",     \
                    #call, __FILE__, __LINE__,                   \
                    hipGetErrorString(err));                     \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    } while (0)

int main() {
    const int N = 256;
    int *h_array = new int[N];
    if (!h_array) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return EXIT_FAILURE;
    }

    // Initialize host array with values 0..255
    for (int i = 0; i < N; ++i) {
        h_array[i] = i;
    }

    // Allocate device memory
    int *d_array;
    CUDA_CHECK(hipMalloc((void**)&d_array, N * sizeof(int)));

    // Copy from host to device
    CUDA_CHECK(hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice));

    // (Optional) Verify by copying back to host
    int *h_copy = new int[N];
    CUDA_CHECK(hipMemcpy(h_copy, d_array, N * sizeof(int), hipMemcpyDeviceToHost));

    // Print first 10 values to confirm
    printf("First 10 values after copy back from device:\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d ", h_copy[i]);
    }
    printf("\n");

    // Clean up
    delete[] h_array;
    delete[] h_copy;
    CUDA_CHECK(hipFree(d_array));

    return EXIT_SUCCESS;
}
```