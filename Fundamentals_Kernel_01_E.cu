/*
Aim of the program (exact wording from the prompt):
Add `cudaDeviceSynchronize()` after the launch and print 'Kernel synchronized!' after it.

Thinking and reasoning about the implementation:
- The prompt asks to modify a CUDA program by adding a synchronization call after a kernel launch and printing a message afterwards.
- Since no original code is provided, I will write a minimal complete CUDA program that demonstrates this behaviour.
- The program will:
  1. Allocate a small array on the host.
  2. Copy it to device memory.
  3. Launch a simple kernel that increments each element.
  4. Immediately after the launch, call `cudaDeviceSynchronize()` to wait for the kernel to finish.
  5. Print "Kernel synchronized!" to indicate the sync was successful.
  6. Copy the results back and print them for verification.
- All error checks are minimal but sufficient for a small demo.
- The code is self‑contained and can be compiled with `nvcc`.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_one(int *d_arr, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_arr[idx] += 1;
    }
}

int main(void) {
    const int N = 10;
    int h_arr[N];
    for (int i = 0; i < N; ++i) h_arr[i] = i;

    int *d_arr = NULL;
    size_t size = N * sizeof(int);
    hipError_t err;

    err = hipMalloc((void **)&d_arr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy H2D failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return 1;
    }

    // Launch kernel: one block with N threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add_one<<<blocksPerGrid, threadsPerBlock>>>(d_arr, N);

    // Add the requested synchronization and print statement
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return 1;
    }
    printf("Kernel synchronized!\n");

    err = hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy D2H failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return 1;
    }

    printf("Resulting array: ");
    for (int i = 0; i < N; ++i) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}
