/*
Aim of the program: Create two timers to measure two different parts of your program sequentially.

Thinking:
- Use CUDA events for GPU timing because they provide accurate timing of device operations.
- First timer will measure the time taken for host-to-device memory copy (cudaMemcpy).
- Second timer will measure the execution time of a simple kernel that processes data on the GPU.
- Events are recorded before and after the operation, then cudaEventElapsedTime gives the elapsed time in milliseconds.
- Since the operations are sequential, no need for overlapping or streams; single stream suffices.
- Use a simple kernel that increments each element of an array.
- Allocate memory on host and device, initialize host array, copy to device, run kernel, copy back, and cleanup.
- Print elapsed times for both parts.

Potential error handling:
- Use CUDA_CHECK macro to check return status and abort on failure.
- Ensure all resources are freed.

We'll write a minimal, self-contained CUDA program that demonstrates this. 
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err)                                        \
    do {                                                       \
        hipError_t err__ = (err);                             \
        if (err__ != hipSuccess) {                            \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",        \
                    __FILE__, __LINE__, hipGetErrorString(err__)); \
            exit(EXIT_FAILURE);                                \
        }                                                      \
    } while (0)

__global__ void incKernel(int *d_arr, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_arr[idx] += 1;
    }
}

int main(void) {
    const int N = 1 << 20;  // 1M elements
    size_t size = N * sizeof(int);

    // Host memory allocation and initialization
    int *h_arr = (int *)malloc(size);
    if (!h_arr) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }
    for (int i = 0; i < N; ++i) {
        h_arr[i] = 0;
    }

    // Device memory allocation
    int *d_arr;
    CUDA_CHECK(hipMalloc((void **)&d_arr, size));

    // CUDA events for timing
    hipEvent_t startMem, stopMem;
    hipEvent_t startKernel, stopKernel;
    CUDA_CHECK(hipEventCreate(&startMem));
    CUDA_CHECK(hipEventCreate(&stopMem));
    CUDA_CHECK(hipEventCreate(&startKernel));
    CUDA_CHECK(hipEventCreate(&stopKernel));

    // Measure host-to-device memcpy
    CUDA_CHECK(hipEventRecord(startMem, 0));
    CUDA_CHECK(hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipEventRecord(stopMem, 0));
    CUDA_CHECK(hipEventSynchronize(stopMem));

    float memTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&memTime, startMem, stopMem));

    // Launch kernel
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    CUDA_CHECK(hipEventRecord(startKernel, 0));
    incKernel<<<gridSize, blockSize>>>(d_arr, N);
    CUDA_CHECK(hipEventRecord(stopKernel, 0));
    CUDA_CHECK(hipEventSynchronize(stopKernel));

    float kernelTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&kernelTime, startKernel, stopKernel));

    // Optional: copy results back to host
    CUDA_CHECK(hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost));

    // Print timing results
    printf("Host to Device memcpy time:   %.3f ms\n", memTime);
    printf("Kernel execution time:        %.3f ms\n", kernelTime);

    // Clean up
    free(h_arr);
    CUDA_CHECK(hipFree(d_arr));
    CUDA_CHECK(hipEventDestroy(startMem));
    CUDA_CHECK(hipEventDestroy(stopMem));
    CUDA_CHECK(hipEventDestroy(startKernel));
    CUDA_CHECK(hipEventDestroy(stopKernel));

    return 0;
}
