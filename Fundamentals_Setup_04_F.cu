```cpp
/*
Aim: Query and print the `computeMode` of the primary GPU (e.g., Default, Exclusive, Prohibited).

Thinking:
1. The CUDA runtime API provides `hipGetDeviceProperties` to retrieve properties of a device, including the `computeMode` field.
2. `hipDeviceProp_t::computeMode` is an enum of type `hipComputeMode`, with possible values:
   - `hipComputeModeDefault` (0)
   - `hipComputeModeExclusiveProcess` (1)
   - `cudaComputeModeExclusiveThread` (2)
   - `hipComputeModeProhibited` (3)
3. We will query the first device (device 0) which is typically the primary GPU.
4. Use a helper macro `CHECK_CUDA_CALL` to handle errors from CUDA API calls gracefully.
5. Map the enum value to a human-readable string and print it to standard output.
6. The program is self‑contained and can be compiled with `nvcc` as a `.cu` file.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Helper macro for error checking
#define CHECK_CUDA_CALL(call)                                         \
    do {                                                              \
        hipError_t err = (call);                                     \
        if (err != hipSuccess) {                                     \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",              \
                    __FILE__, __LINE__, hipGetErrorString(err));    \
            return EXIT_FAILURE;                                     \
        }                                                             \
    } while (0)

// Function to convert compute mode enum to string
const char* computeModeToString(hipComputeMode mode) {
    switch (mode) {
        case hipComputeModeDefault:
            return "Default";
        case hipComputeModeExclusiveProcess:
            return "Exclusive Process";
        case cudaComputeModeExclusiveThread:
            return "Exclusive Thread";
        case hipComputeModeProhibited:
            return "Prohibited";
        default:
            return "Unknown";
    }
}

int main(void) {
    int device = 0; // primary GPU
    // Ensure the device exists
    int deviceCount = 0;
    CHECK_CUDA_CALL(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable device found.\n");
        return EXIT_FAILURE;
    }
    if (device >= deviceCount) {
        fprintf(stderr, "Device index %d out of range. Only %d device(s) available.\n",
                device, deviceCount);
        return EXIT_FAILURE;
    }

    hipDeviceProp_t prop;
    CHECK_CUDA_CALL(hipGetDeviceProperties(&prop, device));

    const char* modeStr = computeModeToString(prop.computeMode);
    printf("Compute mode of primary GPU (device %d) is: %s\n", device, modeStr);

    return EXIT_SUCCESS;
}
```