```cpp
/*
Aim: Query and print the `computeMode` of the primary GPU (e.g., Default, Exclusive, Prohibited).

Thinking:
- The compute mode of a CUDA device can be obtained from the device properties using `hipGetDeviceProperties`. 
- The `hipDeviceProp_t` struct contains a field `computeMode` which indicates the compute mode for the device. 
- We will query device 0 (the primary GPU) by calling `hipSetDevice(0)` and then `hipGetDeviceProperties(&prop, 0)`. 
- The `computeMode` field is an enum of type `hipComputeMode` with values: 
    * `hipComputeModeDefault`
    * `hipComputeModeExclusive`
    * `hipComputeModeProhibited`
    * `hipComputeModeExclusiveProcess` (in newer CUDA versions)
- We will map these enum values to human‑readable strings for printing. 
- Basic error checking will be performed after each CUDA API call. 
- The program is written in C++ with the CUDA Runtime API and will compile to a .cu file.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

std::string computeModeToString(hipDeviceProp_t const &prop) {
    switch (prop.computeMode) {
        case hipComputeModeDefault:
            return "Default";
        case hipComputeModeExclusive:
            return "Exclusive";
        case hipComputeModeProhibited:
            return "Prohibited";
#ifdef hipComputeModeExclusiveProcess
        case hipComputeModeExclusiveProcess:
            return "ExclusiveProcess";
#endif
        default:
            return "Unknown";
    }
}

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "Failed to get device count: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return -1;
    }

    int device = 0; // primary GPU
    err = hipSetDevice(device);
    if (err != hipSuccess) {
        std::cerr << "Failed to set device " << device << ": " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        std::cerr << "Failed to get device properties for device " << device << ": " 
                  << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << "Device " << device << " (" << prop.name << "): Compute Mode = "
              << computeModeToString(prop) << std::endl;

    return 0;
}
```