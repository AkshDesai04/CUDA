#include "hip/hip_runtime.h"
```cpp
/*
After copying, use `hipFree` to release the device memory.
Thinking: 
We want to demonstrate allocating device memory, copying data to it, launching a kernel, copying back, and freeing device memory. 
We'll allocate an array on host, copy to device using hipMemcpy, launch a simple kernel that increments each element, copy back to host, print results, and finally free device memory with hipFree, as requested. We'll also free host memory with free(). 
We will include basic error checking for CUDA calls. 
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel that adds 1 to each element of the array
__global__ void incrementKernel(int *d_arr, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        d_arr[idx] += 1;
    }
}

int main(void)
{
    const int N = 10;
    const size_t size = N * sizeof(int);
    int *h_arr = (int *)malloc(size);
    if (h_arr == NULL)
    {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    // Initialize host array
    for (int i = 0; i < N; ++i)
    {
        h_arr[i] = i;
    }

    // Allocate device memory
    int *d_arr = NULL;
    hipError_t err = hipMalloc((void **)&d_arr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        free(h_arr);
        return EXIT_FAILURE;
    }

    // Copy from host to device
    err = hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy H2D failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        free(h_arr);
        return EXIT_FAILURE;
    }

    // Launch kernel
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    incrementKernel<<<blocksPerGrid, threadsPerBlock>>>(d_arr, N);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        free(h_arr);
        return EXIT_FAILURE;
    }

    // Wait for GPU to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        free(h_arr);
        return EXIT_FAILURE;
    }

    // Copy result back to host
    err = hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy D2H failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        free(h_arr);
        return EXIT_FAILURE;
    }

    // Print the results
    printf("Result after incrementing on device:\n");
    for (int i = 0; i < N; ++i)
    {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    // Free device memory
    err = hipFree(d_arr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(err));
        free(h_arr);
        return EXIT_FAILURE;
    }

    // Free host memory
    free(h_arr);

    return EXIT_SUCCESS;
}
```