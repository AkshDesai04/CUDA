/*
Ensure you call `cudaEventDestroy` to clean up the event handles.
The aim of this program is to demonstrate the proper creation, usage, and destruction of CUDA event handles for timing a simple vector addition kernel. I chose a straightforward vector addition as it provides a clear workload that can be parallelized, allows us to measure GPU execution time accurately with events, and showcases the necessity of cleaning up resources. The program allocates host pinned memory for efficient transfers, copies data to the device, launches the kernel, records timing events before and after the kernel launch, synchronizes, retrieves elapsed time, and finally frees all allocated memory and destroys the CUDA event objects with `cudaEventDestroy`. This illustrates the complete lifecycle of event usage, ensuring no resource leaks occur.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Error checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Simple vector addition kernel
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main(void) {
    const int N = 1 << 20;               // 1 million elements
    const size_t size = N * sizeof(float);

    // Allocate pinned host memory
    float *h_A, *h_B, *h_C;
    cudaCheckError(hipHostMalloc(&h_A, size, hipHostMallocDefault));
    cudaCheckError(hipHostMalloc(&h_B, size, hipHostMallocDefault));
    cudaCheckError(hipHostMalloc(&h_C, size, hipHostMallocDefault));

    // Initialize host data
    for (int i = 0; i < N; ++i) {
        h_A[i] = (float)i;
        h_B[i] = (float)(i * 2);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    cudaCheckError(hipMalloc(&d_A, size));
    cudaCheckError(hipMalloc(&d_B, size));
    cudaCheckError(hipMalloc(&d_C, size));

    // Copy data to device
    cudaCheckError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Kernel launch configuration
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    cudaCheckError(hipEventCreate(&start));
    cudaCheckError(hipEventCreate(&stop));

    // Record start event
    cudaCheckError(hipEventRecord(start, 0));

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    cudaCheckError(hipGetLastError());

    // Record stop event
    cudaCheckError(hipEventRecord(stop, 0));
    cudaCheckError(hipEventSynchronize(stop));

    // Compute elapsed time
    float milliseconds = 0.0f;
    cudaCheckError(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Kernel execution time: %f ms\n", milliseconds);

    // Copy result back to host
    cudaCheckError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Cleanup: free device memory
    cudaCheckError(hipFree(d_A));
    cudaCheckError(hipFree(d_B));
    cudaCheckError(hipFree(d_C));

    // Cleanup: free host memory
    cudaCheckError(hipHostFree(h_A));
    cudaCheckError(hipHostFree(h_B));
    cudaCheckError(hipHostFree(h_C));

    // Destroy CUDA events to release resources
    cudaCheckError(hipEventDestroy(start));
    cudaCheckError(hipEventDestroy(stop));

    return 0;
}
