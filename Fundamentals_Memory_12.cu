```cpp
/*
Write a C++ wrapper class for device memory. The constructor should accept a size, call `hipMalloc`, and store the pointer. The destructor should call `hipFree`. This is a basic implementation of the RAII (Resource Acquisition Is Initialization) principle for CUDA pointers.

Thought process:
- Need a class that manages a CUDA device pointer.
- Constructor takes a size (in bytes), calls hipMalloc, stores pointer and size.
- Destructor calls hipFree if pointer is not null.
- Prevent copy semantics (copy constructor and copy assignment deleted).
- Provide move semantics for convenience.
- Provide accessor methods to get raw pointer and size.
- Use error handling: if hipMalloc fails, throw std::runtime_error.
- Include necessary headers: <hip/hip_runtime.h>, <stdexcept>.
- Keep implementation in a .cu file; no external dependencies.
- Add a small test in main to demonstrate usage (optional but helpful for compilation).
*/

#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>
#include <cstdio>

// C++ wrapper for CUDA device memory using RAII
class CudaDeviceBuffer
{
public:
    // Constructor: allocate memory of given size (in bytes)
    explicit CudaDeviceBuffer(std::size_t size)
        : ptr_(nullptr), size_(size)
    {
        if (size_ == 0) {
            // Zero-sized allocation is a no-op; keep ptr_ nullptr
            return;
        }
        hipError_t err = hipMalloc(&ptr_, size_);
        if (err != hipSuccess) {
            throw std::runtime_error(std::string("hipMalloc failed: ") + hipGetErrorString(err));
        }
    }

    // Destructor: free device memory
    ~CudaDeviceBuffer()
    {
        if (ptr_) {
            hipError_t err = hipFree(ptr_);
            if (err != hipSuccess) {
                // In destructor, we cannot throw. Log the error.
                std::cerr << "hipFree failed: " << hipGetErrorString(err) << std::endl;
            }
        }
    }

    // Delete copy constructor and copy assignment
    CudaDeviceBuffer(const CudaDeviceBuffer&) = delete;
    CudaDeviceBuffer& operator=(const CudaDeviceBuffer&) = delete;

    // Move constructor
    CudaDeviceBuffer(CudaDeviceBuffer&& other) noexcept
        : ptr_(other.ptr_), size_(other.size_)
    {
        other.ptr_ = nullptr;
        other.size_ = 0;
    }

    // Move assignment
    CudaDeviceBuffer& operator=(CudaDeviceBuffer&& other) noexcept
    {
        if (this != &other) {
            // Free existing resource
            if (ptr_) {
                hipFree(ptr_);
            }
            ptr_ = other.ptr_;
            size_ = other.size_;
            other.ptr_ = nullptr;
            other.size_ = 0;
        }
        return *this;
    }

    // Accessors
    void* data() noexcept { return ptr_; }
    const void* data() const noexcept { return ptr_; }
    std::size_t size() const noexcept { return size_; }

private:
    void* ptr_;
    std::size_t size_;
};

// Example usage: allocate 1024 bytes on device
int main()
{
    try {
        CudaDeviceBuffer buf(1024);
        std::cout << "Allocated " << buf.size() << " bytes on device at " << buf.data() << std::endl;
        // Use the buffer as needed...
    } catch (const std::exception& e) {
        std::cerr << "Exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;
}
```