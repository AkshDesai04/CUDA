```c
/*
Query and print the multiprocessor count for the default device.

Thinking:
1. Include <hip/hip_runtime.h> for CUDA runtime API and <stdio.h> for printing.
2. Use hipGetDeviceCount to ensure at least one CUDA-capable device is present.
3. Pick device 0 as the default device.
4. Use hipGetDeviceProperties to retrieve hipDeviceProp_t struct for that device.
5. Print the multiProcessorCount field from the struct.
6. Add basic error checking after each CUDA API call, printing an error message and exiting if needed.
7. Keep the program minimal and self-contained, compiling with nvcc as a .cu file.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device count: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable devices found.\n");
        return 1;
    }

    int device = 0; // default device
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device properties: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Device %d: %s\n", device, prop.name);
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount);

    return 0;
}
```