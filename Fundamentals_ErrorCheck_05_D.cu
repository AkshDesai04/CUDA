/*
Write a program that checks `hipDeviceCanAccessPeer` with an invalid peer device index.

I started by planning a simple CUDA C program that will:
1. Query the number of CUDA devices with hipGetDeviceCount.
2. If no devices are present, print an error and exit.
3. Select device 0 for the test using hipSetDevice.
4. Construct an invalid peer index. The easiest way to guarantee an invalid index is to use a value equal to the device count (since valid indices range from 0 to count-1). I also considered negative indices but using count works on all systems.
5. Call hipDeviceCanAccessPeer with the selected device (0) and the invalid peer index.
6. The function returns a hipError_t; I will check if it equals hipSuccess. For an invalid peer it should return a non-success error code such as hipErrorInvalidDevice. I will print the returned error code and the corresponding error string using hipGetErrorString for clarity.
7. Finally, exit gracefully.

The program includes basic error checking for CUDA calls and prints informative messages so that the user sees what went wrong when the peer index is invalid.

*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err;

    // Get number of CUDA-capable devices
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found.\n");
        return 1;
    }

    // Select device 0
    int dev = 0;
    err = hipSetDevice(dev);
    if (err != hipSuccess) {
        fprintf(stderr, "hipSetDevice(%d) failed: %s\n", dev, hipGetErrorString(err));
        return 1;
    }

    // Construct an invalid peer index (deviceCount is out of range)
    int invalidPeer = deviceCount; // out-of-range index

    // Check peer access
    bool canAccess;
    err = hipDeviceCanAccessPeer(&canAccess, dev, invalidPeer);
    if (err != hipSuccess) {
        printf("hipDeviceCanAccessPeer returned error for invalid peer index %d: %s\n",
               invalidPeer, hipGetErrorString(err));
    } else {
        printf("Unexpected success: hipDeviceCanAccessPeer returned true for invalid peer index %d.\n",
               invalidPeer);
    }

    return 0;
}
