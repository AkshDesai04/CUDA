#include "hip/hip_runtime.h"
/*
Aim: Iterate through all devices and print the clock rate for each one.
Thinking: The program uses the CUDA runtime API to query the number of CUDA-capable devices, then loops over each device index, retrieves its properties via hipGetDeviceProperties, and prints the clockRate field. The code includes error handling for the API calls and prints informative messages to stderr if any call fails. The output includes the total number of devices and for each device, its name and clock rate in kHz. This program should be compiled with nvcc and run on a system with the CUDA toolkit installed.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Number of CUDA devices: %d\n", deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess) {
            fprintf(stderr, "hipGetDeviceProperties failed for device %d: %s\n", i, hipGetErrorString(err));
            continue;
        }

        printf("Device %d: %s, Clock rate: %d kHz\n", i, prop.name, prop.clockRate);
    }

    return 0;
}
