```cuda
/*
Write a loop that iterates through all available devices and calls `hipSetDevice` followed by `hipDeviceReset` for each one.

Thinking:
1. The goal is to iterate over every CUDA-capable device present in the system.
2. First, obtain the number of devices with hipGetDeviceCount().
3. Loop from 0 to count-1:
   - Call hipSetDevice(i) to select the device.
   - After setting the device, immediately call hipDeviceReset() to reset it.
4. Add basic error checking after each CUDA API call to handle failures gracefully.
5. Print messages to indicate progress and any errors.
6. The program uses the CUDA Runtime API, includes <hip/hip_runtime.h>, and compiles as a .cu file.
7. No external libraries or extra output are required. The program simply performs the loop and exits.
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error getting device count: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    if (deviceCount == 0)
    {
        printf("No CUDA devices found.\n");
        return EXIT_SUCCESS;
    }

    printf("Found %d CUDA device(s).\n", deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        printf("\n--- Device %d ---\n", dev);

        err = hipSetDevice(dev);
        if (err != hipSuccess)
        {
            fprintf(stderr, "hipSetDevice(%d) failed: %s\n", dev, hipGetErrorString(err));
            continue;  // Skip to next device
        }
        printf("Set device to %d successfully.\n", dev);

        err = hipDeviceReset();
        if (err != hipSuccess)
        {
            fprintf(stderr, "hipDeviceReset() failed on device %d: %s\n", dev, hipGetErrorString(err));
            continue;
        }
        printf("Device %d reset successfully.\n", dev);
    }

    printf("\nAll devices processed.\n");
    return EXIT_SUCCESS;
}
```