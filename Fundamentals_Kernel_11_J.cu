```cuda
/*
Aim of the program:
Define a struct with many members. Time the difference between copying the whole struct vs. using hipMemcpy on individual members (the former is much better).

Thinking:
- Create a large struct with several member arrays of different types to mimic a realistic data structure.
- Allocate this struct on both host and device.
- Initialize the host struct with some data.
- Measure the time to copy the entire struct in one hipMemcpy call.
- Measure the time to copy each member array separately using individual hipMemcpy calls.
- Use CUDA events for precise timing.
- Compare the elapsed times and print the results.
- Ensure proper error checking and cleanup.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,  \
                    hipGetErrorString(err));                                \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

// Define a large struct with many members
struct BigStruct {
    float  f[64];   // 64 floats
    double d[32];   // 32 doubles
    int    i[128];  // 128 ints
};

int main(void) {
    // Seed random number generator
    srand((unsigned)time(NULL));

    // Allocate and initialize host struct
    BigStruct h_struct;
    for (int idx = 0; idx < 64; ++idx) h_struct.f[idx] = (float)rand() / RAND_MAX;
    for (int idx = 0; idx < 32; ++idx) h_struct.d[idx] = (double)rand() / RAND_MAX;
    for (int idx = 0; idx < 128; ++idx) h_struct.i[idx] = rand();

    // Allocate device struct
    BigStruct* d_struct;
    CUDA_CHECK(hipMalloc((void**)&d_struct, sizeof(BigStruct)));

    // Events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    float wholeTimeMs = 0.0f;
    float memberTimeMs = 0.0f;

    // --- Copy whole struct in one go ---
    CUDA_CHECK(hipEventRecord(start, 0));
    CUDA_CHECK(hipMemcpy(d_struct, &h_struct, sizeof(BigStruct),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&wholeTimeMs, start, stop));

    // --- Copy each member separately ---
    CUDA_CHECK(hipEventRecord(start, 0));

    // Copy array f
    CUDA_CHECK(hipMemcpy(&d_struct->f, &h_struct.f, sizeof(h_struct.f),
                          hipMemcpyHostToDevice));
    // Copy array d
    CUDA_CHECK(hipMemcpy(&d_struct->d, &h_struct.d, sizeof(h_struct.d),
                          hipMemcpyHostToDevice));
    // Copy array i
    CUDA_CHECK(hipMemcpy(&d_struct->i, &h_struct.i, sizeof(h_struct.i),
                          hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&memberTimeMs, start, stop));

    // Print results
    printf("Time to copy whole struct (one hipMemcpy): %.3f ms\n", wholeTimeMs);
    printf("Time to copy members separately (three hipMemcpy): %.3f ms\n",
           memberTimeMs);

    // Cleanup
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_struct));

    return 0;
}
```