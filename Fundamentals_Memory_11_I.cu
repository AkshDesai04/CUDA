#include "hip/hip_runtime.h"
```cuda
/*
Try to use `hipMemcpyFromSymbol` to read the value of the constant variable back to the host.

My thinking:  
I need a minimal CUDA program that demonstrates how to copy data into a device constant memory
using `hipMemcpyToSymbol` and then copy it back to the host using `hipMemcpyFromSymbol`.  
A constant variable must be declared with `__constant__`.  
To change its value I use `hipMemcpyToSymbol` (since it is read‑only from kernels).  
After that I use `hipMemcpyFromSymbol` to read it back.  
Error checking helper will make debugging easier.  
The program prints the value read back, confirming that the round‑trip worked.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

#define CHECK(call)                                                     \
    do {                                                                \
        hipError_t err = call;                                         \
        if (err != hipSuccess) {                                       \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                 \
                    __FILE__, __LINE__, hipGetErrorString(err));       \
            exit(EXIT_FAILURE);                                         \
        }                                                               \
    } while (0)

// Device constant memory variable
__constant__ int d_const;

// Kernel that reads the constant variable (optional, just for demonstration)
__global__ void kernel_read_const(int *out) {
    *out = d_const;
}

int main(void) {
    // Host value to copy into constant memory
    int h_val = 42;

    // Copy host value to device constant memory
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_const), &h_val, sizeof(int), 0, hipMemcpyHostToDevice));

    // Allocate device memory for kernel output (optional)
    int *d_out;
    CHECK(hipMalloc(&d_out, sizeof(int)));

    // Launch kernel that reads the constant variable
    kernel_read_const<<<1,1>>>(d_out);
    CHECK(hipDeviceSynchronize());

    // Copy result from kernel back to host
    int h_out = 0;
    CHECK(hipMemcpyFromSymbol(&h_out, HIP_SYMBOL(d_const), sizeof(int), 0, hipMemcpyDeviceToHost));

    printf("Value copied to constant memory: %d\n", h_val);
    printf("Value read back via hipMemcpyFromSymbol: %d\n", h_out);

    // Optional: also read kernel output
    int kernel_out = 0;
    CHECK(hipMemcpy(&kernel_out, d_out, sizeof(int), hipMemcpyDeviceToHost));
    printf("Kernel read constant value: %d\n", kernel_out);

    // Clean up
    CHECK(hipFree(d_out));

    return 0;
}
```