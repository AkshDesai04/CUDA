/*
Aim of the program:
Write to a negative index, like `d_ptr[-1]`, and check the report.

Thinking:
- The goal is to deliberately trigger an invalid memory write on the GPU by accessing a negative index of a device pointer.
- We will allocate a small device array and launch a kernel with a single thread that performs the write `d_ptr[-1] = 42;`.
- Since this address is out of bounds, the CUDA runtime (or a memory checker like cuda-memcheck) should report an error.
- After launching the kernel, we call `cudaDeviceSynchronize()` to force the kernel to complete and propagate any errors.
- We then retrieve the last CUDA error using `cudaGetLastError()` and print it. If the write is caught, this should report an invalid device memory address or similar error.
- The program includes a simple error-checking macro for convenience and prints whether the kernel executed successfully or if an error occurred.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

// Simple macro to check CUDA API calls
#define CHECK_CUDA(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                    \
                    __FILE__, __LINE__, hipGetErrorString(err));          \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)

// Kernel that writes to a negative index
__global__ void write_negative_index(int *d_ptr) {
    // Intentionally write to an invalid negative index
    d_ptr[-1] = 42;
}

int main(void) {
    const size_t N = 10;               // Size of device array
    int *d_ptr = nullptr;

    // Allocate device memory
    CHECK_CUDA(hipMalloc((void**)&d_ptr, N * sizeof(int)));

    // Optional: initialize memory to zero
    CHECK_CUDA(hipMemset(d_ptr, 0, N * sizeof(int)));

    // Launch kernel with one block of one thread
    write_negative_index<<<1, 1>>>(d_ptr);

    // Synchronize to ensure kernel completion and propagate errors
    hipError_t sync_err = hipDeviceSynchronize();
    if (sync_err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize error: %s\n",
                hipGetErrorString(sync_err));
    }

    // Check for any kernel launch errors
    hipError_t last_err = hipGetLastError();
    if (last_err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n",
                hipGetErrorString(last_err));
    } else {
        printf("Kernel launched successfully (no launch error reported).\n");
    }

    // Clean up
    CHECK_CUDA(hipFree(d_ptr));

    // Final status
    if (sync_err != hipSuccess || last_err != hipSuccess) {
        printf("Negative index write caused an error as expected.\n");
    } else {
        printf("No error reported for negative index write (unexpected).\n");
    }

    return 0;
}
