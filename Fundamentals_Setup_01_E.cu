#include "hip/hip_runtime.h"
```cuda
/*
Aim of the program:
Query and print the memory clock rate of the primary CUDA device in kilohertz (kHz).

Thinking:
- The CUDA Runtime API provides hipGetDeviceProperties which fills a hipDeviceProp_t struct.
- The struct contains a member `memoryClockRate` that is reported in kilohertz.
- To target the primary device, we assume device ID 0 is the primary.
- We will perform basic error checking: ensure a CUDA device is present, and the query succeeds.
- The program will print the memory clock rate directly, as it is already in kHz.
- No kernel launches are required; this is a simple host-side query.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found.\n");
        return 1;
    }

    int deviceId = 0; // primary device
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, deviceId);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Memory Clock Rate of device %d (%s): %d kHz\n",
           deviceId, prop.name, prop.memoryClockRate);

    return 0;
}
```