```cpp
/*
 * Aim of the program: Write a program that allocates 10 small chunks of memory and stores their pointers in an array. Then, loop through the array to free them all.
 *
 * My thinking:
 * 1. The program will run on the host and use CUDA runtime API to allocate memory on the GPU.
 * 2. We'll declare an array of 10 pointers (e.g., `int* ptrs[10];`) to hold the device memory addresses.
 * 3. For each pointer, we call `hipMalloc` to allocate a small chunk (e.g., 100 bytes).
 * 4. We'll check the return status of each call and print an error message if allocation fails.
 * 5. After allocations, we'll iterate over the array again, calling `hipFree` for each pointer.
 * 6. We'll also check the status of `hipFree` and report any errors.
 * 7. Finally, we call `hipDeviceReset()` to clean up the device before exiting.
 *
 * The program uses the CUDA runtime API, so it should be compiled with `nvcc`.
 * No device kernels are launched; this is purely host code exercising memory allocation and deallocation.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void) {
    const int num_chunks = 10;
    const size_t chunk_size = 100;  // bytes
    int* ptrs[num_chunks];

    // Allocate memory chunks
    for (int i = 0; i < num_chunks; ++i) {
        hipError_t err = hipMalloc((void**)&ptrs[i], chunk_size);
        if (err != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for chunk %d: %s\n", i, hipGetErrorString(err));
            // Attempt to free any previously allocated chunks before exiting
            for (int j = 0; j < i; ++j) {
                hipFree(ptrs[j]);
            }
            return 1;
        } else {
            printf("Allocated chunk %d at device pointer %p\n", i, (void*)ptrs[i]);
        }
    }

    // Free memory chunks
    for (int i = 0; i < num_chunks; ++i) {
        hipError_t err = hipFree(ptrs[i]);
        if (err != hipSuccess) {
            fprintf(stderr, "hipFree failed for chunk %d: %s\n", i, hipGetErrorString(err));
            // Continue freeing the rest even if one fails
        } else {
            printf("Freed chunk %d\n", i);
        }
    }

    // Reset the device to clean up resources
    hipDeviceReset();
    return 0;
}
```