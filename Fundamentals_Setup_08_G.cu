```cuda
/*
Aim: Check if the device is a multi-GPU board (`isMultiGpuBoard`).

Thinking:
The CUDA Runtime API exposes an attribute called
`cudaDevAttrMultiGpuBoard` via `hipDeviceGetAttribute`.  
When this attribute is queried for a device, it returns 1 if the device
belongs to a multi‑GPU board and 0 otherwise.  We can use this attribute
to determine whether each device on the system is part of such a board.

The program below:
1. Uses `hipGetDeviceCount` to obtain the number of CUDA capable devices.
2. Iterates over each device index.
3. Calls `hipDeviceGetAttribute` with `cudaDevAttrMultiGpuBoard` to query
   the multi-GPU status.
4. Prints the result for each device.
5. Includes simple error checking for CUDA calls.

This code is self‑contained and can be compiled with `nvcc`:
    nvcc -o check_mgpu_board check_mgpu_board.cu
and then executed:
    ./check_mgpu_board
*/

#include <hip/hip_runtime.h>
#include <iostream>

// Helper macro for error checking
#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__      \
                      << " - " << hipGetErrorString(err) << std::endl;       \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

int main() {
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        std::cout << "No CUDA devices found." << std::endl;
        return 0;
    }

    std::cout << "Found " << deviceCount << " CUDA device(s)." << std::endl;

    for (int dev = 0; dev < deviceCount; ++dev) {
        int isMultiGpuBoard = 0;
        // Query the attribute
        CUDA_CHECK(hipDeviceGetAttribute(&isMultiGpuBoard,
                                          cudaDevAttrMultiGpuBoard,
                                          dev));

        std::cout << "Device " << dev << ": ";
        if (isMultiGpuBoard) {
            std::cout << "Is part of a multi-GPU board." << std::endl;
        } else {
            std::cout << "Is NOT part of a multi-GPU board." << std::endl;
        }
    }

    return 0;
}
```