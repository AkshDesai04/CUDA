#include "hip/hip_runtime.h"
```cpp
/*
If you are using CUDA events, wrap `hipEventCreate`, `hipEventRecord`, and `hipEventDestroy`.

Thinking process:
- The goal is to provide a set of wrapper functions around the core CUDA event APIs
  so that error handling, logging, or additional behavior can be added in a single place.
- Each wrapper should mirror the original API signature but add a check for the return
  status and report any errors via `hipGetErrorString`. This keeps the rest of the
  application code cleaner and more readable.
- To demonstrate usage, the program will launch a simple kernel that initializes
  an array, and use the wrapped events to time the kernel execution.
- The main program flow:
  1. Allocate device memory for an array.
  2. Create two events using the wrappers (`start` and `stop`).
  3. Record the start event, launch the kernel, record the stop event.
  4. Synchronize and calculate elapsed time.
  5. Destroy the events and free device memory.
- The program compiles as a standard CUDA .cu file and can be built with `nvcc`.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

// Wrapper for hipEventCreate with error checking
void myEventCreate(hipEvent_t *event, unsigned int flags = 0) {
    hipError_t err = hipEventCreateWithFlags(event, flags);
    if (err != hipSuccess) {
        std::cerr << "hipEventCreateWithFlags failed: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Wrapper for hipEventRecord with error checking
void myEventRecord(hipEvent_t event, hipStream_t stream = 0) {
    hipError_t err = hipEventRecord(event, stream);
    if (err != hipSuccess) {
        std::cerr << "hipEventRecord failed: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Wrapper for hipEventDestroy with error checking
void myEventDestroy(hipEvent_t event) {
    hipError_t err = hipEventDestroy(event);
    if (err != hipSuccess) {
        std::cerr << "hipEventDestroy failed: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Simple kernel that fills an array with sequential values
__global__ void initKernel(int *d_arr, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        d_arr[idx] = idx;
    }
}

int main() {
    const int N = 1 << 20; // 1M elements
    const size_t size = N * sizeof(int);

    // Allocate device memory
    int *d_arr = nullptr;
    hipError_t err = hipMalloc((void**)&d_arr, size);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        return EXIT_FAILURE;
    }

    // Create CUDA events using wrappers
    hipEvent_t start, stop;
    myEventCreate(&start);
    myEventCreate(&stop);

    // Record start event
    myEventRecord(start);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    initKernel<<<blocksPerGrid, threadsPerBlock>>>(d_arr, N);

    // Record stop event
    myEventRecord(stop);

    // Wait for the events to complete
    hipDeviceSynchronize();

    // Calculate elapsed time
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Kernel execution time: " << ms << " ms" << std::endl;

    // Clean up
    myEventDestroy(start);
    myEventDestroy(stop);
    hipFree(d_arr);

    return 0;
}
```