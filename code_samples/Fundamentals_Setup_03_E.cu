/*
Query and print the maximum memory pitch in bytes allowed for memory allocations.
 
In order to satisfy this request I need to query the CUDA device properties and
extract the field that holds the maximum allowed pitch for 2‑D memory
allocations. The CUDA runtime API provides a structure
`hipDeviceProp_t` that contains a member `maxPitch`.  I can obtain the
current device with `hipGetDevice`, fill a `hipDeviceProp_t` with
`hipGetDeviceProperties`, and then print the value.  
To make the program robust I will check the return status of the CUDA
calls and report errors using `hipGetErrorString`.  
The program will be a self‑contained `.cu` file that can be compiled with
`nvcc`.  The `printf` function from `<stdio.h>` will be used to display the
result.  No additional CUDA kernels or memory operations are required,
because the goal is purely to query and display the maximum pitch value.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int dev = 0;
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipGetDevice failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, dev);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipGetDeviceProperties failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Maximum memory pitch for device %d is %llu bytes.\n", dev, (unsigned long long)prop.maxPitch);

    return 0;
}
