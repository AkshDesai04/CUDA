```cpp
/*
Iterate through all devices and print the `concurrentKernels` status for each one.

Thinking:
1. Use hipGetDeviceCount to determine how many CUDA-capable devices are present.
2. Iterate from device 0 to count-1.
3. For each device, call hipGetDeviceProperties to obtain a hipDeviceProp_t struct.
4. The field `concurrentKernels` indicates whether concurrent kernel execution is supported.
   It is an int: 1 if supported, 0 otherwise.
5. Print the device index and the status in a readable form.
6. Handle any CUDA errors by printing an error message and exiting.
7. Ensure the program can be compiled with nvcc and runs on any system with CUDA installed.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device count: %s\n", hipGetErrorString(err));
        return -1;
    }

    if (deviceCount == 0) {
        printf("No CUDA-capable devices detected.\n");
        return 0;
    }

    printf("Detected %d CUDA-capable device(s).\n", deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to get properties for device %d: %s\n", dev, hipGetErrorString(err));
            continue;
        }

        printf("Device %d: \"%s\"\n", dev, prop.name);
        printf("  concurrentKernels support: %s\n", prop.concurrentKernels ? "YES" : "NO");
    }

    return 0;
}
```