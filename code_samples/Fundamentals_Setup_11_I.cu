#include "hip/hip_runtime.h"
```c
/*
Add the `-arch=sm_XX` flag to your `nvcc` command line to compile for a specific compute capability (e.g., `sm_75`).

=== Thinking Process ===
1. The user wants a CUDA .cu file that demonstrates how to compile with a specific architecture flag.
2. The code should be self‑contained and compilable. A simple vector addition kernel is a good example.
3. The comment should contain the exact aim string, followed by my reasoning.
4. After the comment, the actual CUDA code will perform a basic vector addition on the GPU.
5. The code will allocate host and device memory, launch the kernel, copy results back, and print a few values.
6. The user explicitly asked for no other supporting text, only the .cu file content.
*/

#include <cstdio>
#include <cstdlib>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main()
{
    const int N = 1 << 20;           // 1M elements
    const size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print a few results for verification
    for (int i = 0; i < 10; ++i) {
        printf("C[%d] = %f\n", i, h_C[i]);
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
```