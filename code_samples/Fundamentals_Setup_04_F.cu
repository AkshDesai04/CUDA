/*
Query and print the computeMode of the primary GPU (e.g., Default, Exclusive, Prohibited).

Thought process:
- The goal is to determine the compute mode set on the device that the CUDA runtime
  considers primary (the device that the program is currently running on).  
- In CUDA, each device has a compute mode that can be queried via `hipGetDeviceProperties`,
  which returns a `hipDeviceProp_t` structure containing a `computeMode` field.
- The `hipComputeMode` enum includes values: `hipComputeModeDefault`, `hipComputeModeExclusive`,
  `hipComputeModeProhibited`, and `hipComputeModeExclusiveProcess`.  
- The program will:
  1. Retrieve the current device ID using `hipGetDevice`.
  2. Query the device properties with `hipGetDeviceProperties`.
  3. Map the enum value to a readable string.
  4. Print the result along with the device name and ID.
- Error handling will be added to report any failures from CUDA API calls.
- The code is written in standard C++ with CUDA runtime API and will be compiled as a
  `.cu` file. No additional headers beyond `<hip/hip_runtime.h>` and `<iostream>` are required.

*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

std::string computeModeToString(hipComputeMode mode) {
    switch (mode) {
        case hipComputeModeDefault:          return "Default";
        case hipComputeModeExclusive:        return "Exclusive";
        case hipComputeModeProhibited:       return "Prohibited";
        case hipComputeModeExclusiveProcess: return "ExclusiveProcess";
        default:                              return "Unknown";
    }
}

int main() {
    int device = 0;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        std::cerr << "Failed to get current device: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        std::cerr << "Failed to get device properties: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "Compute mode of device " << device << " (" << prop.name << "): "
              << computeModeToString(prop.computeMode) << std::endl;

    return 0;
}
