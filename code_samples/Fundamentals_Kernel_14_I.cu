A CUDA API call can fail in two fundamentally different ways: **synchronously** or **asynchronously**.

| Aspect | Synchronous error | Asynchronous error |
|--------|-------------------|--------------------|
| **When it occurs** | At the time the API function returns. The host thread sees the error immediately. | After the function returns; the error surfaces only when the device is next accessed from the host (e.g., a later `hipMemcpy` or `hipDeviceSynchronize`). |
| **Typical source** | Errors that can be determined on the host side: memory allocation failure (`hipMalloc`), invalid arguments (`hipMemcpy` with wrong pointer, wrong size), context‑creation errors, etc. | Errors that require the device to execute work: kernel launch failures due to bad grid/block dimensions, illegal memory accesses inside a kernel, or other runtime errors that manifest only during execution. |
| **Error codes** | Returned directly by the API call (`hipError_t`). | The call returns `hipSuccess`, but the device state is marked as *failed*. The next API call that checks the device state will return a non‑zero error. |
| **Recovery** | The error can be handled immediately; you can free resources, retry, or exit. | You must call a *synchronizing* function (e.g., `hipDeviceSynchronize`, `hipMemcpy`, or `hipGetLastError`) to detect the error. Once detected, the context may be in an unrecoverable state, so you often need to reset the device or terminate. |
| **Examples** | `hipMalloc(&ptr, size)` fails if `size` is too large or the device has no free memory. | Launching a kernel with `<<<blocks, threads>>>()` where `threads > 1024` or `blocks > maxBlocks` may still return `hipSuccess` immediately, but the launch will fail internally. The error is only seen on a subsequent sync or query. |
| **Timing** | Immediate (host side). | Delayed; depends on the device’s completion of the queued work. |

### How to detect each

* **Synchronous**  
  ```c
  hipError_t err = hipMalloc(&ptr, size);
  if (err != hipSuccess) {
      // handle error right away
  }
  ```

* **Asynchronous**  
  ```c
  kernel<<<blocks, threads>>>(...);   // may launch "silently"
  hipError_t err = hipGetLastError();   // checks the *last* launch error
  if (err != hipSuccess) { ... }

  // or
  err = hipDeviceSynchronize();   // waits for all work to finish
  if (err != hipSuccess) { ... }
  ```

In short, synchronous errors are caught immediately on the host because the failure can be detected without waiting for the GPU. Asynchronous errors are deferred until the device has an opportunity to process the request, so you need to explicitly synchronize or query the device to surface them.