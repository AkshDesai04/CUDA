```cpp
/*
Aim of the program: Write a version of the function for streams: `checkStream(hipStream_t stream)` which calls `hipStreamSynchronize`.

Thinking:
- The task is to provide a CUDA helper function that synchronizes a given CUDA stream.
- We need to include the necessary CUDA headers (`<hip/hip_runtime.h>`) and standard I/O for error reporting (`<cstdio>`).
- The function `checkStream` should take a `hipStream_t` as its argument.
- Inside the function, call `hipStreamSynchronize(stream)` and capture the returned `hipError_t`.
- If the returned error code is not `hipSuccess`, print an error message to `stderr` using `hipGetErrorString`.
- The function does not need to return a value; it performs the synchronization and reports errors if any.
- This file is a complete `.cu` file ready to be compiled with nvcc.
*/

#include <hip/hip_runtime.h>
#include <cstdio>

// Synchronizes the specified CUDA stream and reports errors if they occur.
void checkStream(hipStream_t stream) {
    hipError_t err = hipStreamSynchronize(stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error synchronizing stream: %s\n", hipGetErrorString(err));
    }
}
```