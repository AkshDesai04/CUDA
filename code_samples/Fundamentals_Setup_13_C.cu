#include "hip/hip_runtime.h"
```c
/*
Aim of the program:
Iterate through all devices and print the clock rate for each one.

Thinking:
To accomplish this, I will use the CUDA Runtime API. First, I need to include
the necessary headers: <stdio.h> for I/O and <hip/hip_runtime.h> for CUDA
runtime functions. I will then call hipGetDeviceCount to find out how many
GPU devices are available. For each device index, I will retrieve a
hipDeviceProp_t structure using hipGetDeviceProperties. The clock rate is
stored in the 'clockRate' member of that structure, measured in kHz. I will
convert it to MHz for easier reading. Finally, I will print the device
index along with its clock rate. The program will be a simple C-style
main function that returns 0 on success. This code will be compiled with
nvcc as a .cu file.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Error getting device count: %s\n", hipGetErrorString(err));
        return -1;
    }

    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
        return 0;
    }

    printf("Found %d CUDA-capable device(s).\n", deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Error getting properties for device %d: %s\n",
                    dev, hipGetErrorString(err));
            continue;
        }

        // Clock rate is in kHz; convert to MHz
        float clockMHz = prop.clockRate / 1000.0f;
        printf("Device %d: Clock Rate = %.2f MHz\n", dev, clockMHz);
    }

    return 0;
}
```