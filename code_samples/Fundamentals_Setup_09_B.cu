```cuda
/*
Aim of the program: Query and print the `asyncEngineCount` property, which indicates the number of engines available for overlapping data copies and kernel execution.

Thinking:
1. Use the CUDA runtime API to obtain the number of CUDA-capable devices via hipGetDeviceCount.
2. For each device, retrieve the device properties using hipGetDeviceProperties.
3. Extract the asyncEngineCount field from the hipDeviceProp_t structure.
4. Print the device index, name, and asyncEngineCount to standard output.
5. Include error checking after each CUDA API call to handle potential failures gracefully.
6. The program is written in C with CUDA extensions (.cu) and can be compiled with `nvcc`.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Error retrieving device count: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
        return 0;
    }

    printf("Number of CUDA devices: %d\n", deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Error retrieving properties for device %d: %s\n", dev, hipGetErrorString(err));
            continue;
        }

        printf("\nDevice %d: \"%s\"\n", dev, prop.name);
        printf("  asyncEngineCount: %d\n", prop.asyncEngineCount);
    }

    return 0;
}
```