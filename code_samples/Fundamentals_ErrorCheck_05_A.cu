/*
Aim: First, get the device count. Then, try to set the device to `count` (which is an invalid index, since they are 0-based).

Thinking:
1. Include necessary headers: <cuda_runtime.h> for CUDA runtime API and <stdio.h> for I/O.
2. In main, call cudaGetDeviceCount(&count) to retrieve the number of available CUDA devices.
3. Print the device count for visibility.
4. Attempt to set the device using cudaSetDevice(count). Since valid device indices are 0 to count-1, using count should trigger an error (cudaErrorInvalidDevice).
5. Capture the return value of cudaSetDevice and print an error message if it fails, using cudaGetErrorString to obtain a human-readable description.
6. Optionally, call cudaGetLastError to clear any pending error (though not strictly necessary for this simple demo).
7. Exit with appropriate return code based on success or failure.
8. Compile with nvcc, e.g., nvcc -o invalid_device invalid_device.cu
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err;

    // Get the number of CUDA-capable devices
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Number of CUDA devices: %d\n", deviceCount);

    // Attempt to set the device to an invalid index (count)
    printf("Attempting to set device to index %d (invalid index)...\n", deviceCount);
    err = hipSetDevice(deviceCount);
    if (err != hipSuccess) {
        printf("hipSetDevice returned error: %s\n", hipGetErrorString(err));
    } else {
        printf("hipSetDevice succeeded (unexpected).\n");
    }

    // Optionally, clear any error state
    hipGetLastError();

    return 0;
}
