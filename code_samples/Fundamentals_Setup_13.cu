#include "hip/hip_runtime.h"
/*
Write a program that queries and prints the clock rate of the primary GPU in kilohertz (kHz).

Thinking process:

1. The goal is to use CUDA Runtime API to retrieve the clock rate of the primary GPU.
2. The primary GPU is usually device 0; we can use hipGetDevice to confirm current device or simply use 0.
3. Use `hipGetDeviceProperties` to fill a `hipDeviceProp_t` structure.
4. The `clockRate` member of `hipDeviceProp_t` holds the clock rate in kHz.
5. Print the value using `printf` with a clear message.
6. Include basic error checking: verify the return status of `hipGetDeviceProperties`.
7. Compile with `nvcc` and run on a system with CUDA installed.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

int main(void) {
    int device = 0;  // Primary GPU
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Error getting device properties: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Primary GPU (%d) clock rate: %d kHz\n", device, prop.clockRate);
    return 0;
}
