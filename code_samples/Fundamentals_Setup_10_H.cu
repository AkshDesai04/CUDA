#include "hip/hip_runtime.h"
/*
Query and print the GPU's clock rate in GHz.

Thought Process:
1. The goal is to write a CUDA program in C that retrieves the GPU's clock rate and prints it in GHz.
2. CUDA provides the `hipGetDeviceProperties` function which fills a `hipDeviceProp_t` struct that contains the `clockRate` field in kHz.
3. The program will:
   - Include `<stdio.h>` for I/O and `<hip/hip_runtime.h>` for CUDA runtime functions.
   - Use `hipGetDeviceCount` to confirm that at least one device exists.
   - Choose device 0 for simplicity.
   - Call `hipGetDeviceProperties` for that device.
   - Convert the `clockRate` from kHz to GHz: `clockRate_GHz = clockRate_kHz / 1e6`.
   - Print the result with a reasonable precision.
4. Error handling will be performed by checking the return status of CUDA API calls.
5. The program will compile with `nvcc` and run on a system with a CUDA-capable GPU.

This file is a complete CUDA C source that can be compiled as a `.cu` file.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device count: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable devices found.\n");
        return 1;
    }

    int device = 0;  // Use the first device
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get properties for device %d: %s\n", device, hipGetErrorString(err));
        return 1;
    }

    // clockRate is in kHz; convert to GHz
    double clockGhz = prop.clockRate / 1e6;

    printf("GPU (%s) clock rate: %.3f GHz\n", prop.name, clockGhz);

    return 0;
}
