#include "hip/hip_runtime.h"
```cpp
/*
Launch the infinite loop kernel on a non-default stream. Does it still hang the application when you call `hipDeviceSynchronize()`?

Thinking:
- The purpose of this program is to demonstrate that launching a kernel that runs forever (infinite loop) on a stream other than the default stream (stream 0) will still cause a call to `hipDeviceSynchronize()` to block the host indefinitely.
- We create a non-default stream with `hipStreamCreate()`, launch a kernel that never exits (`while(true){}`) into that stream, and then call `hipDeviceSynchronize()`.
- Since the kernel is stuck in an infinite loop, the device will never finish the kernel, so `hipDeviceSynchronize()` will never return. The program will hang at that point, confirming that the stream property does not affect the blocking behavior of `hipDeviceSynchronize()`.
- We include basic error checking for stream creation, kernel launch, and the synchronize call, but the program will not reach the final print because the sync never returns.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

// Kernel that runs an infinite loop
__global__ void infinite_loop()
{
    while (true)
    {
        // Do nothing, just spin
    }
}

// Helper macro for checking CUDA errors
#define CHECK_CUDA(call)                                          \
    do {                                                          \
        hipError_t err = call;                                   \
        if (err != hipSuccess) {                                 \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",           \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    } while (0)

int main()
{
    printf("Creating a non-default stream...\n");
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    printf("Launching infinite loop kernel on the non-default stream...\n");
    infinite_loop<<<1, 1, 0, stream>>>();
    CHECK_CUDA(hipGetLastError());

    printf("Kernel launched. Calling hipDeviceSynchronize() to wait for completion...\n");
    CHECK_CUDA(hipDeviceSynchronize()); // This call will block forever

    // This line will never be reached
    printf("This message will not be printed because the application hangs at synchronize.\n");

    // Clean up (unreachable in practice)
    hipStreamDestroy(stream);
    return 0;
}
```