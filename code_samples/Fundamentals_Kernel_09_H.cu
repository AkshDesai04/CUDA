/*
Create a `__device__` function that takes a pointer as an argument.

Thinking:
- The goal is to illustrate a __device__ function that accepts a pointer parameter.
- We define a device function that writes a value through a pointer.
- A simple kernel will call this function.
- The host will allocate memory, launch the kernel, copy back the result, and print it.
- Error checking is included for robustness.
*/


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define CUDA_CHECK(call)                                                     \
    do {                                                                     \
        hipError_t err = call;                                              \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",                \
                    __func__, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                             \
        }                                                                    \
    } while (0)

// Device function that takes a pointer and writes a value to the location it points to
__device__ void write_value(int *ptr, int val) {
    *ptr = val;
}

// Kernel that calls the device function
__global__ void kernel(int *d_arr) {
    // For simplicity, use only one thread
    write_value(d_arr, 42);
}

int main() {
    const size_t N = 1;
    int *h_arr = (int*)malloc(N * sizeof(int));
    int *d_arr = nullptr;

    CUDA_CHECK(hipMalloc((void**)&d_arr, N * sizeof(int)));

    // Launch kernel
    kernel<<<1, 1>>>(d_arr);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost));

    printf("Value after kernel execution: %d\n", h_arr[0]);

    // Clean up
    CUDA_CHECK(hipFree(d_arr));
    free(h_arr);

    return 0;
}
