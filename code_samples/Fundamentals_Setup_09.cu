```cpp
/* 
Aim of the program:
Write a program to check and print whether the primary GPU supports running multiple kernels concurrently. 
The property you are looking for is concurrentKernels in hipDeviceProp_t.

Thinking:
1. Use CUDA Runtime API to query device properties.
2. First, get the number of CUDA devices via hipGetDeviceCount.
3. If at least one device is present, select the primary device (device 0).
4. Call hipGetDeviceProperties to fill a hipDeviceProp_t structure.
5. Inspect the concurrentKernels field of the structure; if it is non-zero, the GPU can run multiple kernels concurrently.
6. Print a user-friendly message indicating support or lack thereof.
7. Include error checking after each CUDA API call for robustness.
8. Compile with nvcc to produce a .cu file that can be executed on a machine with a CUDA-capable GPU.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        printf("No CUDA-capable device found.\n");
        return 0;
    }

    int device = 0; // Primary device
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed for device %d: %s\n", device, hipGetErrorString(err));
        return 1;
    }

    printf("Device %d: %s\n", device, prop.name);
    if (prop.concurrentKernels) {
        printf("This GPU supports running multiple kernels concurrently.\n");
    } else {
        printf("This GPU does NOT support running multiple kernels concurrently.\n");
    }

    return 0;
}
```