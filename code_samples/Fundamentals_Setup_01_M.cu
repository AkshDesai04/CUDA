```c
/*
Aim of the program:
Query and print the major and minor compute capability numbers for the primary CUDA device separately.

Thinking:
- The primary CUDA device is the one currently selected by the CUDA runtime.
- We can obtain its ID using hipGetDevice.
- The device properties, including compute capability, are available via hipGetDeviceProperties.
- The major and minor fields of hipDeviceProp_t hold the compute capability numbers.
- We'll print them using printf, with proper error checking for CUDA API calls.
- The program will be a simple console application that includes <hip/hip_runtime.h> and <stdio.h>.
- It will handle potential errors by printing the CUDA error string and exiting with a non-zero status.
- No additional features or command-line arguments are required.
- The code is written in C with CUDA extensions, suitable for compilation with nvcc.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(void)
{
    int device = 0;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        fprintf(stderr, "Error getting current device: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Error getting device properties: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    printf("Compute Capability: Major = %d, Minor = %d\n", prop.major, prop.minor);

    return EXIT_SUCCESS;
}
```