```cuda
/*
Aim: Write a program that deliberately triggers a hipErrorInvalidValue by passing a nullptr as the destination for hipMemcpy. Use the CHECK() macro to catch and report the error cleanly.

Thinking:
- The CHECK macro should wrap a CUDA API call, evaluate its return status, and if it indicates an error, print the error string and exit the program. This keeps error handling concise.
- To trigger hipErrorInvalidValue, we will call hipMemcpy with a null destination pointer. The source can be a small buffer allocated on the host; the size and kind can be anything valid, but the null dest will make the API fail.
- We need to include the necessary headers: <cstdio> for printf, <cstdlib> for exit, and <hip/hip_runtime.h> for CUDA runtime API.
- The program will allocate a small host buffer, then call hipMemcpy(host_ptr, nullptr, 4, hipMemcpyHostToDevice). This should produce an error. The CHECK macro will catch this and print a message.
- We also ensure we free any allocated host memory before exiting (although the program will abort upon error).
- The code will be a single .cu file ready to compile with nvcc.
*/

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define CHECK(call)                                                      \
    do {                                                                 \
        hipError_t err = (call);                                        \
        if (err != hipSuccess) {                                       \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",            \
                    #call, __FILE__, __LINE__, hipGetErrorString(err));\
            exit(EXIT_FAILURE);                                         \
        }                                                                \
    } while (0)

int main(void) {
    const int N = 4;
    int *h_ptr = (int*)malloc(N * sizeof(int));
    if (!h_ptr) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }
    // Initialize host data
    for (int i = 0; i < N; ++i) {
        h_ptr[i] = i;
    }

    // Intentionally pass nullptr as destination to trigger hipErrorInvalidValue
    CHECK(hipMemcpy(nullptr, h_ptr, N * sizeof(int), hipMemcpyHostToDevice));

    // Clean up (will not be reached due to CHECK aborting)
    free(h_ptr);
    return 0;
}
```