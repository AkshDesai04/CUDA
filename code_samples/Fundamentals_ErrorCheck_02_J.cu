#include "hip/hip_runtime.h"
```cuda
/*
Aim: Add the `CHECK` macro around a call to `hipGetLastError()`.

My thinking:
- The goal is to demonstrate the use of a `CHECK` macro that wraps a CUDA API call,
  specifically `hipGetLastError()`.  
- The macro should capture the result of the call, compare it to `hipSuccess`,
  and if an error occurs, print a helpful message including the file name,
  line number, and the CUDA error string, then exit the program.
- To illustrate usage, I will create a simple CUDA kernel that does nothing,
  launch it, and then immediately call `CHECK(hipGetLastError())`.  
- I will also call `CHECK(hipDeviceSynchronize())` to catch any errors that
  might arise during kernel execution.  
- The program will compile to a `.cu` file and can be built with `nvcc`.

The complete code follows:
*/

#include <stdio.h>
#include <stdlib.h>

// CHECK macro to wrap CUDA API calls
#define CHECK(call)                                                     \
    do {                                                                \
        hipError_t err = (call);                                       \
        if (err != hipSuccess) {                                       \
            fprintf(stderr, "CUDA error at %s:%d - %s\n",                \
                    __FILE__, __LINE__, hipGetErrorString(err));       \
            exit(EXIT_FAILURE);                                         \
        }                                                               \
    } while (0)

// A trivial kernel that does nothing
__global__ void noopKernel(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] = data[idx];
    }
}

int main(void) {
    const int N = 256;
    int *d_data = NULL;

    // Allocate device memory
    CHECK(hipMalloc((void**)&d_data, N * sizeof(int)));

    // Launch kernel
    int threadsPerBlock = 64;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    noopKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, N);

    // Check for errors immediately after kernel launch
    CHECK(hipGetLastError());

    // Wait for the kernel to finish
    CHECK(hipDeviceSynchronize());

    // Free device memory
    CHECK(hipFree(d_data));

    printf("Program completed successfully.\n");
    return 0;
}
```