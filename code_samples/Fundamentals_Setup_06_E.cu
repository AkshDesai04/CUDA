```c
/*
Print the maximum y-dimension of a grid of blocks.

The aim of the program is to query the CUDA runtime for the maximum
grid dimensions supported by the current device and then print the
value for the Y‑dimension.  The information is obtained via
hipGetDeviceProperties which fills a hipDeviceProp_t structure.  The
member maxGridSize[1] holds the maximum number of blocks that can be
specified along the Y axis of a grid.  The program handles errors
gracefully by checking the return codes from CUDA API calls and
reporting any failures.  This is a minimal, self‑contained example
suitable for compiling with nvcc.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int device = 0;
    hipDeviceProp_t prop;

    // Get device count
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable devices found.\n");
        return 1;
    }

    // Get properties for device 0 (or chosen device)
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Print the maximum Y dimension of a grid
    printf("Maximum Y-dimension of a grid of blocks: %d\n", prop.maxGridSize[1]);

    return 0;
}
```