```cpp
/* 
Aim of the program: Check if the device supports cooperative launch (`cooperativeLaunch`) and print the result.

Thinking:
1. We need to query each CUDA-capable device present on the system.
2. CUDA exposes a device attribute `hipDeviceAttributeCooperativeLaunch` (via `hipDeviceGetAttribute`) which indicates whether cooperative launch is supported.
3. For each device:
   - Retrieve the attribute value (0 or 1).
   - Print a human-readable message.
4. Handle errors gracefully: check the return codes of CUDA runtime API calls and report any failures.
5. The program will be compiled as a .cu file and run on a machine with CUDA installed.
6. We'll use `printf` for simplicity and avoid additional dependencies.
7. The code uses standard CUDA runtime API and no external libraries.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("Error: hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return 0;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        int cooperativeLaunchSupported = 0;
        err = hipDeviceGetAttribute(&cooperativeLaunchSupported,
                                     hipDeviceAttributeCooperativeLaunch,
                                     dev);
        if (err != hipSuccess) {
            printf("Error: hipDeviceGetAttribute failed for device %d: %s\n",
                   dev, hipGetErrorString(err));
            continue;
        }

        printf("Device %d (%s): Cooperative Launch Supported: %s\n",
               dev,
               /* retrieve device name for clarity */
               [=]() {
                   hipDeviceProp_t prop;
                   hipGetDeviceProperties(&prop, dev);
                   return prop.name;
               }(),
               cooperativeLaunchSupported ? "Yes" : "No");
    }

    return 0;
}
```