```cuda
/*
Aim of the program:
Query and print the maximum pitch in bytes for a memory allocation.

Thinking:
To obtain the maximum allowable pitch for a 2D memory allocation on a CUDA device, the CUDA Runtime API provides the device attribute
hipDeviceAttributeMaxPitch. The function hipDeviceGetAttribute can be used to query this value for each device. 
We first determine how many CUDA-capable devices are available using hipGetDeviceCount. 
For each device, we call hipDeviceGetAttribute with the attribute hipDeviceAttributeMaxPitch and print the result.
Error checking is performed after each CUDA API call to ensure robust operation. 
The output is printed in bytes and uses unsigned long long formatting to accommodate 64-bit values.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
        return 0;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        size_t maxPitch = 0;
        err = hipDeviceGetAttribute((int*)&maxPitch, hipDeviceAttributeMaxPitch, dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipDeviceGetAttribute for device %d failed: %s\n", dev, hipGetErrorString(err));
            continue;
        }
        printf("Device %d: Max pitch for 2D allocation = %llu bytes\n", dev, (unsigned long long)maxPitch);
    }

    return 0;
}
```