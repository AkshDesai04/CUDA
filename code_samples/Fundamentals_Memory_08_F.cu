/*
Aim of the program: Wrap the event timer logic in a C++ class `GpuTimer` for convenience.

Thinking:
- The goal is to encapsulate CUDA event timing into a reusable C++ class. 
- The class should create two CUDA events (start and stop) in the constructor.
- Provide methods: startTimer(), stopTimer(), elapsedTime() returning milliseconds, and optionally reset().
- Destructor should destroy the events.
- Include error checking macro to catch CUDA API errors.
- Demonstrate usage in a main() with a dummy kernel launch to showcase timing.
- Use CUDA runtime API; include <cuda_runtime.h> and <iostream>.
- Ensure code compiles as a .cu file with nvcc.
- Keep the code self-contained, no external dependencies.
*/

#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error in file '" << __FILE__                   \
                      << "' in line " << __LINE__ << ": "                     \
                      << hipGetErrorString(err) << std::endl;                \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

class GpuTimer {
public:
    GpuTimer() {
        CUDA_CHECK(hipEventCreate(&start_));
        CUDA_CHECK(hipEventCreate(&stop_));
    }

    ~GpuTimer() {
        CUDA_CHECK(hipEventDestroy(start_));
        CUDA_CHECK(hipEventDestroy(stop_));
    }

    void startTimer() {
        CUDA_CHECK(hipEventRecord(start_, 0));
    }

    void stopTimer() {
        CUDA_CHECK(hipEventRecord(stop_, 0));
        CUDA_CHECK(hipEventSynchronize(stop_));
    }

    float elapsedTime() const {
        float ms = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&ms, start_, stop_));
        return ms;
    }

private:
    hipEvent_t start_;
    hipEvent_t stop_;
};

// Dummy kernel for demonstration
__global__ void dummyKernel(int *a) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] = idx;
}

int main() {
    const int N = 1 << 20; // 1M elements
    int *devPtr = nullptr;
    size_t size = N * sizeof(int);

    CUDA_CHECK(hipMalloc(&devPtr, size));

    GpuTimer timer;
    timer.startTimer();

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    dummyKernel<<<blocksPerGrid, threadsPerBlock>>>(devPtr);

    timer.stopTimer();

    float elapsed = timer.elapsedTime();
    std::cout << "Kernel execution time: " << elapsed << " ms" << std::endl;

    CUDA_CHECK(hipFree(devPtr));
    return 0;
}
