#include "hip/hip_runtime.h"
```cu
/*
Aim: Query and print the memory clock rate of the primary GPU.

Thinking process:

1. The CUDA Runtime API provides a function hipGetDeviceProperties that fills a hipDeviceProp_t structure with device information.
2. The memory clock rate is stored in the memoryClockRate field of this structure, expressed in kHz.
3. To get the "primary GPU", we simply query device 0 (the first GPU) which is typically the primary one.
4. We need to handle possible errors from the CUDA API calls. A simple error checking macro will be defined for readability.
5. The output should be printed in a human-readable format, converting kHz to MHz for convenience.
6. The program will be a minimal self-contained CUDA C file that can be compiled with nvcc and executed on a machine with a CUDA-capable GPU.

Implementation steps:
- Include <hip/hip_runtime.h> and <stdio.h>.
- Define a macro CHECK_CUDA(call) that checks the return value of a CUDA API call and exits on error.
- In main():
    * Call CHECK_CUDA(hipGetDeviceCount(&deviceCount)) to ensure at least one GPU exists.
    * Use device 0; call CHECK_CUDA(hipSetDevice(0)).
    * Declare a hipDeviceProp_t struct and call CHECK_CUDA(hipGetDeviceProperties(&prop, 0)).
    * Print the memory clock rate: prop.memoryClockRate (kHz). Convert to MHz by dividing by 1000.0.
- Return 0.

This program is straightforward and demonstrates basic CUDA device introspection.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call)                                 \
    do {                                                 \
        hipError_t err = (call);                        \
        if (err != hipSuccess) {                        \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                          \
        }                                                \
    } while (0)

int main(void) {
    int deviceCount = 0;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable devices found.\n");
        return EXIT_FAILURE;
    }

    int deviceId = 0; // Primary GPU
    CHECK_CUDA(hipSetDevice(deviceId));

    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, deviceId));

    // memoryClockRate is in kHz; convert to MHz
    double memoryClockMHz = prop.memoryClockRate / 1000.0;
    printf("Device %d (%s) memory clock rate: %.2f MHz\n",
           deviceId, prop.name, memoryClockMHz);

    return EXIT_SUCCESS;
}
```