```c
/*
Aim of the program: Query and print the maximum number of registers available per block.

Thought process:
- We want to use CUDA Runtime API to retrieve device properties.
- The attribute for maximum registers per block is hipDeviceAttributeMaxRegistersPerBlock.
- We can get the current device using hipGetDevice, then use hipDeviceGetAttribute to fetch the attribute.
- For robustness, we will also handle errors and print a friendly message if the query fails.
- The program will be a minimal console application that prints the result and exits.
- The code will include necessary headers, perform the query, print the result, and return an appropriate status.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int device = 0;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get CUDA device: %s\n", hipGetErrorString(err));
        return 1;
    }

    int maxRegsPerBlock = 0;
    err = hipDeviceGetAttribute(&maxRegsPerBlock,
                                 hipDeviceAttributeMaxRegistersPerBlock,
                                 device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to query MaxRegistersPerBlock: %s\n",
                hipGetErrorString(err));
        return 1;
    }

    printf("Maximum registers available per block on device %d: %d\n",
           device, maxRegsPerBlock);

    return 0;
}
```