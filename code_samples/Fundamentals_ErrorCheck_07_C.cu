#include "hip/hip_runtime.h"
```cpp
/*
How are synchronous errors reported? (Directly via the return code).

Thinking: 
In CUDA, certain API calls and kernel launches can fail synchronously, meaning
the failure is detected and reported immediately, allowing the program to
react without waiting for the GPU to finish. These synchronous errors are
reported directly via a return code of type `hipError_t`. For example, a
kernel launch with an illegal grid or block configuration fails immediately
and can be detected by checking the return value of `hipGetLastError()`.  
Other CUDA runtime API functions such as `hipMalloc`, `hipMemcpy`, or
`hipDeviceSynchronize` also return a `hipError_t` that indicates success
or failure. This program demonstrates synchronous error reporting in two
ways:
1.  A deliberately malformed kernel launch that fails immediately.
2.  An API call that fails because it requests more memory than is
    available, returning an error code directly.

The code prints the error code and a human‑readable message whenever a
synchronous error occurs, illustrating how to handle such errors in
production code. The program is self‑contained and can be compiled with
`nvcc -o sync_error_demo sync_error_demo.cu`.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

/* Helper macro to check CUDA error codes and print messages */
#define CHECK_CUDA(call)                                                         \
    do {                                                                         \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",          \
                    __FILE__, __LINE__, hipGetErrorString(err));                \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

/* Simple kernel that does nothing */
__global__ void do_nothing_kernel() {
    // No operation
}

int main(void) {
    /* -------------------------------------------------------------
     * 1. Synchronous error from an illegal kernel launch
     * ------------------------------------------------------------- */
    printf("Launching kernel with illegal configuration (grid.x = 0)...\n");
    /* Launching with grid.x = 0 is illegal and should fail synchronously. */
    do_nothing_kernel<<<0, 1>>>();
    /* Immediately check for launch errors */
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Synchronous kernel launch error detected: %s\n",
               hipGetErrorString(err));
    } else {
        printf("Kernel launch succeeded (unexpected).\n");
    }

    /* -------------------------------------------------------------
     * 2. Synchronous error from a CUDA runtime API call
     * ------------------------------------------------------------- */
    printf("\nAttempting to allocate an excessively large amount of memory...\n");
    /* Try to allocate 2^62 bytes (~4 EiB) which will definitely fail. */
    void* d_ptr = nullptr;
    size_t hugeSize = (size_t)1 << 62;
    err = hipMalloc(&d_ptr, hugeSize);
    if (err != hipSuccess) {
        printf("Synchronous hipMalloc error detected: %s\n",
               hipGetErrorString(err));
    } else {
        printf("hipMalloc succeeded (unexpected).\n");
        /* Free memory if allocation somehow succeeded */
        CHECK_CUDA(hipFree(d_ptr));
    }

    /* Clean exit */
    printf("\nProgram completed.\n");
    return 0;
}
```