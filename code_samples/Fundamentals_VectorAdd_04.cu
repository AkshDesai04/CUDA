/*
Implement element-wise vector subtraction (`C[i] = A[i] - B[i]`).

Thinking:
- A CUDA kernel will perform the subtraction for each element.
- Each thread processes one element: index = blockIdx.x * blockDim.x + threadIdx.x.
- The kernel will check bounds to avoid out-of-range memory access.
- Host code will:
  1. Allocate host and device memory for vectors A, B, C.
  2. Initialize vectors A and B with sample data.
  3. Copy data from host to device.
  4. Launch the subtraction kernel with an appropriate grid/block configuration.
  5. Copy the result vector C back to host.
  6. Verify the result by printing a few elements.
  7. Free allocated memory.
- Error checking will be performed after CUDA API calls.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Error checking macro
#define CUDA_CHECK(call)                                            \
    do {                                                            \
        hipError_t err = call;                                     \
        if (err != hipSuccess) {                                   \
            fprintf(stderr, "CUDA error at %s:%d - %s\n",            \
                    __FILE__, __LINE__, hipGetErrorString(err));   \
            exit(EXIT_FAILURE);                                     \
        }                                                           \
    } while (0)

// Kernel for element-wise vector subtraction
__global__ void vectorSubtraction(const float* A, const float* B, float* C, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] - B[idx];
    }
}

int main(void) {
    const size_t N = 1 << 20; // 1M elements
    const size_t bytes = N * sizeof(float);

    // Host allocation
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    if (!h_A || !h_B || !h_C) {
        fprintf(stderr, "Failed to allocate host vectors.\n");
        return EXIT_FAILURE;
    }

    // Initialize host vectors
    for (size_t i = 0; i < N; ++i) {
        h_A[i] = (float)i;
        h_B[i] = (float)(N - i);
    }

    // Device allocation
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_C, bytes));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    // Kernel launch configuration
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vectorSubtraction<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    // Verify a few results
    printf("Sample results:\n");
    for (int i = 0; i < 10; ++i) {
        printf("C[%d] = %f\n", i, h_C[i]);
    }

    // Clean up
    free(h_A);
    free(h_B);
    free(h_C);
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    return 0;
}
