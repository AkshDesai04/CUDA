```cpp
/*
Query and print the maximum number of threads allowed in a single thread block for the primary CUDA device. This value, `maxThreadsPerBlock`, is a key constraint when designing kernels.

--- Thinking Process ---
To solve this problem we need to query the CUDA driver for the properties of the primary device (device 0). The CUDA Runtime API provides the function `hipGetDeviceProperties` which fills a `hipDeviceProp_t` structure with various device characteristics, including `maxThreadsPerBlock`. The steps are:

1. Include the necessary headers: `<iostream>` for printing and `<hip/hip_runtime.h>` for CUDA runtime API.
2. Declare a `hipDeviceProp_t` variable.
3. Call `hipGetDeviceProperties(&prop, 0)` to fill it for the primary device. We must check the returned `hipError_t` to ensure the call succeeded.
4. Extract `prop.maxThreadsPerBlock` and print it to standard output.
5. Return `0` from `main` to indicate successful execution.

No kernel launch is required for this simple query. The program is straightforward and portable across CUDA-capable devices.
*/

#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, 0); // primary device = 0

    if (err != hipSuccess) {
        std::cerr << "Error querying device properties: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    std::cout << "Maximum number of threads per block (maxThreadsPerBlock): "
              << prop.maxThreadsPerBlock << std::endl;

    return 0;
}
```