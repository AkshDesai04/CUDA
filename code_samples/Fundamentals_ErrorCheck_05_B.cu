```cpp
/* 
Aim of the program: 
Look up the documentation for `hipErrorInvalidDevice`. What other functions can return this error?

Thinking:
The goal is to demonstrate that the CUDA error code `hipErrorInvalidDevice` can be produced by several API calls when an invalid device index is supplied. Typical functions that accept a device ID include:
- hipSetDevice
- hipGetDeviceProperties
- hipDeviceReset
- hipDeviceGetAttribute
- hipGetDevice (though this retrieves the current device, passing an invalid pointer might indirectly cause issues)
- hipDeviceGetPCIBusId (when combined with an invalid device)

To illustrate this, the program will:
1. Attempt to set an invalid device index using `hipSetDevice`.
2. Query properties for an invalid device using `hipGetDeviceProperties`.
3. Reset an invalid device using `hipDeviceReset`.
4. Retrieve an attribute for an invalid device using `hipDeviceGetAttribute`.
Each call will print the returned error code and its human-readable string. This demonstrates that all these functions can return `hipErrorInvalidDevice` when given an out‑of‑range device ID. */

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int invalidDevice = 999;  // Intentionally out of range

    // 1. hipSetDevice with invalid device
    hipError_t err = hipSetDevice(invalidDevice);
    printf("hipSetDevice(%d) returned: %s (%d)\n",
           invalidDevice, hipGetErrorString(err), err);

    // 2. hipGetDeviceProperties with invalid device
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, invalidDevice);
    printf("hipGetDeviceProperties(&prop, %d) returned: %s (%d)\n",
           invalidDevice, hipGetErrorString(err), err);

    // 3. hipDeviceReset with invalid device
    // First set a valid device so that reset is meaningful
    int devCount = 0;
    hipGetDeviceCount(&devCount);
    if (devCount > 0) {
        hipSetDevice(0);
    }
    err = hipDeviceReset();
    // Note: hipDeviceReset operates on the current device, so if we had an invalid device set
    // earlier, this would return hipErrorInvalidDevice. Since we reset to a valid device,
    // this should succeed. To force an error, we deliberately set an invalid device again.
    hipSetDevice(invalidDevice);
    err = hipDeviceReset();
    printf("hipDeviceReset() after setting invalid device returned: %s (%d)\n",
           hipGetErrorString(err), err);

    // 4. hipDeviceGetAttribute with invalid device
    int attribute = 0;
    err = hipDeviceGetAttribute(&attribute,
                                 hipDeviceAttributeMaxThreadsPerBlock,
                                 invalidDevice);
    printf("hipDeviceGetAttribute(..., %d) returned: %s (%d)\n",
           invalidDevice, hipGetErrorString(err), err);

    return 0;
}
```