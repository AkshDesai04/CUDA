#include "hip/hip_runtime.h"
```cpp
/*
Does `hipDeviceSynchronize` have a performance cost? (Yes, it stalls the CPU).
Explain when it's appropriate to use (debugging, final result needed) and when not (in a tight loop of asynchronous operations).

Thinking:
- The goal of this example is to illustrate that `hipDeviceSynchronize()` blocks the CPU until all previously issued GPU work is finished.
- We implement a simple vector addition kernel. The host code will launch this kernel twice:
  1. With `hipDeviceSynchronize()` immediately after launch (synchronous usage).
  2. Without a synchronization point in a tight loop, demonstrating asynchronous execution and how the CPU can do other work in parallel.
- Timing is performed with `std::chrono` on the host to show the difference in CPU wall‑clock time.
- We also use CUDA events to measure GPU execution time, which remains the same for both cases; the difference lies in the CPU time due to the block.
- The code demonstrates:
  * When to use `hipDeviceSynchronize()` (e.g., after kernel launches when you need the results before proceeding, or during debugging to catch errors).
  * When to avoid it (inside tight loops or pipelines where you want to overlap GPU work with CPU work).
- The program prints out timings so you can see the performance cost.
*/

#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

__global__
void vectorAdd(const float* A, const float* B, float* C, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) C[idx] = A[idx] + B[idx];
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error (" << msg << "): " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    const size_t N = 1 << 20;          // 1M elements
    const size_t bytes = N * sizeof(float);

    // Allocate host memory
    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    std::vector<float> h_C(N, 0.0f);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc(&d_A, bytes), "hipMalloc d_A");
    checkCudaError(hipMalloc(&d_B, bytes), "hipMalloc d_B");
    checkCudaError(hipMalloc(&d_C, bytes), "hipMalloc d_C");

    // Copy data to device (synchronous copy)
    checkCudaError(hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice), "Memcpy H2D A");
    checkCudaError(hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice), "Memcpy H2D B");

    // Kernel launch parameters
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // ----- Case 1: Synchronous usage (with hipDeviceSynchronize) -----
    {
        std::cout << "=== Synchronous execution with hipDeviceSynchronize ===" << std::endl;

        auto start = std::chrono::high_resolution_clock::now();

        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        // Check for launch errors
        checkCudaError(hipGetLastError(), "Kernel launch");

        // Synchronize: blocks the CPU until the kernel finishes
        checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> cpu_time = end - start;

        // Copy result back to host
        checkCudaError(hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost), "Memcpy D2H");

        std::cout << "CPU wall‑clock time (including sync): " << cpu_time.count() << " s" << std::endl;
        std::cout << "First element of result: " << h_C[0] << std::endl;
    }

    // ----- Case 2: Asynchronous usage (no sync in a tight loop) -----
    {
        std::cout << "\n=== Asynchronous execution (no hipDeviceSynchronize in loop) ===" << std::endl;

        const int num_iterations = 100;
        std::vector<float> h_D(num_iterations * N, 0.0f); // buffer for results

        auto start = std::chrono::high_resolution_clock::now();

        // Launch multiple kernels asynchronously
        for (int i = 0; i < num_iterations; ++i) {
            vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
            // We intentionally do NOT call hipDeviceSynchronize() here.
            // The kernel launches are queued; the CPU continues immediately.
        }

        // After launching all kernels, we do a single synchronization
        checkCudaError(hipDeviceSynchronize(), "Final hipDeviceSynchronize");

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> cpu_time = end - start;

        // Copy result back to host (only one result needed to verify correctness)
        checkCudaError(hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost), "Memcpy D2H");

        std::cout << "CPU wall‑clock time (no sync per launch): " << cpu_time.count() << " s" << std::endl;
        std::cout << "First element of result: " << h_C[0] << std::endl;
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
```