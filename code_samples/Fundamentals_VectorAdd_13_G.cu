/*
Implement `C[i] = (A[i] == B[i]) ? 1 : 0`.

Thinking:
- The aim is to create a CUDA program that performs an element‑wise comparison between two integer arrays A and B.
- For each index i, if A[i] equals B[i], set C[i] to 1; otherwise set C[i] to 0.
- The program will allocate memory on both host and device, copy data to the device, launch a kernel that does the comparison, and copy the result back.
- We'll use a simple kernel where each thread handles one element.
- The program includes basic error handling and prints a few results for verification.
- Size of the arrays can be changed; for this example, we use 1024 elements.
- Thread block size of 256 is chosen for good occupancy.
- The code is fully self‑contained and can be compiled with nvcc: nvcc -o compare compare.cu
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA kernel for element-wise comparison
__global__ void compare_arrays(const int *A, const int *B, int *C, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        C[idx] = (A[idx] == B[idx]) ? 1 : 0;
    }
}

// Utility macro for checking CUDA errors
#define CHECK_CUDA(call)                                         \
    do {                                                         \
        hipError_t err = call;                                  \
        if (err != hipSuccess) {                                \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",         \
                    __FILE__, __LINE__, hipGetErrorString(err));\
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    } while (0)

int main(void)
{
    const int n = 1024;          // Number of elements
    size_t bytes = n * sizeof(int);

    // Allocate host memory
    int *h_A = (int *)malloc(bytes);
    int *h_B = (int *)malloc(bytes);
    int *h_C = (int *)malloc(bytes);

    if (!h_A || !h_B || !h_C)
    {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    // Initialize host arrays with random data
    for (int i = 0; i < n; ++i)
    {
        h_A[i] = rand() % 10;
        h_B[i] = rand() % 10;
    }

    // Allocate device memory
    int *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void **)&d_A, bytes));
    CHECK_CUDA(hipMalloc((void **)&d_B, bytes));
    CHECK_CUDA(hipMalloc((void **)&d_C, bytes));

    // Copy data from host to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    // Launch kernel
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    compare_arrays<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
    CHECK_CUDA(hipGetLastError());   // Check for launch errors
    CHECK_CUDA(hipDeviceSynchronize()); // Wait for kernel to finish

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    // Print first 10 results
    printf("Index\tA[i]\tB[i]\tC[i]\n");
    for (int i = 0; i < 10 && i < n; ++i)
    {
        printf("%d\t%d\t%d\t%d\n", i, h_A[i], h_B[i], h_C[i]);
    }

    // Clean up
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);

    return EXIT_SUCCESS;
}
