```cuda
/* 
Aim: Try to call `hipFree` on a host pointer from `malloc`. What error does this produce?

Thinking:
1. Allocate memory on the host using `malloc`. This memory is ordinary CPU memory, not device memory.
2. Attempt to free that host pointer with `hipFree`, which is designed to free device memory allocated by `hipMalloc` or similar functions.
3. According to the CUDA Runtime API documentation, passing a non-device pointer to `hipFree` should return an error code indicating an invalid device pointer. In the current CUDA runtime, this is `hipErrorInvalidDevicePointer` (error code 3).
4. The program will capture the error returned by `hipFree`, print the error string using `hipGetErrorString`, and exit.
5. This demonstrates that `hipFree` cannot be used to free host memory, and the runtime reports the appropriate error.

The code below implements this test and prints the error information. */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main(void) {
    /* Allocate host memory */
    float *host_ptr = (float *)malloc(100 * sizeof(float));
    if (!host_ptr) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    /* Attempt to free host memory with hipFree */
    hipError_t err = hipFree(host_ptr);

    /* Print the error */
    printf("Attempted hipFree on host pointer returned error code: %d\n", err);
    printf("Error string: %s\n", hipGetErrorString(err));

    /* Clean up host memory */
    free(host_ptr);

    return (err == hipSuccess) ? EXIT_SUCCESS : EXIT_FAILURE;
}
```