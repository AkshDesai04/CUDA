```cpp
/*
Aim of the program: Query and print the `regsPerBlock` property.

Thought process:
- The `regsPerBlock` property is part of `hipDeviceProp_t`, which can be obtained via
  `hipGetDeviceProperties()`. It indicates the number of registers per block that
  the device can allocate.
- We will write a CUDA C++ program that queries the number of devices using
  `hipGetDeviceCount()`. For each device, we call `hipGetDeviceProperties()` to
  retrieve the device properties struct.
- Then we print the `regsPerBlock` value for each device to the console.
- Error checking will be performed after each CUDA API call. If an error occurs,
  we will print an error message and exit the program.
- The program uses standard C++ I/O (`iostream`) to display results.

The code below is self-contained and should compile with `nvcc`. It requires a CUDA
enabled compiler and a compatible device.

*/

#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "Error: hipGetDeviceCount failed: " << hipGetErrorString(err) << std::endl;
        return EXIT_FAILURE;
    }

    if (deviceCount == 0) {
        std::cerr << "No CUDA-capable devices found." << std::endl;
        return EXIT_FAILURE;
    }

    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, dev);
        if (err != hipSuccess) {
            std::cerr << "Error: hipGetDeviceProperties failed for device " << dev
                      << ": " << hipGetErrorString(err) << std::endl;
            continue;
        }

        std::cout << "Device " << dev << " (" << prop.name << "):" << std::endl;
        std::cout << "  regsPerBlock = " << prop.regsPerBlock << std::endl;
    }

    return EXIT_SUCCESS;
}
```