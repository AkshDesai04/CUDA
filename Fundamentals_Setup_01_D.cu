/*
Query and print the number of multiprocessors on the primary CUDA device (device 0).

Thinking:
To obtain the number of multiprocessors on the first CUDA device, we need to:
1. Include the CUDA runtime header (`cuda_runtime.h`) and the standard iostream.
2. Set device 0 as the current device using `cudaSetDevice(0)` (though this is optional for querying properties, it ensures we are working with the primary device).
3. Retrieve the device properties structure via `cudaGetDeviceProperties`.
4. The field `multiProcessorCount` in the returned structure holds the number of SMs.
5. Print this value to standard output.
6. Wrap CUDA API calls in a helper macro (`CUDA_CHECK`) that aborts on error for simplicity.
7. Return `0` from `main`.
*/

#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = (call);                                             \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error in " << __FILE__ << " at line "         \
                      << __LINE__ << ": " << hipGetErrorString(err)         \
                      << std::endl;                                          \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

int main() {
    // Set device 0 as current device (optional but explicit)
    CUDA_CHECK(hipSetDevice(0));

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    std::cout << "Number of multiprocessors on device 0: "
              << prop.multiProcessorCount << std::endl;

    return 0;
}
