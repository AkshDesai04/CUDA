/*
Aim: Use `atomicAdd` to safely have multiple threads increment a counter member within the single shared struct instance.
Thinking: 
- We create a struct SharedCounter with a single integer counter.
- Each block has one instance of this struct in shared memory.
- Thread 0 initializes the counter to zero.
- All threads in the block call atomicAdd on the address of s.counter to increment it.
- After synchronization, thread 0 writes the final counter value to a global array.
- The host launches multiple blocks, each with multiple threads, and then prints the counts from each block.
- This demonstrates atomicAdd usage on a shared memory struct member.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

typedef struct {
    int counter;
} SharedCounter;

__global__ void counterKernel(int threadsPerBlock, int *blockCounts)
{
    __shared__ SharedCounter s;

    // Initialize counter once per block
    if (threadIdx.x == 0) {
        s.counter = 0;
    }
    __syncthreads();

    // Each thread increments the counter atomically
    atomicAdd(&(s.counter), 1);

    __syncthreads();

    // Store the result for this block
    if (threadIdx.x == 0) {
        blockCounts[blockIdx.x] = s.counter;
    }
}

int main(void)
{
    const int numBlocks = 10;
    const int threadsPerBlock = 32;

    int *d_blockCounts = NULL;
    int *h_blockCounts = NULL;

    size_t countsSize = numBlocks * sizeof(int);

    // Allocate device memory for block counts
    hipError_t err = hipMalloc((void**)&d_blockCounts, countsSize);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Allocate host memory for results
    h_blockCounts = (int*)malloc(countsSize);
    if (!h_blockCounts) {
        fprintf(stderr, "Host malloc failed\n");
        hipFree(d_blockCounts);
        return 1;
    }

    // Launch kernel
    counterKernel<<<numBlocks, threadsPerBlock>>>(threadsPerBlock, d_blockCounts);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        free(h_blockCounts);
        hipFree(d_blockCounts);
        return 1;
    }

    // Copy results back to host
    err = hipMemcpy(h_blockCounts, d_blockCounts, countsSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
        free(h_blockCounts);
        hipFree(d_blockCounts);
        return 1;
    }

    // Print results
    printf("Block counts:\n");
    for (int i = 0; i < numBlocks; ++i) {
        printf("  Block %2d: %d\n", i, h_blockCounts[i]);
    }

    // Clean up
    free(h_blockCounts);
    hipFree(d_blockCounts);

    return 0;
}