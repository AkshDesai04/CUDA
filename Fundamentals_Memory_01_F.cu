```cuda
/* 
Aim: Try to `hipMalloc` zero bytes. Does it return an error or a valid pointer?

This program tests the behavior of the CUDA runtime when allocating
zero bytes with `hipMalloc`. According to the CUDA Runtime API,
`hipMalloc` should return `hipSuccess` even when the requested size
is zero, but the semantics of the returned pointer are not explicitly
defined. Some implementations return a non‑null pointer that cannot
be dereferenced, while others may return a null pointer. The purpose
of this test is to observe the returned error code, the value of the
pointer, and whether `hipFree` accepts it without causing a crash.

The program performs the following steps:
1. Calls `hipMalloc` with a size of 0 and stores the result in a
   `void*` pointer.
2. Prints the returned error code and the string representation
   of the error.
3. Prints the pointer value and indicates whether it is NULL.
4. Frees the pointer (if non‑NULL) to avoid memory leaks.
5. Exits with status 0.

Compile with: 
    nvcc -o cuda_zero_malloc cuda_zero_malloc.cu

Run with:
    ./cuda_zero_malloc
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    void *devPtr = NULL;
    size_t size = 0;

    // Attempt to allocate zero bytes on the device
    hipError_t err = hipMalloc(&devPtr, size);

    // Print the error code and message
    printf("hipMalloc(%zu) returned error code: %d (%s)\n",
           size, err, hipGetErrorString(err));

    // Print the pointer value
    printf("Returned pointer: %p\n", devPtr);
    if (devPtr == NULL) {
        printf("Pointer is NULL.\n");
    } else {
        printf("Pointer is NOT NULL.\n");
    }

    // Free the allocated memory if the pointer is non-NULL
    if (devPtr != NULL) {
        hipError_t freeErr = hipFree(devPtr);
        printf("hipFree returned error code: %d (%s)\n",
               freeErr, hipGetErrorString(freeErr));
    } else {
        // Even if NULL, hipFree is safe to call
        hipError_t freeErr = hipFree(devPtr);
        printf("hipFree(NULL) returned error code: %d (%s)\n",
               freeErr, hipGetErrorString(freeErr));
    }

    return 0;
}
```