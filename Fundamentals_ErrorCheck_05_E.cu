```c
/*
Aim of the program: Try to get device properties for an invalid device index with hipGetDeviceProperties.
 
Thinking process:
1. I want to demonstrate what happens when hipGetDeviceProperties is called with a device index that is not valid.
2. CUDA device indices range from 0 to (deviceCount - 1). Any index outside this range is invalid.
3. I'll first query the number of CUDA-capable devices with hipGetDeviceCount. This is optional, but useful to determine an out-of-range index.
4. I'll construct an invalid index by adding 1 to the device count (or using -1). Both are invalid, but adding 1 ensures it is out of bounds even if count is 0.
5. I'll call hipGetDeviceProperties with this invalid index. The function returns a hipError_t value indicating the error.
6. I'll print the returned error code and its human-readable string using hipGetErrorString to show the failure.
7. The program will compile as a standard .cu file, include <hip/hip_runtime.h> and <stdio.h>, and contain a simple main() that performs these steps.
8. No additional functionality is needed; the goal is solely to trigger and display the error for an invalid device index.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device count: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Choose an invalid device index: deviceCount is out of range (valid indices are 0..deviceCount-1)
    int invalidIndex = deviceCount; // If deviceCount is 0, this is also invalid

    // Alternatively, could use -1 to guarantee invalid index:
    // int invalidIndex = -1;

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, invalidIndex);
    if (err != hipSuccess) {
        printf("Attempted to get properties for invalid device index %d.\n", invalidIndex);
        printf("hipGetDeviceProperties returned error: %s (%d)\n", hipGetErrorString(err), err);
    } else {
        // This branch should not execute for an invalid index, but handle just in case
        printf("Device %d properties retrieved (unexpectedly):\n", invalidIndex);
        printf("  Name: %s\n", prop.name);
        printf("  Total Global Memory: %lu bytes\n", prop.totalGlobalMem);
    }

    return 0;
}
```