/*
Aim: Write a version of the function for streams: `checkStream(cudaStream_t stream)` which calls `cudaStreamSynchronize`.

Thinking: 
- The goal is to provide a small CUDA helper that synchronizes a given CUDA stream and reports any errors that occur during the synchronization. 
- The function will call `cudaStreamSynchronize(stream)`, capture the returned `cudaError_t`, and if it is not `cudaSuccess`, print an error message and terminate the program. 
- For consistency with typical CUDA error‑checking conventions, we also provide a generic `check()` function that handles any `cudaError_t` and prints a message with the file and line number. 
- A simple `main()` function demonstrates usage: it creates a CUDA stream, performs a trivial kernel launch (to have something running on the stream), and then calls `checkStream()` to wait for completion and verify success. 
- The code uses the CUDA runtime API (`#include <cuda_runtime.h>`) and is written in C with CUDA extensions, compiling as a .cu file. 
- The entire file is self‑contained: includes, helper functions, a trivial kernel, and a `main()` that exercises the `checkStream` wrapper.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/* Helper function to check CUDA error codes */
static void check(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error [%s:%d]: %s (%d)\n",
                file, line, hipGetErrorString(err), err);
        exit(EXIT_FAILURE);
    }
}

#define CHECK(call) check((call), __FILE__, __LINE__)

/* Wrapper that synchronizes a stream and checks for errors */
static void checkStream(hipStream_t stream)
{
    hipError_t err = hipStreamSynchronize(stream);
    if (err != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed: %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/* A trivial kernel that does nothing but will run on the stream */
__global__ void dummyKernel()
{
    /* Empty kernel body */
}

int main(void)
{
    /* Create a CUDA stream */
    hipStream_t stream;
    CHECK(hipStreamCreate(&stream));

    /* Launch a trivial kernel on the stream */
    dummyKernel<<<1, 1, 0, stream>>>();
    CHECK(hipGetLastError());  /* check kernel launch */

    /* Synchronize the stream using our wrapper */
    checkStream(stream);

    /* Destroy the stream */
    CHECK(hipStreamDestroy(stream));

    printf("Stream synchronized successfully.\n");
    return 0;
}
