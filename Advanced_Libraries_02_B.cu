/*
Aim: Generate random integers instead of floats.

Thinking:
- The original problem requested generating random integers rather than floats.
- We'll use the CUDA CURAND library's host API to generate random unsigned integers directly on the GPU.
- The function `curandGenerate` outputs 32‑bit unsigned integers, which we can treat as random integers.
- We'll allocate device memory, generate N random integers, copy them back to the host, and print them.
- We'll include error checking for CURAND and CUDA calls.
- The program is self‑contained, compiles with `nvcc`, and prints the random integers to stdout.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 10  // Number of random integers to generate

int main(void) {
    hiprandGenerator_t gen;
    hiprandStatus_t status;

    // Create a pseudo‑random generator
    status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    if (status != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "Error: hiprandCreateGenerator failed (status %d)\n", status);
        return EXIT_FAILURE;
    }

    // Seed the generator
    status = hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    if (status != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "Error: hiprandSetPseudoRandomGeneratorSeed failed (status %d)\n", status);
        hiprandDestroyGenerator(gen);
        return EXIT_FAILURE;
    }

    // Allocate device memory for N unsigned ints
    unsigned int *d_random;
    hipError_t cudaStatus = hipMalloc((void **)&d_random, N * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc failed (status %d)\n", cudaStatus);
        hiprandDestroyGenerator(gen);
        return EXIT_FAILURE;
    }

    // Generate N random unsigned integers on the device
    status = hiprandGenerate(gen, d_random, N);
    if (status != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "Error: hiprandGenerate failed (status %d)\n", status);
        hipFree(d_random);
        hiprandDestroyGenerator(gen);
        return EXIT_FAILURE;
    }

    // Copy the random integers back to host
    unsigned int *h_random = (unsigned int *)malloc(N * sizeof(unsigned int));
    if (!h_random) {
        fprintf(stderr, "Error: host allocation failed\n");
        hipFree(d_random);
        hiprandDestroyGenerator(gen);
        return EXIT_FAILURE;
    }

    cudaStatus = hipMemcpy(h_random, d_random, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy failed (status %d)\n", cudaStatus);
        free(h_random);
        hipFree(d_random);
        hiprandDestroyGenerator(gen);
        return EXIT_FAILURE;
    }

    // Print the random integers
    printf("Random integers:\n");
    for (int i = 0; i < N; ++i) {
        printf("%u ", h_random[i]);
    }
    printf("\n");

    // Clean up
    free(h_random);
    hipFree(d_random);
    hiprandDestroyGenerator(gen);

    return EXIT_SUCCESS;
}
