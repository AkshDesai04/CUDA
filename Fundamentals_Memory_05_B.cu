```cpp
/*
Aim of the program:
Try to call hipFree on the same pointer twice. What happens? Use error checking to see the result of the second call.

Thinking:
The objective is to demonstrate the behavior of CUDA's memory deallocation routine when invoked twice on the same device pointer. 
We will:
1. Allocate a small block of device memory using hipMalloc.
2. Free that memory with hipFree and check the return status.
3. Attempt to free the same pointer a second time and capture the return status.
4. Print the numerical error code and the human‑readable error string for both calls.
5. Observe that the first call succeeds (hipSuccess), while the second call typically returns an error
   such as hipErrorInvalidDevicePointer or hipErrorInvalidValue, indicating that the pointer is no longer
   valid for deallocation.

We will use the standard CUDA runtime API, and basic error checking via the returned hipError_t
and hipGetErrorString. No kernel launches or complex device code are required.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

int main() {
    const size_t numElements = 10;
    const size_t size = numElements * sizeof(int);
    int *d_ptr = nullptr;

    // Allocate device memory
    hipError_t err = hipMalloc((void**)&d_ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    printf("hipMalloc succeeded, pointer = %p\n", (void*)d_ptr);

    // First hipFree
    err = hipFree(d_ptr);
    if (err == hipSuccess) {
        printf("First hipFree succeeded.\n");
    } else {
        printf("First hipFree failed: %s\n", hipGetErrorString(err));
    }

    // Second hipFree on the same pointer
    err = hipFree(d_ptr);
    if (err == hipSuccess) {
        printf("Second hipFree succeeded (unexpected).\n");
    } else {
        printf("Second hipFree failed as expected: %s\n", hipGetErrorString(err));
    }

    return 0;
}
```