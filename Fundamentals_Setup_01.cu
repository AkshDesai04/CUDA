```cu
/*
Aim of the program: 
Write a CUDA program to query and print the number of CUDA-enabled devices available on your system. The program should use the `hipGetDeviceCount` API call and print a user-friendly message, such as 'Found X CUDA-capable devices.'

Thinking:
1. We need to include the CUDA runtime header `cuda_runtime.h` to use `hipGetDeviceCount`.
2. Use `int main()` as the entry point.
3. Declare an integer variable to hold the device count.
4. Call `hipGetDeviceCount(&count)` and capture the returned `hipError_t`.
5. Check if the call succeeded. If not, print an error message using `hipGetErrorString(err)` and exit with a non-zero status.
6. If successful, print the message "Found X CUDA-capable devices." using `std::cout`.
7. Return 0 to indicate success.
8. Since we are writing a .cu file, we use `#include <hip/hip_runtime.h>` and `#include <iostream>`.
9. No kernel launch or other CUDA features are needed; this is purely a host query program.
10. Ensure the code compiles with `nvcc`. 
*/

#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        std::cerr << "hipGetDeviceCount failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << "Found " << deviceCount << " CUDA-capable devices." << std::endl;
    return 0;
}
```