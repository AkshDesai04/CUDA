```cpp
/* 
Query and print the maximum dimensions (width and height) of a 2D texture for device 0.
Thinking:
- Use the CUDA Runtime API to set the device to 0.
- Retrieve device properties via hipGetDeviceProperties.
- The struct hipDeviceProp_t contains a member maxTexture2D of type cudaTextureAlignment, which has width and height fields.
- After fetching properties, print the width and height using printf.
- Add basic error checking for CUDA API calls.
- Provide a minimal main() that performs these steps and exits.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

int main() {
    int device = 0;
    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set device %d: %s\n", device, hipGetErrorString(err));
        return 1;
    }

    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device properties for device %d: %s\n", device, hipGetErrorString(err));
        return 1;
    }

    printf("Maximum 2D texture dimensions for device %d:\n", device);
    printf("  Width  = %u\n", prop.maxTexture2D.width);
    printf("  Height = %u\n", prop.maxTexture2D.height);

    return 0;
}
```