/*
Write a loop that iterates through all available devices and calls `cudaSetDevice` followed by `cudaDeviceReset` for each one.

Thought process:
- The program should query how many CUDA devices are present with `cudaGetDeviceCount`.
- It should iterate from 0 to deviceCount-1.
- For each device index, first call `cudaSetDevice(i)` to make that device the current one.
- Then call `cudaDeviceReset()` which resets the device and cleans up resources.
- Error checking is important: check return status of both CUDA API calls and report failures.
- Use standard C/C++ output to indicate progress.
- Wrap the logic in `int main()` returning 0 on success.
- Include necessary headers: <cuda_runtime.h> and <cstdio> or <iostream>.
- Keep the code minimal but robust, and compileable with `nvcc`.
- No additional text or explanation outside the code is required.
*/

#include <hip/hip_runtime.h>
#include <cstdio>

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    printf("Number of CUDA devices: %d\n", deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        printf("Setting device %d\n", dev);
        err = hipSetDevice(dev);
        if (err != hipSuccess) {
            fprintf(stderr, "cudaSetDevice(%d) failed: %s\n", dev, hipGetErrorString(err));
            // Continue to next device to attempt resetting others
            continue;
        }

        printf("Resetting device %d\n", dev);
        err = hipDeviceReset();
        if (err != hipSuccess) {
            fprintf(stderr, "cudaDeviceReset() on device %d failed: %s\n", dev, hipGetErrorString(err));
            // Continue to next device
            continue;
        }

        printf("Device %d reset successfully.\n", dev);
    }

    printf("All devices processed.\n");
    return EXIT_SUCCESS;
}
