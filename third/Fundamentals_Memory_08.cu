```c
/* 
Time the data transfer speed (bandwidth) of `hipMemcpy`. Transfer a large array (e.g., 256MB) from host to device. Record the time before and after the copy using a CUDA event timer (`hipEventCreate`, `hipEventRecord`, `hipEventSynchronize`, `hipEventElapsedTime`). Calculate and print the bandwidth in GB/s.

My thinking:
- Allocate a large array on the host (256 MB). Using a `char` array keeps the size calculation straightforward.
- Allocate a device buffer of the same size with `hipMalloc`.
- Create two CUDA events (`start` and `stop`) to measure the elapsed time of the copy operation.
- Record the start event, perform `hipMemcpy` from host to device, record the stop event, then synchronize the stop event to ensure the copy has finished.
- Use `hipEventElapsedTime` to obtain the elapsed time in milliseconds.
- Convert the elapsed time to seconds and calculate bandwidth as (size in bytes) / (elapsed time in seconds) / 1e9 to get GB/s.
- Print the bandwidth to standard output.
- Perform basic error checking after each CUDA API call for robustness.
- Clean up by freeing host and device memory and destroying events.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    const size_t size_bytes = 256UL * 1024 * 1024; // 256 MB
    char *h_ptr = NULL;
    char *d_ptr = NULL;
    hipError_t err;

    // Allocate host memory
    err = hipHostMalloc((void**)&h_ptr, size_bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Host allocation failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Allocate device memory
    err = hipMalloc((void**)&d_ptr, size_bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Device allocation failed: %s\n", hipGetErrorString(err));
        hipHostFree(h_ptr);
        return -1;
    }

    // Create CUDA events
    hipEvent_t start, stop;
    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        fprintf(stderr, "hipEventCreate start failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }
    err = hipEventCreate(&stop);
    if (err != hipSuccess) {
        fprintf(stderr, "hipEventCreate stop failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    // Record start event
    hipEventRecord(start, 0);

    // Perform the copy from host to device
    err = hipMemcpy(d_ptr, h_ptr, size_bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    // Record stop event
    hipEventRecord(stop, 0);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, start, stop);

    // Compute bandwidth (GB/s)
    float elapsed_sec = elapsed_ms / 1000.0f;
    float bandwidth_gbps = (float)size_bytes / elapsed_sec / 1e9f;

    printf("Data transfer of %.0f MB took %.3f ms, bandwidth = %.3f GB/s\n",
           (float)size_bytes / (1024 * 1024), elapsed_ms, bandwidth_gbps);

cleanup:
    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(d_ptr);

    // Free host memory
    hipHostFree(h_ptr);

    return 0;
}
```