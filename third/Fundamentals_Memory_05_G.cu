#include "hip/hip_runtime.h"
```cuda
/*
Aim: What happens if you allocate memory but your program crashes before `hipFree` is called? Does the memory get reclaimed?

Thinking:
- Device memory is allocated by the CUDA driver on behalf of the process.
- If the process terminates unexpectedly (crash, abort, or explicit exit), the driver
  detects the termination and automatically frees all resources that were allocated
  for that process, including device memory.
- Therefore, the memory is reclaimed when the process ends, even if `hipFree` was never
  called.
- If the process continues to run but never frees the memory, it remains allocated
  until the process exits or until `hipFree` is called.
- In practice, you should free memory explicitly, but the driver’s cleanup on process
  exit prevents leaks that would otherwise affect other processes.
- This program demonstrates allocating memory, printing device free memory before
  and after allocation, then intentionally aborting before calling `hipFree`.  While
  the reclamation occurs when the process exits, we cannot directly observe it from
  within the same process; however, the driver documentation guarantees the
  cleanup.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void dummyKernel(int *d_arr, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_arr[idx] = idx * 2;
    }
}

int main() {
    size_t freeMem, totalMem;
    hipError_t err;

    // Query free memory before allocation
    err = hipMemGetInfo(&freeMem, &totalMem);
    if (err != hipSuccess) {
        fprintf(stderr, "Error querying memory info before allocation: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    printf("Before allocation: Free memory = %zu bytes, Total memory = %zu bytes\n",
           freeMem, totalMem);

    const int N = 1 << 20; // 1M integers
    int *d_arr = NULL;
    err = hipMalloc((void**)&d_arr, N * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    // Query free memory after allocation
    err = hipMemGetInfo(&freeMem, &totalMem);
    if (err != hipSuccess) {
        fprintf(stderr, "Error querying memory info after allocation: %s\n", hipGetErrorString(err));
        hipFree(d_arr); // Clean up before exiting on error
        return EXIT_FAILURE;
    }
    printf("After allocation: Free memory = %zu bytes, Total memory = %zu bytes\n",
           freeMem, totalMem);

    // Launch a simple kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    dummyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_arr, N);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return EXIT_FAILURE;
    }
    hipDeviceSynchronize();

    // Intentionally abort before calling hipFree to simulate a crash
    printf("Simulating crash: aborting before hipFree\n");
    abort(); // This will terminate the process abruptly

    // The following code will not be executed
    // hipFree(d_arr);
    // hipDeviceReset();

    return EXIT_SUCCESS;
}
```