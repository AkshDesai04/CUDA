```cpp
/*
Aim of the program:
Write a program that queries and prints the total amount of global memory available on device 0 in megabytes (MB). 
You will need to get the `totalGlobalMem` property from `hipDeviceProp_t` and convert it from bytes to MB.

Thinking:
1. Include necessary headers: <cstdio> for printf and <hip/hip_runtime.h> for CUDA API.
2. In main, declare a hipDeviceProp_t struct and use hipGetDeviceProperties(&prop, 0) to fill it for device 0.
3. Check for errors from hipGetDeviceProperties. If an error occurs, print an error message and exit with non-zero status.
4. Convert prop.totalGlobalMem (bytes) to megabytes by dividing by (1024 * 1024). Use a double or float for fractional MB.
5. Print the result using printf, formatting with e.g., "%.2f MB" for two decimal places.
6. Return 0 on success.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    int device = 0;
    hipError_t err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Error getting device properties: %s\n", hipGetErrorString(err));
        return 1;
    }

    double totalMemMB = static_cast<double>(prop.totalGlobalMem) / (1024.0 * 1024.0);
    printf("Total global memory on device %d: %.2f MB\n", device, totalMemMB);

    return 0;
}
```