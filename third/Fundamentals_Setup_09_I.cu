/*
Print the amount of shared memory available per multiprocessor.

To solve this problem we need to query the device properties using CUDA’s
runtime API.  The structure cudaDeviceProp contains a field
sharedMemPerMultiprocessor that holds the amount of shared memory that
each multiprocessor on the GPU can provide.  We will:
1. Call cudaGetDeviceProperties for the first device (device 0).
2. Check for errors from the CUDA API.
3. Print the value in bytes, using the %zu format specifier for size_t.

The program is written in CUDA C and can be compiled with nvcc.  It
requires no kernel launches; all work is done on the host side.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    hipDeviceProp_t prop;
    int device = 0; // default to the first CUDA device

    hipError_t err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipGetDeviceProperties failed: %s\n",
                hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    printf("Shared memory per multiprocessor: %zu bytes\n",
           prop.sharedMemPerMultiprocessor);

    return EXIT_SUCCESS;
}
