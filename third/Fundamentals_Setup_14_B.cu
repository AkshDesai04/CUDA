```cpp
/*
Aim of the program:
Based on the asyncEngineCount, print a message like 'Device has X copy engines.'
If the count is greater than 0, also print 'Data copy can be overlapped with kernel execution.'

Thinking process:
1. Use the CUDA Runtime API to query device properties via hipGetDeviceProperties.
2. The asyncEngineCount field in hipDeviceProp_t tells us how many asynchronous copy engines a device has.
3. The program will:
   - Determine the number of CUDA devices available.
   - For each device, retrieve its properties.
   - Print "Device has X copy engines." where X is asyncEngineCount.
   - If asyncEngineCount > 0, also print "Data copy can be overlapped with kernel execution."
4. Error handling will be minimal for brevity: if hipGetDeviceProperties fails, print an error and exit.
5. The code is a self‑contained CUDA C++ source file that can be compiled with nvcc.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return 0;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Error: hipGetDeviceProperties for device %d failed: %s\n",
                    dev, hipGetErrorString(err));
            continue;
        }

        printf("Device %d (%s) has %d copy engines.\n",
               dev, prop.name, prop.asyncEngineCount);

        if (prop.asyncEngineCount > 0) {
            printf("Data copy can be overlapped with kernel execution.\n");
        }
    }

    return 0;
}
```