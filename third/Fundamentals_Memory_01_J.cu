```cuda
/*
Aim of the program:
Allocate on host, allocate on device, but forget the hipMemcpy call. What is in the device memory?

Thinking:
The CUDA runtime does not automatically copy data from host to device. After calling hipMalloc,
the device memory contains whatever happened to be in that region previously; it is not
guaranteed to be zeroed or contain the host data. In many systems the memory may be
zeroed for security reasons, but this is not a documented behavior and should not be
reliably depended upon. This program demonstrates that by allocating a small array on
the host, allocating an array on the device, *not* copying the host data to the device,
and then copying the device array back to the host to inspect its contents. The output
will show that the device memory holds arbitrary values (often zero, but not guaranteed),
highlighting that forgetting the hipMemcpy results in uninitialized device memory.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    const int N = 10;
    int h_arr[N];
    for (int i = 0; i < N; ++i) h_arr[i] = i + 1;  // Host array initialized to 1..10

    int *d_arr = nullptr;
    hipError_t err = hipMalloc((void**)&d_arr, N * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Intentionally omit hipMemcpy from host to device.

    int h_arr_copy[N];
    err = hipMemcpy(h_arr_copy, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return -1;
    }

    printf("Contents of device memory after hipMalloc (no memcpy):\n");
    for (int i = 0; i < N; ++i) {
        printf("%d ", h_arr_copy[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}
```