```cpp
/* 
Try to call `hipFree` on a host pointer from `malloc`. What error does this produce?

Thinking:
- The aim is to demonstrate the behavior of calling hipFree on a host pointer.
- Allocate memory on the host using standard C `malloc`.
- Pass that host pointer to `hipFree` which expects a device pointer.
- Capture the returned hipError_t value.
- Use `hipGetErrorString` to obtain a human-readable description.
- Print both the numeric error code and the description.
- According to CUDA documentation, passing an invalid (host) pointer to hipFree
  should yield `CUDA_ERROR_INVALID_DEVICE_POINTER` (error code 11).
- The program will compile with nvcc and run on a system with CUDA installed.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main(void) {
    // Allocate memory on the host
    size_t size = 256;
    void *hostPtr = malloc(size);
    if (hostPtr == NULL) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    // Attempt to free the host pointer using hipFree
    hipError_t err = hipFree(hostPtr);

    // Check the result
    if (err != hipSuccess) {
        printf("hipFree failed with error code %d: %s\n", err, hipGetErrorString(err));
    } else {
        printf("hipFree succeeded (unexpected).\n");
    }

    // Clean up the host memory
    free(hostPtr);

    return 0;
}
```