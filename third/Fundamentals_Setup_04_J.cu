```cpp
/*
Aim: Query and print whether the device can map host memory (`canMapHostMemory` property).

Thinking:
- The goal is to inspect CUDA device capabilities, specifically the flag `canMapHostMemory`
  which indicates whether the device can map host memory (e.g., for pinned memory access).
- To achieve this, the program will:
  1. Include the CUDA runtime header `<hip/hip_runtime.h>`.
  2. Query the number of CUDA-capable devices using `hipGetDeviceCount`.
  3. Iterate over each device (or just the first one) to retrieve its properties via
     `hipGetDeviceProperties`.
  4. Print the value of `props.canMapHostMemory` for each device.
- The program will include basic error checking for CUDA API calls.
- It will be a simple console application that outputs the information to standard output.
- The output will be a single line indicating whether each device can map host memory,
  e.g., "Device 0 can map host memory: 1" (where 1 means true).
- The code will be self‑contained and compile with `nvcc` producing a `.cu` file.

*/

#include <cstdio>
#include <hip/hip_runtime.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0)
    {
        printf("No CUDA-capable devices found.\n");
        return 0;
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, dev);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error: hipGetDeviceProperties for device %d failed: %s\n",
                    dev, hipGetErrorString(err));
            continue;
        }

        printf("Device %d: \"%s\"\n", dev, prop.name);
        printf("  canMapHostMemory: %s\n",
               prop.canMapHostMemory ? "Yes (1)" : "No (0)");
        printf("\n");
    }

    return 0;
}
```