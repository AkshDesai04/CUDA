```cpp
/*
Write a program that prints the CUDA driver version and the CUDA runtime version detected on your system. Use `hipDriverGetVersion` and `hipRuntimeGetVersion` and explain in a comment what the difference between the two is.

My thinking:
- The program is a simple C++ (or C) CUDA host code that calls the two CUDA Runtime API functions: hipDriverGetVersion and hipRuntimeGetVersion.
- We need to include <hip/hip_runtime.h> for runtime API and <hip/hip_runtime.h> for driver API. Both provide the functions.
- The program should print the integer versions (e.g., 11020 for CUDA 11.2). These integers encode major*1000 + minor*10 + patch.
- The comment explaining the difference: Driver API is lower level, deals with GPU device drivers; runtime API is higher level, abstracts driver for user code. Driver API can run on systems without a full CUDA installation; runtime needs the runtime library. Usually, the driver version is >= runtime version.
- We'll check the return status of each call and print an error if they fail.
- We'll use printf from <cstdio> or std::cout. For simplicity, use printf.
- We'll convert the integer version to human-readable form by printing the integer value.
- We'll compile with nvcc. The code is self-contained.
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main(void) {
    int driverVersion = 0;
    int runtimeVersion = 0;

    // Get CUDA driver version
    hipError_t driverErr = hipDriverGetVersion(&driverVersion);
    if (driverErr != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed: %s\n", hipGetErrorString(driverErr));
        return 1;
    }

    // Get CUDA runtime version
    hipError_t runtimeErr = hipRuntimeGetVersion(&runtimeVersion);
    if (runtimeErr != hipSuccess) {
        fprintf(stderr, "hipRuntimeGetVersion failed: %s\n", hipGetErrorString(runtimeErr));
        return 1;
    }

    printf("CUDA Driver Version: %d\n", driverVersion);
    printf("CUDA Runtime Version: %d\n", runtimeVersion);

    return 0;
}
```