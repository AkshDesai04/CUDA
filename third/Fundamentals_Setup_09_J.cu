```c
/*
Aim of the program: Print the maximum number of resident blocks per multiprocessor.

Thinking:
1. The goal is to query CUDA device properties to obtain the value for the maximum resident blocks that can be scheduled on a single multiprocessor (SM) at a time. This value is stored in the hipDeviceProp_t structure field `maxBlocksPerMultiprocessor`.
2. To do this, we use the CUDA Runtime API. Specifically, we call `hipGetDeviceProperties(&prop, dev)` to fill a `hipDeviceProp_t` struct for a given device index (we use 0 for the first device).
3. Once the properties are retrieved, we simply print the field `prop.maxBlocksPerMultiprocessor` using standard C I/O (`printf`).
4. We also perform basic error checking: if `hipGetDeviceProperties` returns a non-zero error code, we print an error message and exit.
5. The program is intended to be compiled with `nvcc`. Therefore we include `<hip/hip_runtime.h>` for CUDA runtime functions and `<stdio.h>` for printing.
6. No other supporting code or output is required; the program prints only the requested value.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
    hipDeviceProp_t prop;
    int device = 0; // Use the first CUDA device

    hipError_t err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to get device properties for device %d: %s\n", device, hipGetErrorString(err));
        return -1;
    }

    printf("Maximum resident blocks per multiprocessor: %d\n", prop.maxBlocksPerMultiprocessor);

    return 0;
}
```