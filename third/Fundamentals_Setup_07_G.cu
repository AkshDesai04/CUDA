```c
/*
 * Aim of the program: Print the `major` and `minor` compute capability numbers for the primary device.
 *
 * Thinking:
 * 1. To obtain the compute capability of the current primary CUDA device, we need to query the device properties.
 * 2. The CUDA Runtime API provides `hipGetDevice` to get the current device id, and `hipGetDeviceProperties`
 *    to retrieve a `hipDeviceProp_t` structure containing information about the device, including the major
 *    and minor compute capability fields.
 * 3. We'll call `hipGetDevice(&dev)` to get the device id, then `hipGetDeviceProperties(&prop, dev)` to fill the
 *    properties struct. After that, we simply print `prop.major` and `prop.minor`.
 * 4. Error handling is minimal; we just check the return status of the CUDA API calls and exit with a message if
 *    something goes wrong.
 * 5. The output will be a simple text line: "Compute capability: X.Y" where X and Y are the major and minor numbers.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int device = 0;
    hipError_t err;

    // Get the current device
    err = hipGetDevice(&device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get current device: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Retrieve device properties
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device properties: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Print the compute capability
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);

    return 0;
}
```