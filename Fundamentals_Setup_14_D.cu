#include "hip/hip_runtime.h"
```cuda
/*
Aim of the program: Add a comment explaining what a CUDA stream is and how it relates to overlapping operations.

My thinking:
- The user asked for a CUDA program that includes a comment explaining what a CUDA stream is and how it can be used to overlap operations. 
- To demonstrate this concept, the code will allocate memory, create two CUDA streams, perform an asynchronous memory copy in one stream, launch a kernel in the other stream, and use hipDeviceSynchronize() to wait for completion.
- The comment explaining CUDA streams will be added inside the code near the stream creation, providing a clear description for anyone reading the source.
- The program will be minimal but functional: a simple vector addition kernel, asynchronous memory copy, and two separate streams to show potential overlap.
- We'll also include basic error checking for robustness.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

/* Macro for checking CUDA errors */
#define CUDA_CHECK(err)                                                      \
    if (err != hipSuccess) {                                                \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,     \
                hipGetErrorString(err));                                    \
        exit(EXIT_FAILURE);                                                  \
    }

/* Simple kernel: add two vectors */
__global__ void vecAdd(const float *A, const float *B, float *C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) C[idx] = A[idx] + B[idx];
}

int main(void)
{
    const int N = 1 << 20;               // 1M elements
    const size_t bytes = N * sizeof(float);

    /* Allocate host memory */
    float *h_A = (float *)malloc(bytes);
    float *h_B = (float *)malloc(bytes);
    float *h_C = (float *)malloc(bytes);

    /* Initialize host arrays */
    for (int i = 0; i < N; ++i) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    /* Allocate device memory */
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void **)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_C, bytes));

    /* Create two non-blocking streams */
    hipStream_t streamMemcpy, streamKernel;
    CUDA_CHECK(hipStreamCreateWithFlags(&streamMemcpy, hipStreamNonBlocking));
    CUDA_CHECK(hipStreamCreateWithFlags(&streamKernel, hipStreamNonBlocking));

    /* Asynchronously copy host to device in streamMemcpy */
    CUDA_CHECK(hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice, streamMemcpy));
    CUDA_CHECK(hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice, streamMemcpy));

    /* 
     * CUDA Stream explanation:
     * A CUDA stream is a sequence of operations that execute on the device in order. 
     * Operations in different streams may execute concurrently, provided there are no
     * dependencies between them and the hardware resources allow it. By using multiple
     * streams, we can overlap memory transfers (host-to-device or device-to-host)
     * with kernel execution, thereby improving overall performance. 
     * In this example, the memory copy operations are queued in streamMemcpy, while
     * the kernel launch is queued in streamKernel. If the GPU has enough resources,
     * the kernel execution can overlap with the memory copies.
     */

    /* Launch kernel in streamKernel */
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streamKernel>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());

    /* Asynchronously copy result back to host */
    CUDA_CHECK(hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost, streamKernel));

    /* Wait for all operations to finish */
    CUDA_CHECK(hipStreamSynchronize(streamMemcpy));
    CUDA_CHECK(hipStreamSynchronize(streamKernel));

    /* Verify results (simple check) */
    for (int i = 0; i < N; ++i) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            fprintf(stderr, "Mismatch at index %d: %f != %f\n", i, h_C[i], h_A[i] + h_B[i]);
            break;
        }
    }
    printf("Vector addition completed successfully.\n");

    /* Clean up */
    CUDA_CHECK(hipStreamDestroy(streamMemcpy));
    CUDA_CHECK(hipStreamDestroy(streamKernel));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
```