#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define N 1024  // Matrix size (1024x1024)

void generateRandomMatrix(float *matrix, int size) {
    for (int i = 0; i < size * size; ++i) {
        matrix[i] = (float)(rand() % 100);  // Random values between 0 and 99
    }
}

int main() {
    float *A, *B, *C;   // Host matrices
    float *d_A, *d_B, *d_C;   // Device matrices

    size_t size = N * N * sizeof(float);

    // Allocate host memory
    A = (float *)malloc(size);
    B = (float *)malloc(size);
    C = (float *)malloc(size);

    // Generate random matrices
    generateRandomMatrix(A, N);
    generateRandomMatrix(B, N);

    // Allocate device memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform matrix multiplication using cuBLAS (C = A * B)
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print part of the result (optional)
    // for (int i = 0; i < 5; i++) {
    //     for (int j = 0; j < 5; j++) {
    //         printf("%f ", C[i * N + j]);
    //     }
    //     printf("\n");
    // }

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}
