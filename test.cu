#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 1024  // Matrix size
#define BLOCK_SIZE 16  // Block size for kernel launch

// CUDA kernel to perform matrix multiplication
__global__ void matrixMultiply(int *A, int *B, int *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

// Function to generate random integer matrix
void generateRandomMatrix(int *matrix, int N) {
    for (int i = 0; i < N * N; ++i) {
        matrix[i] = rand() % 100;  // Random integers between 0 and 99
    }
}

// Function to print a matrix (for debugging)
void printMatrix(int *matrix, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", matrix[i * N + j]);
        }
        printf("\n");
    }
}

int main() {
    int *A, *B, *C;  // Host matrices
    int *d_A, *d_B, *d_C;  // Device matrices
    size_t size = N * N * sizeof(int);

    // Initialize random seed
    srand(time(0));

    // Allocate memory for matrices on host
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    // Generate random matrices A and B
    generateRandomMatrix(A, N);
    generateRandomMatrix(B, N);

    // Print matrices for debugging (optional)
    // printMatrix(A, N);
    // printMatrix(B, N);

    // Use the default GPU
    hipSetDevice(0);  // Automatically uses the default device (GPU 0)
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s\n", prop.name);

    // Allocate memory on the device
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the kernel for matrix multiplication
    matrixMultiply<<<grid, block>>>(d_A, d_B, d_C, N);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result (optional)
    // printMatrix(C, N);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
