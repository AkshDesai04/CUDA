```c
/* 
Aim of the program:
Check if the device supports peer-to-peer access with other GPUs (`p2pAccessSupported`). 
This means for each pair of GPUs in the system we will query the CUDA runtime API
`hipDeviceCanAccessPeer()` to determine whether one device can directly read from
another. The program enumerates all devices, and for each unique pair prints a
message indicating whether P2P is supported. If only a single GPU is present
the program will note that there are no peers to check.

Thinking:
1. Use `hipGetDeviceCount()` to get the number of GPUs.
2. If less than 2, print a message that no peers exist.
3. Otherwise, loop over all device indices `i` and `j` where `j > i` to avoid
   duplicate pairs.
4. For each pair, call `hipDeviceCanAccessPeer(i, j)`. This returns an
   integer: 1 if peer access is possible, 0 otherwise. It also may return an
   error if the call itself fails; check the returned `hipError_t` and print
   a helpful message.
5. Compile with `nvcc` and run; output will be printed to stdout.

This program is minimal, self-contained, and uses only the CUDA runtime API,
so it will compile on any system with the CUDA toolkit installed.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Error getting device count: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Number of CUDA devices detected: %d\n", deviceCount);

    if (deviceCount < 2) {
        printf("Not enough devices to check peer-to-peer access.\n");
        return 0;
    }

    // Iterate over all unique device pairs
    for (int src = 0; src < deviceCount; ++src) {
        for (int dst = src + 1; dst < deviceCount; ++dst) {
            int canAccess = 0;
            err = hipDeviceCanAccessPeer(&canAccess, src, dst);
            if (err != hipSuccess) {
                fprintf(stderr,
                        "Error checking p2p from device %d to device %d: %s\n",
                        src, dst, hipGetErrorString(err));
                continue;
            }

            printf("Device %d can%s access Device %d directly.\n",
                   src,
                   canAccess ? "" : " NOT",
                   dst);

            // Also check the reverse direction
            err = hipDeviceCanAccessPeer(&canAccess, dst, src);
            if (err != hipSuccess) {
                fprintf(stderr,
                        "Error checking p2p from device %d to device %d: %s\n",
                        dst, src, hipGetErrorString(err));
                continue;
            }

            printf("Device %d can%s access Device %d directly.\n",
                   dst,
                   canAccess ? "" : " NOT",
                   src);
        }
    }

    return 0;
}
```