/*
First, get the device count. Then, try to set the device to `count` (which is an invalid index, since they are 0-based).

**Thinking Process:**

1. The program’s aim is to demonstrate the behavior of the CUDA runtime when an invalid device index is used.  
2. We will call `cudaGetDeviceCount` to obtain the number of CUDA-capable devices available on the system.  
3. `cudaGetDeviceCount` returns the count as a positive integer (e.g., 1, 2, …).  
4. CUDA device indices are 0‑based, so the valid range for `cudaSetDevice` is `[0, count-1]`.  
5. By intentionally passing the value `count` to `cudaSetDevice`, we provide an index that is *one* larger than the highest valid index.  
6. This should trigger an error from the runtime – typically `cudaErrorInvalidDevice`.  
7. The program will capture this error, print the error code and message, and exit.  
8. This exercise is useful for testing error handling and for understanding the importance of valid device indices.

The code below performs these steps and prints the result of the attempted device set operation.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("Failed to get device count: %s (%d)\n", hipGetErrorString(err), err);
        return -1;
    }

    printf("Number of CUDA devices found: %d\n", deviceCount);

    // Intentionally use an invalid device index: count (which is one greater than the max valid index)
    int invalidIndex = deviceCount;
    err = hipSetDevice(invalidIndex);
    if (err != hipSuccess) {
        printf("Attempt to set device to %d failed: %s (%d)\n", invalidIndex, hipGetErrorString(err), err);
    } else {
        printf("Successfully set device to %d (unexpected)\n", invalidIndex);
    }

    return 0;
}
