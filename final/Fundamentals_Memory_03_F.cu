```cpp
/*
Allocate `d_a` and `d_b`. Use `hipMemcpy(d_b, h_a, size, hipMemcpyHostToDevice)` to initialize `d_a`. What is the error? Change `h_a` to `d_a`. What is the error? Explain why the kind parameter is critical.

My thinking:
- The goal is to illustrate two common mistakes when using hipMemcpy.
- First, the user incorrectly copies host data into the wrong device pointer (`d_b`) while intending to initialize `d_a`. The copy succeeds from the host viewpoint, but `d_a` remains uninitialized, leading to wrong results downstream. The error is logical, not a runtime error, because the source and destination pointers are both valid memory addresses (host to device), so hipMemcpy will not report an error, but the data will be wrong.
- Second, the user then tries to copy from device to device by passing a device pointer (`d_a`) as the source but still using the kind `hipMemcpyHostToDevice`. The runtime expects a host pointer for the source, so it will flag an error (invalid value). The kind parameter tells the runtime which memory space each argument refers to. Using the wrong kind leads to either undefined behaviour or explicit errors.
- This demonstrates that the `kind` parameter is critical: it specifies the direction of the copy and, implicitly, which side (host or device) each pointer refers to. Mismatching this can result in data corruption or runtime errors.

The code below allocates two device arrays (`d_a`, `d_b`) and a host array (`h_a`). It then performs the two erroneous copies and prints any CUDA error codes. Finally, it shows the correct usage for each operation.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    const int N = 10;
    const size_t size = N * sizeof(float);

    /* Host data */
    float h_a[N];
    for (int i = 0; i < N; ++i) h_a[i] = (float)i;

    /* Device pointers */
    float *d_a = nullptr;
    float *d_b = nullptr;

    hipError_t err;

    /* Allocate device memory */
    err = hipMalloc((void**)&d_a, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc d_a failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipMalloc((void**)&d_b, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc d_b failed: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        return 1;
    }

    /* --------------------------------------------------------------------
     * Mistake 1: Copy host data into d_b but claim to initialize d_a.
     * The copy operation is legal (Host -> Device) but writes to the wrong
     * device buffer. No CUDA error is reported; the data in d_a remains
     * uninitialized (contains garbage). This is a logical error.
     * -------------------------------------------------------------------- */
    err = hipMemcpy(d_b, h_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpy(d_b, h_a, size, hipMemcpyHostToDevice): %s\n",
                hipGetErrorString(err));
    } else {
        printf("hipMemcpy(d_b, h_a, size, hipMemcpyHostToDevice) succeeded.\n");
        printf("d_a remains uninitialized; data will be wrong if used.\n");
    }

    /* --------------------------------------------------------------------
     * Mistake 2: Copy from d_a to d_b but still use hipMemcpyHostToDevice.
     * The runtime expects a host pointer as the source, but receives a device
     * pointer. This leads to a hipErrorInvalidValue.
     * -------------------------------------------------------------------- */
    err = hipMemcpy(d_b, d_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpy(d_b, d_a, size, hipMemcpyHostToDevice): %s\n",
                hipGetErrorString(err));
    } else {
        printf("Unexpected success: hipMemcpy(d_b, d_a, size, hipMemcpyHostToDevice) succeeded.\n");
    }

    /* --------------------------------------------------------------------
     * Correct usage:
     *   1. Initialize d_a from host: hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice)
     *   2. Copy from d_a to d_b (device-to-device): hipMemcpy(d_b, d_a, size, hipMemcpyDeviceToDevice)
     * -------------------------------------------------------------------- */
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Correct copy d_a <- h_a failed: %s\n", hipGetErrorString(err));
    } else {
        printf("Correctly copied h_a to d_a.\n");
    }

    err = hipMemcpy(d_b, d_a, size, hipMemcpyDeviceToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Correct copy d_b <- d_a failed: %s\n", hipGetErrorString(err));
    } else {
        printf("Correctly copied d_a to d_b (device-to-device).\n");
    }

    /* Optional: verify contents of d_b by copying back to host */
    float h_b[N];
    err = hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Copy back d_b to h_b failed: %s\n", hipGetErrorString(err));
    } else {
        printf("Contents of d_b after correct copy:\n");
        for (int i = 0; i < N; ++i) {
            printf("%f ", h_b[i]);
        }
        printf("\n");
    }

    /* Clean up */
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
```