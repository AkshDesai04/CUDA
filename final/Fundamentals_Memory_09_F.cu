```cpp
/*
Write a function that takes a boolean `use_pinned_memory` and performs the timed transfer, abstracting the allocation/deallocation logic.

Thought process:
- The goal is to encapsulate all the allocation, copy, and cleanup steps in a single function that can be reused for benchmarking or other purposes.
- The function should accept a `bool use_pinned_memory` flag to decide whether the host memory should be page‑locked (pinned) or regular pageable memory.
- We'll need to allocate host memory using either `hipHostMalloc` (for pinned) or `malloc` (for pageable). Correspondingly we must free with `hipHostFree` or `free`.
- Device memory will always be allocated with `hipMalloc` and freed with `hipFree`.
- Timing: Use CUDA events (`hipEvent_t`) to accurately measure GPU copy time. Record a start event, perform `hipMemcpy`, record a stop event, then compute elapsed time.
- Return the elapsed time in milliseconds. In case of any error, print an error message and return a negative value.
- The function will also print details of the transfer, such as bytes transferred and memory type used, for debugging purposes.
- After the transfer, all allocated memory is deallocated to avoid leaks.
- We'll provide a simple `main` that calls this function with both pinned and non‑pinned options to demonstrate usage.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// Helper macro for checking CUDA errors
#define CUDA_CHECK(err)                                      \
    do {                                                     \
        hipError_t _e = (err);                              \
        if (_e != hipSuccess) {                             \
            std::cerr << "CUDA error at " << __FILE__        \
                      << ":" << __LINE__ << ": "            \
                      << hipGetErrorString(_e) << std::endl; \
            std::exit(EXIT_FAILURE);                         \
        }                                                    \
    } while (0)

// Function that abstracts allocation, transfer, timing, and deallocation
double timedTransfer(bool use_pinned_memory, size_t size_in_bytes) {
    void* h_ptr = nullptr;
    void* d_ptr = nullptr;

    // Allocate host memory
    if (use_pinned_memory) {
        CUDA_CHECK(hipHostMalloc(&h_ptr, size_in_bytes));
    } else {
        h_ptr = std::malloc(size_in_bytes);
        if (!h_ptr) {
            std::cerr << "Failed to allocate pageable host memory." << std::endl;
            return -1.0;
        }
    }

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_ptr, size_in_bytes));

    // Initialize host memory (optional)
    std::memset(h_ptr, 0xAA, size_in_bytes);

    // Timing events
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Record start event
    CUDA_CHECK(hipEventRecord(start, 0));

    // Perform host-to-device copy
    CUDA_CHECK(hipMemcpy(d_ptr, h_ptr, size_in_bytes, hipMemcpyHostToDevice));

    // Record stop event
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));

    // Compute elapsed time
    float elapsed_ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));

    // Clean up events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // Deallocate memory
    if (use_pinned_memory) {
        CUDA_CHECK(hipHostFree(h_ptr));
    } else {
        std::free(h_ptr);
    }
    CUDA_CHECK(hipFree(d_ptr));

    // Print transfer details
    std::cout << "Transferred " << size_in_bytes / (1024.0 * 1024.0)
              << " MB (" << (use_pinned_memory ? "Pinned" : "Pageable")
              << " host) in " << elapsed_ms << " ms." << std::endl;

    return static_cast<double>(elapsed_ms);
}

int main() {
    const size_t transfer_size = 256 * 1024 * 1024; // 256 MB

    std::cout << "Benchmarking with pageable host memory:" << std::endl;
    timedTransfer(false, transfer_size);

    std::cout << "\nBenchmarking with pinned host memory:" << std::endl;
    timedTransfer(true, transfer_size);

    return 0;
}
```