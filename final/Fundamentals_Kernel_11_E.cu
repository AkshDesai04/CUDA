```cpp
/*
From the host, use `hipMemset` to zero out the struct's memory on the device after allocating it.

Thinking:
1. Define a simple struct on the host and device side (same layout) to hold some example data.
2. Allocate memory for this struct on the device with hipMalloc.
3. Use hipMemset from the host to set all bytes of the allocated struct to zero.
4. Copy the struct back to the host with hipMemcpy to verify that all fields are zeroed.
5. Provide minimal error checking for CUDA API calls.
6. The code is a self‑contained .cu file that compiles with nvcc and prints the values of the struct before and after the memset operation.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

// Simple struct with different types
struct Data {
    int   i;
    float f;
    double d;
};

// Utility macro for CUDA error checking
#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",                 \
                    #call, __FILE__, __LINE__, hipGetErrorString(err));      \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

int main() {
    // Allocate host instance and initialize with non-zero values
    Data hostData{42, 3.14f, 2.718};
    printf("Host before memset: i=%d, f=%f, d=%f\n", hostData.i, hostData.f, hostData.d);

    // Device pointer
    Data* devPtr = nullptr;
    CUDA_CHECK(hipMalloc(&devPtr, sizeof(Data)));

    // Copy initial data to device
    CUDA_CHECK(hipMemcpy(devPtr, &hostData, sizeof(Data), hipMemcpyHostToDevice));

    // Use hipMemset to zero out the struct on the device
    CUDA_CHECK(hipMemset(devPtr, 0, sizeof(Data)));

    // Copy back to host to verify
    CUDA_CHECK(hipMemcpy(&hostData, devPtr, sizeof(Data), hipMemcpyDeviceToHost));

    printf("Host after memset: i=%d, f=%f, d=%f\n", hostData.i, hostData.f, hostData.d);

    // Clean up
    CUDA_CHECK(hipFree(devPtr));

    return 0;
}
```