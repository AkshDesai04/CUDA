#include "hip/hip_runtime.h"
```cuda
/*
Aim: Query and print the memory clock rate of the primary GPU.

Thinking:
- The program will use the CUDA Runtime API to query properties of the primary device (device 0).
- hipGetDeviceProperties fills a hipDeviceProp_t struct, which contains a field `memoryClockRate` in kHz.
- We'll convert this value to MHz for a more readable output.
- Basic error checking will be performed for CUDA API calls.
- The program will compile as a .cu file and can be run with `nvcc` to display the memory clock rate.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Helper macro for error checking
#define CUDA_CHECK(call)                                                     \
    do {                                                                     \
        hipError_t err = (call);                                            \
        if (err != hipSuccess) {                                            \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                     \
                    __FILE__, __LINE__, hipGetErrorString(err));           \
            return EXIT_FAILURE;                                            \
        }                                                                    \
    } while (0)

int main(void) {
    int device = 0; // Primary GPU
    hipDeviceProp_t prop;

    // Get properties of the device
    CUDA_CHECK(hipGetDeviceProperties(&prop, device));

    // memoryClockRate is in kHz; convert to MHz
    float memory_clock_mhz = prop.memoryClockRate / 1000.0f;

    printf("Primary GPU (device %d) memory clock rate: %.1f MHz\n",
           device, memory_clock_mhz);

    return EXIT_SUCCESS;
}
```