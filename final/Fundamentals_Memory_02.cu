```cuda
/*
Aim: Extend the previous program. After copying data to the device, create a second host array of 256 integers. Copy the data from the device array back to this new host array using hipMemcpy with the hipMemcpyDeviceToHost kind. Verify that every element in the new host array matches the original.

Thinking:
1. Allocate a host array `h_src` of 256 integers and initialize it with known values (e.g., 0..255).
2. Allocate device memory `d_data` of size 256 * sizeof(int).
3. Copy `h_src` to `d_data` using hipMemcpy with hipMemcpyHostToDevice.
4. Allocate a second host array `h_dst` of 256 integers to receive the copy back.
5. Copy from `d_data` to `h_dst` using hipMemcpy with hipMemcpyDeviceToHost.
6. Verify that each element of `h_dst` matches the corresponding element in `h_src`.
   - If all match, print "Verification passed".
   - Otherwise, print the index of the first mismatch and its values.
7. Clean up: free device memory and exit.
8. Include basic CUDA error checking after each CUDA API call.
9. Keep the code simple and self‑contained, using only standard headers and CUDA runtime API.

This file can be compiled with `nvcc`:
    nvcc -o verify copy_verify.cu
and run with `./verify`.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

#define CHECK_CUDA(call)                                                        \
    do {                                                                        \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error in function '" << __FUNCTION__            \
                      << "' at line " << __LINE__ << ": "                      \
                      << hipGetErrorString(err) << std::endl;                 \
            exit(EXIT_FAILURE);                                                \
        }                                                                       \
    } while (0)

int main() {
    const int N = 256;
    int h_src[N];
    int h_dst[N];

    // Initialize source array with values 0..255
    for (int i = 0; i < N; ++i) {
        h_src[i] = i;
    }

    // Allocate device memory
    int *d_data = nullptr;
    CHECK_CUDA(hipMalloc((void**)&d_data, N * sizeof(int)));

    // Copy from host to device
    CHECK_CUDA(hipMemcpy(d_data, h_src, N * sizeof(int), hipMemcpyHostToDevice));

    // Copy back from device to second host array
    CHECK_CUDA(hipMemcpy(h_dst, d_data, N * sizeof(int), hipMemcpyDeviceToHost));

    // Verify that each element matches
    bool ok = true;
    for (int i = 0; i < N; ++i) {
        if (h_dst[i] != h_src[i]) {
            std::cerr << "Mismatch at index " << i
                      << ": src=" << h_src[i]
                      << " dst=" << h_dst[i] << std::endl;
            ok = false;
            break;
        }
    }

    if (ok) {
        std::cout << "Verification passed: all elements match." << std::endl;
    } else {
        std::cerr << "Verification failed." << std::endl;
    }

    // Clean up
    CHECK_CUDA(hipFree(d_data));

    return ok ? EXIT_SUCCESS : EXIT_FAILURE;
}
```