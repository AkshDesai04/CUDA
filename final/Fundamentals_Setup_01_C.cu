```cuda
/*
Write a program to explicitly set the active CUDA device to 0 using `hipSetDevice`. Verify the call was successful by then calling `hipGetDevice` and printing the result.

Thinking:
1. The program should be a self‑contained CUDA source file (.cu) that can be compiled with nvcc.
2. We need to include the CUDA runtime header `<hip/hip_runtime.h>` and standard IO headers for printing.
3. In `main`, we will:
   a. Call `hipSetDevice(0)` and capture the return value.
   b. If the call fails, print an error message using `hipGetErrorString` and exit.
   c. Call `hipGetDevice(&deviceId)` to retrieve the current active device.
   d. Print the device ID and a success message.
4. We should check the return value of `hipGetDevice` as well.
5. For simplicity, we use `printf` from `<cstdio>`. No kernel launch is required.
6. The program will return 0 on success, or a non‑zero error code on failure.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

int main(void)
{
    int device = 0;
    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set device %d: %s\n", device, hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    int currentDevice = -1;
    err = hipGetDevice(&currentDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get current device: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    printf("Successfully set and retrieved CUDA device. Current device: %d\n", currentDevice);
    return EXIT_SUCCESS;
}
```