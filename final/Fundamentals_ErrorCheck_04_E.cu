/*
Aim: Trigger the error by passing a `nullptr` for the event in `cudaEventCreate`.

Thinking: To trigger a CUDA runtime error, we intentionally pass a null pointer to cudaEventCreate, which expects a pointer to a cudaEvent_t. Passing nullptr violates the API contract, leading to cudaErrorInvalidValue. The program will call cudaEventCreate with a null pointer, check the returned error code, and print it using cudaGetErrorString. This demonstrates error handling.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    // Intentionally pass a nullptr to cudaEventCreate
    hipEvent_t *event = nullptr;
    hipError_t err = hipEventCreate(event);

    // Check and report the error
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipEventCreate failed: %s\n", hipGetErrorString(err));
    }
    else
    {
        // This branch should not be executed because the call should fail
        hipEventDestroy(*event);
    }

    // Reset the device before exiting
    hipDeviceReset();

    return 0;
}
