```cuda
/*

Aim of the program:
Extend the IPC example to share a CUDA event. Process A records an event, gets an IPC handle for it, passes it to Process B, which then waits on that event.

Thoughts:
This example demonstrates how to use CUDA IPC (Inter‑Process Communication) with events. Two processes are represented in the same executable, distinguished by a command‑line argument ("producer" or "consumer"). 
- The producer creates a CUDA event, records it on a stream, obtains its IPC handle using `hipIpcGetEventHandle`, and writes that handle to a binary file (`event_handle.bin`). 
- The consumer reads the handle from the file, imports it with `hipIpcOpenEventHandle`, and then waits for the event using `hipEventSynchronize`. 

The file is used as a simple communication medium; in real applications you might use sockets, shared memory, or other IPC mechanisms. All CUDA API calls are checked for errors, and the program prints informative messages. The example compiles with `nvcc`.

*/

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <iostream>
#include <unistd.h>

void checkCudaErr(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        const char *errStr;
        hipDrvGetErrorString(err, &errStr);
        std::cerr << msg << ": " << errStr << std::endl;
        exit(EXIT_FAILURE);
    }
}

void producer() {
    hipDevice_t device;
    hipCtx_t ctx;
    hipStream_t stream;
    hipEvent_t event;
    hipIpcEventHandle_t eventHandle;

    // Initialize CUDA
    checkCudaErr(hipInit(0), "hipInit");
    checkCudaErr(hipDeviceGet(&device, 0), "hipDeviceGet");
    checkCudaErr(hipCtxCreate(&ctx, 0, device), "hipCtxCreate");

    // Create a stream
    checkCudaErr(hipStreamCreateWithFlags(&stream, hipStreamDefault), "hipStreamCreateWithFlags");

    // Create an event
    checkCudaErr(hipEventCreateWithFlags(&event, hipEventDefault), "hipEventCreateWithFlags");

    // Record the event on the stream
    checkCudaErr(hipEventRecord(event, stream), "hipEventRecord");

    // Get IPC handle for the event
    checkCudaErr(hipIpcGetEventHandle(&eventHandle, event), "hipIpcGetEventHandle");

    // Write the handle to a file
    std::ofstream ofs("event_handle.bin", std::ios::binary);
    if (!ofs) {
        std::cerr << "Failed to open file for writing IPC handle." << std::endl;
        exit(EXIT_FAILURE);
    }
    ofs.write(reinterpret_cast<const char*>(&eventHandle), sizeof(eventHandle));
    ofs.close();
    std::cout << "Producer: IPC event handle written to 'event_handle.bin'.\n";

    // Optional: Wait a bit to keep the event alive
    sleep(1);

    // Clean up
    checkCudaErr(hipEventDestroy(event), "hipEventDestroy");
    checkCudaErr(hipStreamDestroy(stream), "hipStreamDestroy");
    checkCudaErr(hipCtxDestroy(ctx), "hipCtxDestroy");
}

void consumer() {
    hipDevice_t device;
    hipCtx_t ctx;
    hipEvent_t event;

    // Initialize CUDA
    checkCudaErr(hipInit(0), "hipInit");
    checkCudaErr(hipDeviceGet(&device, 0), "hipDeviceGet");
    checkCudaErr(hipCtxCreate(&ctx, 0, device), "hipCtxCreate");

    // Read the IPC handle from the file
    std::ifstream ifs("event_handle.bin", std::ios::binary);
    if (!ifs) {
        std::cerr << "Failed to open IPC handle file." << std::endl;
        exit(EXIT_FAILURE);
    }
    hipIpcEventHandle_t eventHandle;
    ifs.read(reinterpret_cast<char*>(&eventHandle), sizeof(eventHandle));
    ifs.close();

    // Open the event handle
    checkCudaErr(hipIpcOpenEventHandle(&event, &eventHandle), "hipIpcOpenEventHandle");
    std::cout << "Consumer: IPC event handle opened.\n";

    // Wait for the event to complete
    std::cout << "Consumer: Waiting for event to complete...\n";
    checkCudaErr(hipEventSynchronize(event), "hipEventSynchronize");
    std::cout << "Consumer: Event completed.\n";

    // Clean up
    checkCudaErr(hipEventDestroy(event), "hipEventDestroy");
    checkCudaErr(hipCtxDestroy(ctx), "hipCtxDestroy");
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " [producer|consumer]" << std::endl;
        return EXIT_FAILURE;
    }

    std::string role(argv[1]);
    if (role == "producer") {
        producer();
    } else if (role == "consumer") {
        consumer();
    } else {
        std::cerr << "Unknown role: " << role << ". Use 'producer' or 'consumer'." << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
```