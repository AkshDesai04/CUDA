#include "hip/hip_runtime.h"
```cpp
/*
Aim of the program:
Use `hipMemPrefetchAsync` to proactively migrate data to a specific processor (CPU or a specific GPU) before it's accessed to hide migration latency.

Thinking:
- To demonstrate `hipMemPrefetchAsync`, we use Unified Memory (`hipMallocManaged`) because it allows the memory to be visible to both CPU and GPU.
- The program allocates a large array of floats and initializes it on the CPU.
- Before launching a simple kernel that adds two arrays, we proactively prefetch the managed memory to the GPU device using `hipMemPrefetchAsync`. This moves the data to GPU memory ahead of the kernel launch, hiding the migration latency that would otherwise occur at first GPU access.
- After the kernel finishes, we prefetch the data back to the CPU in the same proactive way, so that the CPU can read the results without waiting for the implicit migration.
- CUDA events are used to time the kernel execution and the migration prefetches. By comparing the timings with and without prefetching, we can observe the benefit.
- The code checks for errors at each CUDA API call, prints the timings, and verifies that the computation result is correct.
- The example uses device 0 (or the first available GPU). If the system has multiple GPUs, the code can be adapted to prefetch to a specific GPU by passing its device ID to `hipMemPrefetchAsync`.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define CHECK(call)                                                    \
    do {                                                               \
        hipError_t err = call;                                        \
        if (err != hipSuccess) {                                      \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",          \
                    #call, __FILE__, __LINE__, hipGetErrorString(err));\
            exit(EXIT_FAILURE);                                        \
        }                                                              \
    } while (0)

// Simple vector addition kernel
__global__ void vectorAdd(const float *a, const float *b, float *c, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) c[idx] = a[idx] + b[idx];
}

int main(void) {
    // Determine the device to use (device 0)
    int device = 0;
    int deviceCount = 0;
    CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable devices found.\n");
        return EXIT_FAILURE;
    }
    if (device >= deviceCount) device = deviceCount - 1;
    CHECK(hipSetDevice(device));

    const size_t N = 1 << 24; // 16M elements (~64 MB)
    size_t bytes = N * sizeof(float);

    // Allocate unified memory
    float *a, *b, *c;
    CHECK(hipMallocManaged(&a, bytes));
    CHECK(hipMallocManaged(&b, bytes));
    CHECK(hipMallocManaged(&c, bytes));

    // Initialize arrays on host
    for (size_t i = 0; i < N; ++i) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Create CUDA events for timing
    hipEvent_t startEvent, stopEvent;
    CHECK(hipEventCreate(&startEvent));
    CHECK(hipEventCreate(&stopEvent));

    // ----- Pre-fetch data to GPU before kernel -----
    // Record event before prefetch
    CHECK(hipEventRecord(startEvent, 0));
    // Prefetch a, b, c to device
    CHECK(hipMemPrefetchAsync(a, bytes, device, 0));
    CHECK(hipMemPrefetchAsync(b, bytes, device, 0));
    CHECK(hipMemPrefetchAsync(c, bytes, device, 0));
    // Wait for prefetch to finish
    CHECK(hipEventRecord(stopEvent, 0));
    CHECK(hipEventSynchronize(stopEvent));

    float prefetchtime_ms = 0.0f;
    CHECK(hipEventElapsedTime(&prefetchtime_ms, startEvent, stopEvent));

    // ----- Launch kernel -----
    CHECK(hipEventRecord(startEvent, 0));
    size_t blockSize = 256;
    size_t gridSize = (N + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(a, b, c, N);
    CHECK(hipGetLastError());
    CHECK(hipEventRecord(stopEvent, 0));
    CHECK(hipEventSynchronize(stopEvent));

    float kernelTime_ms = 0.0f;
    CHECK(hipEventElapsedTime(&kernelTime_ms, startEvent, stopEvent));

    // ----- Pre-fetch result back to CPU after kernel -----
    CHECK(hipEventRecord(startEvent, 0));
    CHECK(hipMemPrefetchAsync(c, bytes, hipCpuDeviceId, 0));
    CHECK(hipEventRecord(stopEvent, 0));
    CHECK(hipEventSynchronize(stopEvent));

    float postPrefetchTime_ms = 0.0f;
    CHECK(hipEventElapsedTime(&postPrefetchTime_ms, startEvent, stopEvent));

    // Verify results
    bool success = true;
    for (size_t i = 0; i < N; ++i) {
        if (c[i] != 3.0f) {
            success = false;
            fprintf(stderr, "Mismatch at index %zu: %f\n", i, c[i]);
            break;
        }
    }

    printf("Vector addition of %zu elements completed.\n", N);
    printf("Data prefetch to GPU: %.3f ms\n", prefetchtime_ms);
    printf("Kernel execution:     %.3f ms\n", kernelTime_ms);
    printf("Data prefetch to CPU: %.3f ms\n", postPrefetchTime_ms);
    printf("Computation %s.\n", success ? "succeeded" : "failed");

    // Clean up
    CHECK(hipFree(a));
    CHECK(hipFree(b));
    CHECK(hipFree(c));
    CHECK(hipEventDestroy(startEvent));
    CHECK(hipEventDestroy(stopEvent));

    return 0;
}
```