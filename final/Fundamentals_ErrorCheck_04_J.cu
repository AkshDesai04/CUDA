/*
Pass a size of 0 to `cudaMemcpy`. Is this an error?

My reasoning:
- According to the CUDA Runtime API, cudaMemcpy takes a size argument in bytes.
- If size is 0, the function should effectively do nothing (no data transfer).
- The API documentation states that passing a size of 0 is allowed and should return cudaSuccess.
- We will test this by calling cudaMemcpy with a 0 size and checking the returned error code.
- We'll allocate some dummy memory, but not copy anything.
- We expect the function to return cudaSuccess, confirming that size 0 is not an error.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    // Allocate some host memory
    size_t hostSize = 10;
    int *h_data = (int*)malloc(hostSize * sizeof(int));
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return -1;
    }

    // Allocate device memory
    int *d_data = NULL;
    hipError_t err = hipMalloc((void**)&d_data, hostSize * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        free(h_data);
        return -1;
    }

    // Perform cudaMemcpy with size 0 (Host to Device)
    err = hipMemcpy(d_data, h_data, 0, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy HtoD with size 0 failed: %s\n", hipGetErrorString(err));
    } else {
        printf("hipMemcpy HtoD with size 0 succeeded (expected).\n");
    }

    // Perform cudaMemcpy with size 0 (Device to Host)
    err = hipMemcpy(h_data, d_data, 0, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy DtoH with size 0 failed: %s\n", hipGetErrorString(err));
    } else {
        printf("hipMemcpy DtoH with size 0 succeeded (expected).\n");
    }

    // Clean up
    hipFree(d_data);
    free(h_data);

    return 0;
}
