```cuda
/*
Aim of the program: Use the cuRAND library to generate a large array of random numbers directly on the GPU. Create a generator, set its seed, and call hiprandGenerateUniform to fill a device array with floating-point numbers between 0.0 and 1.0.

Thinking:
1. We need to include CUDA runtime, cuRAND headers and iostream for output.
2. Define a macro CURAND_CHECK to wrap cuRAND API calls for error checking.
3. In main, choose a large array size, e.g., 10 million floats.
4. Allocate device memory with hipMalloc.
5. Create a cuRAND generator of type HIPRAND_RNG_PSEUDO_DEFAULT using hiprandCreateGenerator.
6. Set the seed with hiprandSetPseudoRandomGeneratorSeed.
7. Call hiprandGenerateUniform to fill the device array.
8. Copy a small portion back to host to verify randomness.
9. Clean up: free device memory, destroy generator, check for CUDA errors.
10. Print a few numbers to confirm they're in [0,1].
*/

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <vector>

// Macro for checking cuRAND API calls
#define CURAND_CHECK(call)                                                    \
    do {                                                                       \
        hiprandStatus_t err = call;                                            \
        if (err != HIPRAND_STATUS_SUCCESS) {                                   \
            std::cerr << "cuRAND error at " << __FILE__ << ":" << __LINE__    \
                      << " code=" << err << std::endl;                        \
            exit(EXIT_FAILURE);                                               \
        }                                                                      \
    } while (0)

// Macro for checking CUDA Runtime API calls
#define CUDA_CHECK(call)                                                      \
    do {                                                                       \
        hipError_t err = call;                                                \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__      \
                      << " code=" << err << " : " << hipGetErrorString(err)  \
                      << std::endl;                                          \
            exit(EXIT_FAILURE);                                               \
        }                                                                      \
    } while (0)

int main() {
    const size_t N = 10'000'000; // 10 million random numbers
    float *dev_random = nullptr;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&dev_random, N * sizeof(float)));

    // Create cuRAND generator
    hiprandGenerator_t gen;
    CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    // Set seed
    unsigned long long seed = 1234ULL;
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

    // Generate uniform random numbers in [0,1)
    CURAND_CHECK(hiprandGenerateUniform(gen, dev_random, N));

    // Copy a few values back to host for verification
    std::vector<float> host_random(10);
    CUDA_CHECK(hipMemcpy(host_random.data(), dev_random, 10 * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "First 10 random numbers:" << std::endl;
    for (size_t i = 0; i < host_random.size(); ++i) {
        std::cout << host_random[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    CUDA_CHECK(hipFree(dev_random));
    CURAND_CHECK(hiprandDestroyGenerator(gen));

    return 0;
}
```