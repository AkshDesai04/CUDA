/*
Is a kernel launch configuration error (e.g. too many threads) synchronous or asynchronous? (Asynchronous, but the error is usually available immediately via `cudaGetLastError`).

Thinking:
- Kernel launches are asynchronous: the host thread does not wait for the kernel to finish.
- Errors due to launch configuration (e.g., too many threads per block) are detected by the runtime at launch time.
- However, the error is not thrown until the host calls a CUDA runtime API that synchronizes the device or queries for errors.
- The standard way to check for such errors immediately after a launch is cudaGetLastError().
- This program queries the device properties to determine the maximum threads per block.
- It then intentionally launches a kernel with a block size exceeding this maximum to provoke a cudaErrorInvalidConfiguration.
- After the launch, it calls cudaGetLastError() and prints the error message, demonstrating that the error is available immediately.
- Finally, it calls cudaDeviceSynchronize() to ensure all previous operations are complete, and checks the error again (which should be cudaSuccess).
*/

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void dummyKernel()
{
    // Empty kernel
}

int main()
{
    hipDeviceProp_t devProp;
    int device = 0;
    hipError_t err = hipGetDevice(&device);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDevice failed: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    err = hipGetDeviceProperties(&devProp, device);
    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    printf("Device %d: %s\n", device, devProp.name);
    printf("Maximum threads per block: %d\n", devProp.maxThreadsPerBlock);

    // Intentionally use too many threads per block to trigger an error
    int blockSize = devProp.maxThreadsPerBlock + 1000; // Exceeds limit
    int gridSize = 1;

    printf("\nLaunching kernel with blockSize=%d (exceeds max threads per block)\n", blockSize);
    dummyKernel<<<gridSize, blockSize>>>();

    // Immediately check for launch configuration errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("cudaGetLastError() after launch: %s\n", hipGetErrorString(err));
    } else {
        printf("cudaGetLastError() after launch: CUDA_SUCCESS\n");
    }

    // Now synchronize and check again
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("cudaDeviceSynchronize() error: %s\n", hipGetErrorString(err));
    } else {
        printf("cudaDeviceSynchronize() succeeded.\n");
    }

    // Check error again after synchronization
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("cudaGetLastError() after synchronization: %s\n", hipGetErrorString(err));
    } else {
        printf("cudaGetLastError() after synchronization: CUDA_SUCCESS\n");
    }

    return EXIT_SUCCESS;
}
