```cuda
/*
Aim of the program: 
Use `hipGetDeviceProperties` and print the `driverVersion` field from the properties struct.

Thinking process:
1. Include necessary headers: <stdio.h> for printing and <hip/hip_runtime.h> for CUDA runtime API.
2. Define a helper macro `CHECK_CUDA` to wrap CUDA API calls and report errors with line numbers.
3. In `main`, determine the number of CUDA-capable devices with `hipGetDeviceCount`.
4. If at least one device exists, select device 0 and call `hipGetDeviceProperties` to fill a `hipDeviceProp_t` struct.
5. Extract the `driverVersion` field from the struct. The field is encoded as `major << 16 | minor`, so we decode it for readability.
6. Print the driver version in a human‑readable format.
7. Handle cases where no device is available or CUDA errors occur.
8. Return 0 on success.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                           \
    do {                                                           \
        hipError_t err = (call);                                  \
        if (err != hipSuccess) {                                 \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",           \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            return EXIT_FAILURE;                                  \
        }                                                          \
    } while (0)

int main(void) {
    int deviceCount = 0;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("No CUDA-capable device detected.\n");
        return EXIT_FAILURE;
    }

    int device = 0; // use the first device
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, device));

    int driverVersion = prop.driverVersion;
    int major = driverVersion >> 16;
    int minor = driverVersion & 0xFFFF;

    printf("Device %d: %s\n", device, prop.name);
    printf("Driver version: %d.%d\n", major, minor);

    return EXIT_SUCCESS;
}
```