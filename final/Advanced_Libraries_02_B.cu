```cpp
/*
Generate random integers instead of floats.
The aim of this program is to replace the generation of random floating-point numbers
with the generation of random integer values.  The original version of the
CUDA program used `hiprandGenerateUniform` to fill a device array with floats.
In this rewritten version, the program uses `hiprandGenerate`, which produces
random 32‑bit unsigned integers.  The program demonstrates how to set up a
CUDA random number generator, allocate memory on the device, generate an
array of random integers, copy the result back to the host, and display a
subset of the values.

Key points considered during the rewrite:

1. **Random number generator choice** – `hiprandGenerate` returns
   unsigned integers directly, so no conversion from floats is needed.
2. **Memory allocation** – both device and host arrays are allocated
   with the same size in terms of `unsigned int`.
3. **Error handling** – a helper function `checkCuda` and
   `checkCurand` are provided to wrap CUDA and CURAND API calls.
4. **Seed initialization** – the seed is set to the current time to
   ensure a different sequence on each run.
5. **Output** – the program prints the first 10 random integers
   to illustrate the result.

The resulting code is a complete, self‑contained CUDA program that can
be compiled with `nvcc`.  It demonstrates how to replace float
generation with integer generation while preserving the overall
structure of the original program.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

/* Helper macro to check CUDA runtime API calls */
#define checkCuda(call)                                                    \
    do {                                                                   \
        hipError_t err = call;                                            \
        if (err != hipSuccess) {                                         \
            fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",  \
                    __FILE__, __LINE__, err, hipGetErrorName(err),       \
                    hipGetErrorString(err));                             \
            exit(EXIT_FAILURE);                                           \
        }                                                                  \
    } while (0)

/* Helper macro to check CURAND API calls */
#define checkCurand(call)                                                  \
    do {                                                                   \
        hiprandStatus_t err = call;                                         \
        if (err != HIPRAND_STATUS_SUCCESS) {                               \
            fprintf(stderr, "CURAND error at %s:%d code=%d \"%s\" \n",     \
                    __FILE__, __LINE__, err, getCurandErrorString(err));  \
            exit(EXIT_FAILURE);                                           \
        }                                                                  \
    } while (0)

/* Translate CURAND error code to human readable string */
const char* getCurandErrorString(hiprandStatus_t err) {
    switch (err) {
        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";
        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";
        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";
        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";
        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
        case HIPRAND_STATUS_LAUNCH_FAILURE:
            return "HIPRAND_STATUS_LAUNCH_FAILURE";
        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";
        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";
        case HIPRAND_STATUS_ARCH_MISMATCH:
            return "HIPRAND_STATUS_ARCH_MISMATCH";
        case HIPRAND_STATUS_INTERNAL_ERROR:
            return "HIPRAND_STATUS_INTERNAL_ERROR";
        default:
            return "Unknown CURAND error";
    }
}

int main(void) {
    /* Number of random integers to generate */
    const size_t N = 1000000;          /* 1 million integers */

    /* Device memory for random integers */
    unsigned int *d_randInts = NULL;
    checkCuda(hipMalloc((void**)&d_randInts, N * sizeof(unsigned int)));

    /* Create CURAND generator */
    hiprandGenerator_t gen;
    checkCurand(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    /* Set seed */
    unsigned long long seed = (unsigned long long)time(NULL);
    checkCurand(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

    /* Generate N random unsigned integers on the device */
    checkCurand(hiprandGenerate(gen, d_randInts, N));

    /* Allocate host memory to copy back results */
    unsigned int *h_randInts = (unsigned int*)malloc(N * sizeof(unsigned int));
    if (h_randInts == NULL) {
        fprintf(stderr, "Host memory allocation failed\n");
        exit(EXIT_FAILURE);
    }

    /* Copy results from device to host */
    checkCuda(hipMemcpy(h_randInts, d_randInts, N * sizeof(unsigned int),
                         hipMemcpyDeviceToHost));

    /* Print the first 10 random integers */
    printf("First 10 random integers:\n");
    for (int i = 0; i < 10 && i < (int)N; ++i) {
        printf("%u\n", h_randInts[i]);
    }

    /* Clean up */
    free(h_randInts);
    checkCuda(hipFree(d_randInts));
    checkCurand(hiprandDestroyGenerator(gen));

    printf("Random integer generation completed successfully.\n");
    return 0;
}
```