#include "hip/hip_runtime.h"
```cuda
/*
 * Aim of the program:
 * Use `hipDeviceSynchronize` and proper error checking. An out-of-bounds access often results in `hipErrorIllegalAddress`. Catch and report this error.
 *
 * Thinking:
 * 1. We want a minimal, self‑contained CUDA program that deliberately triggers an illegal memory access on the device.
 * 2. To do this, we allocate a small array on the device, but launch a kernel with more threads than elements.
 * 3. Each thread writes to its index in the array. Threads with indices beyond the array size will write out of bounds.
 * 4. After the kernel launch we use `hipGetLastError()` to catch any launch‑time errors.
 * 5. Then we call `hipDeviceSynchronize()` to force completion of the kernel execution on the device.
 * 6. If the kernel performed an illegal address access, `hipDeviceSynchronize()` will return `hipErrorIllegalAddress`.
 * 7. We wrap CUDA API calls in a `CHECK_CUDA()` macro that prints the error string and aborts on failure.
 * 8. The program prints whether an illegal address error was detected or if execution succeeded.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

/* Macro for checking CUDA API call results */
#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                            \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                      \
                    __FILE__, __LINE__, hipGetErrorString(err));            \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while (0)

/* Kernel that intentionally writes out of bounds */
__global__ void write_out_of_bounds(int *data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    /* Each thread writes to its index regardless of bounds */
    data[idx] = idx * 2;
}

int main(void) {
    const int N = 10;              /* Size of the device array */
    const int threadsPerBlock = 32;
    const int blocks = 1;          /* Launch fewer blocks but more threads */

    /* Allocate device memory */
    int *d_array;
    CHECK_CUDA(hipMalloc((void **)&d_array, N * sizeof(int)));

    /* Launch kernel with more threads than elements */
    write_out_of_bounds<<<blocks, threadsPerBlock>>>(d_array, N);

    /* Check for launch errors */
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
    }

    /* Synchronize device and check for illegal address errors */
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        if (err == hipErrorIllegalAddress) {
            fprintf(stderr, "Detected illegal memory access during kernel execution.\n");
        } else {
            fprintf(stderr, "hipDeviceSynchronize error: %s\n", hipGetErrorString(err));
        }
    } else {
        printf("Kernel executed successfully (no illegal address detected).\n");
    }

    /* Clean up */
    CHECK_CUDA(hipFree(d_array));

    return 0;
}
```