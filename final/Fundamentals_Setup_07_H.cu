```cuda
/*
Aim of the program:
Check if the device supports cooperative launch (`cooperativeLaunch`) and print the result.

Thinking:
1. Cooperative launch support is queried via the CUDA Runtime API attribute `hipDeviceAttributeCooperativeLaunch`.
2. The program will:
   - Include <hip/hip_runtime.h> and <stdio.h>.
   - Retrieve the number of CUDA devices using `hipGetDeviceCount`.
   - Select the first device (device 0) for simplicity.
   - Use `hipDeviceGetAttribute` with the attribute `hipDeviceAttributeCooperativeLaunch` to determine support.
   - Handle errors from CUDA API calls and print human‑readable output: "Yes" if supported, "No" otherwise.
3. The program will be written in plain C syntax and compiled with `nvcc` as a `.cu` file.
4. No external libraries are required; all functionality is provided by the CUDA Runtime.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Error retrieving device count: %s\n", hipGetErrorString(err));
        return 1;
    }

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return 0;
    }

    // Use the first device (device 0)
    int device = 0;
    err = hipSetDevice(device);
    if (err != hipSuccess) {
        fprintf(stderr, "Error setting device %d: %s\n", device, hipGetErrorString(err));
        return 1;
    }

    int cooperativeLaunchSupported = 0;
    err = hipDeviceGetAttribute(&cooperativeLaunchSupported,
                                 hipDeviceAttributeCooperativeLaunch,
                                 device);
    if (err != hipSuccess) {
        fprintf(stderr, "Error querying cooperative launch attribute: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Device %d supports cooperative launch: %s\n",
           device,
           cooperativeLaunchSupported ? "Yes" : "No");

    return 0;
}
```