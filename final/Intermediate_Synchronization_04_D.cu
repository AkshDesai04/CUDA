#include "hip/hip_runtime.h"
```cuda
/* 
Use other atomic functions like `atomicSub`, `atomicMin`, `atomicMax`, `atomicExch`.

Thinking:
- The goal is to demonstrate the usage of several atomic operations provided by CUDA that are not the most common ones (`atomicAdd`, `atomicInc`). 
- We will create a small kernel that performs each of the requested atomic operations on device memory.
- `atomicSub` will decrement a shared counter.
- `atomicMin` and `atomicMax` will maintain the minimum and maximum values found in an input array.
- `atomicExch` will replace the value at each index with a constant (here 100) and can be used to retrieve the old value if desired.
- The host code will allocate and initialize the necessary arrays, launch the kernel, copy the results back, and print them to verify correctness.
- Simple error-checking macros will be used for clarity.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                         \
    do {                                                         \
        hipError_t err = call;                                  \
        if (err != hipSuccess) {                                \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",         \
                    __FILE__, __LINE__, hipGetErrorString(err));\
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    } while (0)

// Kernel that demonstrates atomicSub, atomicMin, atomicMax, atomicExch
__global__ void atomicDemoKernel(int *counter,
                                 int *minVal,
                                 int *maxVal,
                                 int *exchange,
                                 const int *input,
                                 int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // atomicSub: decrement the shared counter by 1
    atomicSub(counter, 1);

    // atomicMin: update global minimum with input[idx]
    atomicMin(minVal, input[idx]);

    // atomicMax: update global maximum with input[idx]
    atomicMax(maxVal, input[idx]);

    // atomicExch: replace exchange[idx] with 100, returning old value
    int oldVal = atomicExch(&exchange[idx], 100);
    // (optional) we could store oldVal somewhere, but for this demo we ignore it
}

int main(void)
{
    const int N = 1024;
    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Host memory
    int *h_input = (int *)malloc(N * sizeof(int));
    int *h_exchange = (int *)malloc(N * sizeof(int));
    int h_counter, h_minVal, h_maxVal;

    // Initialize input array with values 0..N-1
    for (int i = 0; i < N; ++i) {
        h_input[i] = i;
        h_exchange[i] = -1; // initial dummy value
    }

    // Initial counter value: N (since each thread will decrement by 1)
    h_counter = N;
    // Set minVal to a large number, maxVal to a small number
    h_minVal = INT_MAX;
    h_maxVal = INT_MIN;

    // Device memory
    int *d_input, *d_exchange, *d_counter, *d_minVal, *d_maxVal;
    CUDA_CHECK(hipMalloc((void **)&d_input, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_exchange, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_counter, sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_minVal, sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&d_maxVal, sizeof(int)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_exchange, h_exchange, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_counter, &h_counter, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_minVal, &h_minVal, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_maxVal, &h_maxVal, sizeof(int), hipMemcpyHostToDevice));

    // Launch kernel
    atomicDemoKernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_counter,
                                                 d_minVal,
                                                 d_maxVal,
                                                 d_exchange,
                                                 d_input,
                                                 N);
    CUDA_CHECK(hipGetLastError());

    // Copy results back to host
    CUDA_CHECK(hipMemcpy(&h_counter, d_counter, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&h_minVal, d_minVal, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&h_maxVal, d_maxVal, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_exchange, d_exchange, N * sizeof(int), hipMemcpyDeviceToHost));

    // Print results
    printf("Final counter (should be 0): %d\n", h_counter);
    printf("Minimum value in array: %d\n", h_minVal);
    printf("Maximum value in array: %d\n", h_maxVal);
    // Verify that all exchange values are now 100
    int allExchanged = 1;
    for (int i = 0; i < N; ++i) {
        if (h_exchange[i] != 100) {
            allExchanged = 0;
            break;
        }
    }
    printf("All exchange values replaced with 100: %s\n",
           allExchanged ? "Yes" : "No");

    // Clean up
    free(h_input);
    free(h_exchange);
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_exchange));
    CUDA_CHECK(hipFree(d_counter));
    CUDA_CHECK(hipFree(d_minVal));
    CUDA_CHECK(hipFree(d_maxVal));

    return 0;
}
```