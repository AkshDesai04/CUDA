#include "hip/hip_runtime.h"
```cpp
/*
Use `hipMemcpyToSymbol` and `hipMemcpyFromSymbol` to initialize and read the `__device__` variable from the host.

Thinking:
The goal is to demonstrate how host code can set a global device variable using hipMemcpyToSymbol, launch a kernel that operates on it, and then retrieve its value back to the host using hipMemcpyFromSymbol. 
We define a simple __device__ integer, set it to a known value on the host, launch a kernel that increments it, copy it back to the host, and print the result. 
Error checking is included after each CUDA call. 
*/

#include <iostream>
#include <hip/hip_runtime.h>

// Global device variable
__device__ int devVar;

// Kernel that modifies the device variable
__global__ void incKernel() {
    devVar += 10;
}

// Helper macro for error checking
#define CHECK_CUDA(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            std::cerr << "CUDA error at " << __FILE__         \
                      << ":" << __LINE__ << " - "             \
                      << hipGetErrorString(err) << std::endl;\
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

int main() {
    int initVal = 100;
    int hostVar = 0;

    // Initialize the device variable from the host
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(devVar), &initVal, sizeof(int),
                                  0, hipMemcpyHostToDevice));

    // Launch the kernel
    incKernel<<<1,1>>>();
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    // Read the updated value back to the host
    CHECK_CUDA(hipMemcpyFromSymbol(&hostVar, HIP_SYMBOL(devVar), sizeof(int),
                                    0, hipMemcpyDeviceToHost));

    std::cout << "Value of devVar after kernel execution: " << hostVar << std::endl;

    return 0;
}
```