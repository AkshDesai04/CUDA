#include "hip/hip_runtime.h"
```cpp
/* 
Aim: Explain the difference between hipPeekAtLastError and hipGetLastError.

Thinking:
- The CUDA runtime keeps an error state that can be queried after a kernel launch or any API call.
- hipPeekAtLastError() returns the last error that occurred, but it does NOT reset the error state.
- hipGetLastError() also returns the last error that occurred, but in addition it resets the error state to hipSuccess.
- This distinction is important when you want to inspect errors without clearing them (peek), or when you want to handle an error and clear it for subsequent calls (get).

In this small example we launch a kernel that intentionally causes an error
by using an invalid launch configuration (zero blocks). We then call both
hipPeekAtLastError and hipGetLastError to demonstrate the difference in behavior.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dummyKernel()
{
    // Do nothing
}

int main(void)
{
    // Intentionally launch kernel with 0 blocks to cause an error
    dummyKernel<<<0, 1>>>();

    // Check error state with hipPeekAtLastError (does not reset)
    hipError_t peekErr = hipPeekAtLastError();
    if (peekErr != hipSuccess) {
        printf("hipPeekAtLastError: %s\n", hipGetErrorString(peekErr));
    } else {
        printf("hipPeekAtLastError: no error\n");
    }

    // Check error state with hipGetLastError (resets error state)
    hipError_t getErr = hipGetLastError();
    if (getErr != hipSuccess) {
        printf("hipGetLastError: %s\n", hipGetErrorString(getErr));
    } else {
        printf("hipGetLastError: no error\n");
    }

    // Call peek again to show that the error state has been reset
    hipError_t peekErrAfterGet = hipPeekAtLastError();
    if (peekErrAfterGet != hipSuccess) {
        printf("hipPeekAtLastError after get: %s\n", hipGetErrorString(peekErrAfterGet));
    } else {
        printf("hipPeekAtLastError after get: no error\n");
    }

    return 0;
}
```