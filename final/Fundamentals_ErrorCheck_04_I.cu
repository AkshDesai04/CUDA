```cuda
/*
Pass a valid device pointer but a `nullptr` host pointer to `hipMemcpy` HtoD. What error does this cause?

Thought process:
- The program aims to demonstrate that passing a null pointer as the source (host side) while providing a valid destination (device side) to hipMemcpy in a HostToDevice transfer results in an error.
- According to CUDA documentation, hipMemcpy requires both source and destination pointers to be valid addresses for the transfer direction. Supplying a nullptr for the source violates this contract.
- The expected error code returned by hipMemcpy (and retrievable via hipGetLastError) is hipErrorInvalidValue, which indicates that an argument is invalid.
- The code below allocates a device buffer, intentionally sets the host pointer to nullptr, performs hipMemcpy, checks the returned error, and prints the error string. This confirms the expected behavior.
*/

#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
    // Allocate device memory (valid device pointer)
    int *d_ptr = nullptr;
    size_t size = 10 * sizeof(int);
    hipError_t err = hipMalloc((void**)&d_ptr, size);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Intentionally set host pointer to nullptr
    int *h_ptr = nullptr;

    // Attempt to copy from host to device with nullptr host pointer
    err = hipMemcpy(d_ptr, h_ptr, size, hipMemcpyHostToDevice);

    // Check the error
    if (err != hipSuccess) {
        std::cout << "hipMemcpy returned error: " << hipGetErrorString(err) << std::endl;
    } else {
        std::cout << "hipMemcpy succeeded unexpectedly." << std::endl;
    }

    // Clean up device memory
    hipFree(d_ptr);

    return 0;
}
```