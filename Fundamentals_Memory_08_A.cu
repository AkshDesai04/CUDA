```cuda
/* 
Perform the same timing for a device-to-host copy. Is it typically faster, slower, or the same?

I want to write a simple CUDA program that measures how long it takes to copy data
from device memory to host memory using hipMemcpy with hipMemcpyDeviceToHost.  
The aim is to use CUDA events to time the transfer, which gives us GPU-side
timing that is accurate for the actual data transfer.  
The program will:
  1. Allocate a large array on the host and the device (e.g., 100 million floats).
  2. Initialize the device array with some data (e.g., fill with 1.0f).
  3. Perform a warm‑up copy from device to host so that any initial
     overheads are amortized.
  4. Run several iterations of device‑to‑host copies, measuring the elapsed
     time for each iteration with hipEventRecord and hipEventElapsedTime.
  5. Compute and print the average copy time in milliseconds.
  6. Clean up allocated memory.

I’ll also include error checking after each CUDA call, so that any failures
are reported clearly.  The code will be minimal but fully functional and
self‑contained, suitable for compiling with nvcc. 
*/

#include <cstdio>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n",     \
                    __FILE__, __LINE__, err, hipGetErrorName(err),         \
                    hipGetErrorString(err));                               \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)

int main(void) {
    const size_t N = 100 * 1024 * 1024; // 100 million floats (~400 MB)
    const int iterations = 10;

    // Host memory allocation
    float *h_data = (float*)malloc(N * sizeof(float));
    if (!h_data) {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    // Device memory allocation
    float *d_data;
    CUDA_CHECK(hipMalloc((void**)&d_data, N * sizeof(float)));

    // Initialize device array to some value (e.g., 1.0f)
    CUDA_CHECK(hipMemset(d_data, 0, N * sizeof(float))); // zero for simplicity

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Warm-up copy: device to host (ignored in timing)
    CUDA_CHECK(hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost));

    // Timing loop
    float totalTime = 0.0f;
    for (int i = 0; i < iterations; ++i) {
        CUDA_CHECK(hipEventRecord(start, 0));

        CUDA_CHECK(hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost));

        CUDA_CHECK(hipEventRecord(stop, 0));
        CUDA_CHECK(hipEventSynchronize(stop));

        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        totalTime += milliseconds;

        printf("Iteration %2d: %f ms\n", i + 1, milliseconds);
    }

    float avgTime = totalTime / iterations;
    printf("\nAverage device-to-host copy time: %f ms\n", avgTime);

    // Clean up
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_data));
    free(h_data);

    return 0;
}
```