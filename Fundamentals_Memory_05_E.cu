```cuda
/*
Use `nvidia-smi` to watch the GPU memory usage while your program runs. Step through with a debugger to see the memory increase after `hipMalloc` and decrease after `hipFree`.

[Thinking]
The goal is to provide a minimal CUDA program that explicitly allocates a block of GPU memory using `hipMalloc` and then frees it with `hipFree`. By running the program and observing the GPU memory usage with `nvidia-smi`, one should see a spike in allocated memory after the `hipMalloc` call and a drop after the `hipFree` call. The program includes basic error checking and prompts the user to press Enter before freeing the memory, giving time to observe the memory state. This simple example is ideal for stepping through in a debugger, watching the allocation and deallocation, and correlating the changes with `nvidia-smi` output.

Implementation details:
- Allocate 100 MB of device memory to make the memory change noticeable.
- Use `hipGetErrorString` for readable error messages.
- Use `getchar()` to pause execution after allocation so the user can run `nvidia-smi` and inspect the GPU memory usage.
- After the user presses Enter, free the memory and print a confirmation message.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    const size_t size = 100 * 1024 * 1024; // 100 MB
    int *d_ptr = nullptr;

    // Allocate memory on the device
    hipError_t err = hipMalloc((void **)&d_ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("Allocated %zu bytes on GPU.\n", size);

    // Pause so the user can run nvidia-smi and watch memory usage
    printf("Press Enter to free memory and exit.\n");
    getchar();

    // Free the allocated memory
    err = hipFree(d_ptr);
    if (err != hipSuccess) {
        fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("Freed GPU memory.\n");

    return 0;
}
```