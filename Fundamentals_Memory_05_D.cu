```cpp
/*
Aim of the program: Try to call `hipFree` on a host pointer from `malloc`. What error does this produce?

Thinking process:
- The CUDA runtime expects `hipFree` to be called on memory that was allocated with `hipMalloc` or
  `hipHostAlloc` (or other CUDA allocation routines). A pointer obtained from standard C `malloc`
  is a host memory pointer that has no association with the GPU.
- According to the CUDA Runtime API documentation, passing a non-device pointer to `hipFree`
  should result in the error code `hipErrorInvalidDevicePointer` (error code 17).
- To verify this, we write a small CUDA C program that:
  1. Allocates a block of host memory using `malloc`.
  2. Attempts to free that block with `hipFree`.
  3. Checks the returned error code and prints it using `hipGetErrorString`.
  4. Cleans up the host memory with `free` afterwards.
- When compiled and executed, the program should print an error message similar to:
  "hipFree error: invalid device pointer".
- The program demonstrates that `hipFree` cannot be used to free host memory allocated
  with `malloc`, and shows the specific CUDA error returned.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int main(void) {
    const size_t size = 1024;
    // Allocate host memory with malloc
    void *host_ptr = malloc(size);
    if (!host_ptr) {
        fprintf(stderr, "Failed to allocate host memory with malloc.\n");
        return EXIT_FAILURE;
    }

    // Attempt to free the host memory using hipFree
    hipError_t err = hipFree(host_ptr);

    // Print the error returned by hipFree
    if (err != hipSuccess) {
        printf("hipFree error: %s (code %d)\n", hipGetErrorString(err), err);
    } else {
        printf("hipFree succeeded unexpectedly.\n");
    }

    // Clean up the host memory
    free(host_ptr);

    return EXIT_SUCCESS;
}
```