/*
Aim of the program: Query and print the memory bus width of the primary GPU in bits.

Thinking:
1. We need to include the CUDA runtime header <cuda_runtime.h> and standard C headers for I/O.
2. In main(), we'll call cudaGetDeviceCount to ensure at least one GPU is available.
3. We'll use device 0 as the primary GPU.
4. Retrieve cudaDeviceProp struct via cudaGetDeviceProperties.
5. The memory bus width is available in the field memoryBusWidth of cudaDeviceProp.
6. Print the value in bits using printf.
7. Add basic error checking after each CUDA API call to handle potential failures.
8. Compile with nvcc and run; it should display the bus width.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device count: %s\n", hipGetErrorString(err));
        return 1;
    }
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA-capable devices found.\n");
        return 1;
    }

    int device = 0; // primary GPU
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device properties: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Memory bus width of primary GPU (device %d) in bits: %d\n", device, prop.memoryBusWidth);

    return 0;
}
