```cpp
/*
Aim of the program: Trigger `hipErrorInvalidValue` by calling `hipSetDevice` with a negative device number.

Thinking Process:
1. The goal is to deliberately cause an error by using an invalid argument for hipSetDevice.
2. According to CUDA runtime documentation, passing a negative device number should result in
   hipErrorInvalidValue.
3. The program will simply call hipSetDevice(-1) and then check the return value.
4. If an error occurs, it will print the error string using hipGetErrorString.
5. The program is minimal, includes necessary headers, and uses the C API of CUDA.
6. We ensure to compile the file with nvcc and it will produce the expected error when run.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{
    int negativeDevice = -1;
    hipError_t err = hipSetDevice(negativeDevice);

    if (err != hipSuccess)
    {
        printf("hipSetDevice failed as expected: %s\n", hipGetErrorString(err));
    }
    else
    {
        printf("Unexpected success: hipSetDevice returned hipSuccess.\n");
    }

    return 0;
}
```