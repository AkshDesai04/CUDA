```cpp
/*
Aim of the program: Write a program that allocates 100MB of memory on every available GPU.

Thought process:
1. Query the number of CUDA-capable devices with hipGetDeviceCount.
2. For each device:
   - Set the current device with hipSetDevice.
   - Allocate 100 * 1024 * 1024 bytes (100 MB) using hipMalloc.
   - Check for allocation success; if it fails, print the error and continue.
3. Store each allocated pointer in an array so that the allocations remain alive until program exit.
4. After allocating on all devices, print a summary and then exit.
   - Since the program terminates, the allocated device memory will be freed automatically by the OS/runtime.
5. Use a simple error-checking macro for clarity.
6. Keep the code in plain C (using CUDA Runtime API) and compile with nvcc as a .cu file.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

/* Simple macro for checking CUDA errors */
#define CHECK_CUDA(call)                                           \
    do {                                                           \
        hipError_t err = call;                                    \
        if (err != hipSuccess) {                                  \
            fprintf(stderr, "CUDA error at %s:%d - %s\n",          \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            return -1;                                             \
        }                                                          \
    } while (0)

int main(void) {
    int deviceCount = 0;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
        return 0;
    }

    printf("Found %d CUDA device(s).\n", deviceCount);

    /* Allocate an array to store pointers for each device */
    void **devicePointers = (void**)malloc(deviceCount * sizeof(void*));
    if (devicePointers == NULL) {
        fprintf(stderr, "Failed to allocate host memory for device pointers.\n");
        return -1;
    }

    const size_t allocSize = 100 * 1024 * 1024; /* 100 MB */

    for (int dev = 0; dev < deviceCount; ++dev) {
        printf("\nDevice %d:\n", dev);
        CHECK_CUDA(hipSetDevice(dev));

        /* Optional: query device properties */
        hipDeviceProp_t prop;
        CHECK_CUDA(hipGetDeviceProperties(&prop, dev));
        printf("  Name: %s\n", prop.name);
        printf("  Total global memory: %lu bytes\n", prop.totalGlobalMem);

        /* Attempt to allocate 100 MB */
        hipError_t allocErr = hipMalloc(&devicePointers[dev], allocSize);
        if (allocErr == hipSuccess) {
            printf("  Successfully allocated 100 MB on device %d.\n", dev);
        } else {
            printf("  Failed to allocate 100 MB on device %d: %s\n",
                   dev, hipGetErrorString(allocErr));
            devicePointers[dev] = NULL;
        }
    }

    /* Keep allocations alive for a short period (optional) */
    // sleep(5); // Uncomment if you want the program to pause before exiting

    /* Clean up: free device memory */
    for (int dev = 0; dev < deviceCount; ++dev) {
        if (devicePointers[dev] != NULL) {
            hipSetDevice(dev);
            hipFree(devicePointers[dev]);
        }
    }

    free(devicePointers);

    printf("\nAll allocated memory has been freed. Exiting.\n");
    return 0;
}
```