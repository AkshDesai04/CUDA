```cpp
/*
Aim of the program: For each device, query and print whether it supports running kernels concurrently.

Thinking:
- The goal is to examine every CUDA-capable device present on the system and determine if it can run kernels concurrently. 
- CUDA provides the device property 'concurrentKernels' in the hipDeviceProp_t structure, which is set to 1 if the device supports concurrent kernel execution.
- We need to query the number of devices using hipGetDeviceCount.
- For each device, we retrieve its properties with hipGetDeviceProperties.
- We then check the 'concurrentKernels' field and print an appropriate message.
- Basic error checking is added for CUDA API calls to ensure robustness.
- The output will display the device index, name, and whether concurrent kernel execution is supported.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void) {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get device count: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return EXIT_SUCCESS;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, dev);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to get properties for device %d: %s\n", dev, hipGetErrorString(err));
            continue;
        }

        printf("Device %d: %s\n", dev, prop.name);
        if (prop.concurrentKernels) {
            printf("  Supports running kernels concurrently.\n");
        } else {
            printf("  Does NOT support running kernels concurrently.\n");
        }
    }

    return EXIT_SUCCESS;
}
```