#include "hip/hip_runtime.h"
```cpp
/*
Aim: Explain the difference between synchronous and asynchronous CUDA errors. 
Write a code snippet that would cause a synchronous error (e.g., invalid hipMalloc size) 
and another that would cause an asynchronous error (e.g., out-of-bounds access inside a kernel).

Thinking:
1. CUDA operations can fail in two distinct ways:
   - Synchronous errors are reported immediately by the CUDA API function that was called.
     For example, hipMalloc will return a non-zero error code if the requested size is
     invalid (negative, too large, or exceeds the device memory). These errors are
     detected on the host side before any kernel launch or asynchronous operation
     can take place.

   - Asynchronous errors happen inside a kernel or other asynchronous stream
     operations. The API call that initiates the operation (e.g., kernel launch)
     returns successfully, but the actual error (like out‑of‑bounds memory
     access, division by zero, etc.) occurs later when the device processes the
     instruction. The error is captured by hipGetLastError() only after the
     operation has completed or when a synchronization point (hipDeviceSynchronize
     or hipStreamSynchronize) is reached.

2. In the code below:
   - The first part intentionally requests an unrealistically large allocation
     to trigger a synchronous hipMalloc error. We check the return value
     immediately and print the error.

   - The second part allocates a small array, launches a kernel that writes
     past the end of the array (indexing beyond the allocated size). The
     kernel launch returns successfully; the error is captured only when
     hipGetLastError() or hipDeviceSynchronize() is called. We demonstrate
     both error retrieval methods.

3. The program is self-contained and compiles with nvcc. It prints both
   error messages to illustrate the difference between the two types.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

// Simple error checking macro for synchronous errors
#define CHECK_CUDA(call)                                            \
    do {                                                            \
        hipError_t err = (call);                                   \
        if (err != hipSuccess) {                                   \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",             \
                    __FILE__, __LINE__, hipGetErrorString(err));   \
            exit(EXIT_FAILURE);                                     \
        }                                                           \
    } while (0)

// Kernel that will write out-of-bounds for indices >= N
__global__ void outOfBoundsKernel(int *d_arr, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Intentionally write to d_arr even if idx >= N
    d_arr[idx] = idx * 2;
}

int main(void) {
    /* ---------- Synchronous error: invalid hipMalloc size ---------- */
    size_t hugeSize = 1ULL << 62; // ~4EB, far beyond device memory
    int *d_ptr = nullptr;
    hipError_t syncErr = hipMalloc((void**)&d_ptr, hugeSize);
    if (syncErr != hipSuccess) {
        printf("Synchronous error (invalid hipMalloc size): %s\n",
               hipGetErrorString(syncErr));
    } else {
        // Should never get here
        printf("Unexpectedly succeeded in allocating huge memory.\n");
        hipFree(d_ptr);
    }

    /* ---------- Asynchronous error: out-of-bounds kernel access ---------- */
    const int N = 10;
    int *d_arr = nullptr;
    CHECK_CUDA(hipMalloc((void**)&d_arr, N * sizeof(int)));
    CHECK_CUDA(hipMemset(d_arr, 0, N * sizeof(int)));

    // Launch more threads than elements to force out-of-bounds write
    const int threadsPerBlock = 4;
    const int blocks = 4; // 4*4 = 16 threads > N=10
    outOfBoundsKernel<<<blocks, threadsPerBlock>>>(d_arr, N);

    // Check for errors immediately after launch (will still be asynchronous)
    hipError_t asyncErr = hipGetLastError();
    if (asyncErr != hipSuccess) {
        printf("Asynchronous error after kernel launch (early check): %s\n",
               hipGetErrorString(asyncErr));
    } else {
        printf("No error reported immediately after kernel launch.\n");
    }

    // Force synchronization to allow the device to process the kernel
    asyncErr = hipDeviceSynchronize();
    if (asyncErr != hipSuccess) {
        printf("Asynchronous error detected after hipDeviceSynchronize: %s\n",
               hipGetErrorString(asyncErr));
    } else {
        printf("Kernel executed successfully (unexpectedly).\n");
    }

    // Cleanup
    hipFree(d_arr);

    return 0;
}
```