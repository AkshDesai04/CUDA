#include "hip/hip_runtime.h"
```cpp
/* 
Refactor your vector add program to include post-launch error checking.
Think: The user wants a CUDA vector addition example that includes error checking after launching the kernel. We should start with a clear comment block at the top that states the aim verbatim, then provide our reasoning. We need to allocate host and device memory, copy input data, launch the kernel, then immediately check for launch errors with hipGetLastError(), and also check for runtime errors after hipDeviceSynchronize(). If any errors occur, print a message and exit. The kernel itself will perform element-wise addition. The host code will also verify the result or simply print a few elements to confirm. We should keep the code minimal but functional, using standard CUDA headers, error-checking macros, and proper cleanup. No extra output or text should be produced beyond the .cu file content. 
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel for element-wise vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

// Helper macro for CUDA error checking
#define CHECK_CUDA(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n", \
                    #call, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

int main(void) {
    const int N = 1 << 20; // 1M elements
    const size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(N - i);
    }

    // Allocate device memory
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;
    CHECK_CUDA(hipMalloc((void**)&d_A, size));
    CHECK_CUDA(hipMalloc((void**)&d_B, size));
    CHECK_CUDA(hipMalloc((void**)&d_C, size));

    // Copy input data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Post-launch error checking
    hipError_t err = hipGetLastError(); // Check for launch errors
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Wait for kernel to finish and check for runtime errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA runtime error after kernel execution: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Verify a few results
    bool success = true;
    for (int i = 0; i < 5; ++i) {
        float expected = h_A[i] + h_B[i];
        if (fabs(h_C[i] - expected) > 1e-5f) {
            fprintf(stderr, "Verification failed at index %d: %f != %f\n", i, h_C[i], expected);
            success = false;
            break;
        }
    }
    if (success) {
        printf("Vector addition successful for first 5 elements.\n");
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
```