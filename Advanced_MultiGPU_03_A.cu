#include "hip/hip_runtime.h"
```c
/*
 * Aim of the program:
 * Measure the bandwidth of the P2P `hipMemcpy` and compare it to the bandwidth
 * of a manual copy through the host (GPU1 -> Host -> GPU0).
 *
 * Thinking and design:
 * 1. Determine the number of available CUDA devices. The benchmark requires at
 *    least two GPUs. If fewer than two are available, the program exits with a
 *    message.
 *
 * 2. Allocate a large buffer on each GPU. We use 512 MB (1<<29 bytes) which
 *    provides enough data to get a stable measurement. Device 0 holds the
 *    destination buffer; Device 1 holds the source buffer.
 *
 * 3. Enable peer-to-peer (P2P) access between the two GPUs using
 *    `hipDeviceEnablePeerAccess`. P2P may not be available on all hardware,
 *    so we check the return code. If P2P is not supported, we report it and
 *    skip the P2P copy measurement, but still perform the host path copy.
 *
 * 4. Initialize the source buffer on GPU1. We simply set all bytes to 0xAA
 *    using `hipMemset`. This is fast and sufficient for bandwidth measurement.
 *
 * 5. Timing is performed with CUDA events (`hipEvent_t`). Each copy operation
 *    is wrapped by a start/stop event pair, and the elapsed time is obtained
 *    via `hipEventElapsedTime` (returns milliseconds).
 *
 * 6. P2P copy measurement:
 *    - Set the current device to GPU1.
 *    - Record start event.
 *    - Perform `hipMemcpyPeer` from GPU1 buffer to GPU0 buffer.
 *    - Record stop event.
 *    - Compute elapsed time and bandwidth (bytes / seconds).
 *
 * 7. Host copy measurement:
 *    - Allocate a pinned host buffer of the same size using `hipHostAlloc`
 *      to ensure high throughput.
 *    - Copy from GPU1 to host (`hipMemcpy`).
 *    - Copy from host to GPU0 (`hipMemcpy`).
 *    - The two transfers are timed separately or together; here we time
 *      the entire round‑trip in a single measurement.
 *
 * 8. Print the measured bandwidths in GB/s for both P2P and host‑mediated
 *    transfers. Also report whether P2P was available.
 *
 * 9. Clean up: free all device and host memory, disable peer access,
 *    and destroy CUDA events.
 *
 * 10. Error handling: All CUDA API calls are wrapped in a macro that checks
 *     the returned `hipError_t` and prints an informative message before
 *     exiting. This ensures that any failure is clearly reported.
 *
 * The program is self‑contained and can be compiled with:
 *     nvcc -arch=sm_70 -o bandwidth_test bandwidth_test.cu
 * (Adjust the architecture as needed for your GPUs.)
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/* Macro for checking CUDA errors */
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                    \
                    __FILE__, __LINE__, hipGetErrorString(err));           \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)

int main(void) {
    const size_t bufSize = 512ULL * 1024 * 1024;  // 512 MB
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount < 2) {
        fprintf(stderr, "Need at least 2 CUDA devices, found %d.\n", deviceCount);
        return EXIT_FAILURE;
    }

    int dev0 = 0;
    int dev1 = 1;

    /* Allocate device memory */
    void *devBuf0 = NULL, *devBuf1 = NULL;
    CUDA_CHECK(hipSetDevice(dev0));
    CUDA_CHECK(hipMalloc(&devBuf0, bufSize));
    CUDA_CHECK(hipSetDevice(dev1));
    CUDA_CHECK(hipMalloc(&devBuf1, bufSize));

    /* Enable peer access */
    int canAccessPeer = 0;
    CUDA_CHECK(hipDeviceCanAccessPeer(&canAccessPeer, dev0, dev1));
    int p2pAvailable = 0;
    if (canAccessPeer) {
        /* Enable peer access from dev1 to dev0 */
        CUDA_CHECK(hipSetDevice(dev1));
        hipError_t err = hipDeviceEnablePeerAccess(dev0, 0);
        if (err == hipSuccess) {
            p2pAvailable = 1;
        } else if (err == hipErrorPeerAccessAlreadyEnabled) {
            p2pAvailable = 1;
        } else {
            fprintf(stderr, "Failed to enable P2P from device %d to %d: %s\n",
                    dev1, dev0, hipGetErrorString(err));
        }
    }

    /* Initialize source buffer on GPU1 */
    CUDA_CHECK(hipSetDevice(dev1));
    CUDA_CHECK(hipMemset(devBuf1, 0xAA, bufSize));

    /* Create CUDA events for timing */
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    /* Measure P2P copy if available */
    float p2pTimeMs = 0.0f;
    if (p2pAvailable) {
        CUDA_CHECK(hipSetDevice(dev1));  // Set to source device
        CUDA_CHECK(hipEventRecord(start, 0));
        CUDA_CHECK(hipMemcpyPeer(devBuf0, dev0, devBuf1, dev1, bufSize));
        CUDA_CHECK(hipEventRecord(stop, 0));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&p2pTimeMs, start, stop));
    }

    /* Measure host-mediated copy */
    /* Allocate pinned host memory */
    void *hostBuf = NULL;
    CUDA_CHECK(hipHostAlloc(&hostBuf, bufSize, hipHostMallocDefault));

    CUDA_CHECK(hipEventRecord(start, 0));
    /* Copy from GPU1 to host */
    CUDA_CHECK(hipMemcpy(hostBuf, devBuf1, bufSize, hipMemcpyDeviceToHost));
    /* Copy from host to GPU0 */
    CUDA_CHECK(hipMemcpy(devBuf0, hostBuf, bufSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    float hostTimeMs;
    CUDA_CHECK(hipEventElapsedTime(&hostTimeMs, start, stop));

    /* Compute bandwidths */
    double p2pBandwidthGBs = 0.0;
    if (p2pAvailable) {
        p2pBandwidthGBs = (double)bufSize / (p2pTimeMs / 1e3) / 1e9;
    }
    double hostBandwidthGBs = (double)bufSize / (hostTimeMs / 1e3) / 1e9;

    /* Print results */
    printf("Buffer size: %.2f MB\n", (double)bufSize / (1024 * 1024));
    if (p2pAvailable) {
        printf("P2P copy time: %.3f ms, bandwidth: %.3f GB/s\n",
               p2pTimeMs, p2pBandwidthGBs);
    } else {
        printf("P2P copy not available.\n");
    }
    printf("Host-mediated copy time: %.3f ms, bandwidth: %.3f GB/s\n",
           hostTimeMs, hostBandwidthGBs);

    /* Cleanup */
    CUDA_CHECK(hipFree(devBuf0));
    CUDA_CHECK(hipFree(devBuf1));
    CUDA_CHECK(hipHostFree(hostBuf));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    /* Disable peer access if enabled */
    if (p2pAvailable) {
        CUDA_CHECK(hipSetDevice(dev1));
        CUDA_CHECK(hipDeviceDisablePeerAccess(dev0));
    }

    return EXIT_SUCCESS;
}
```