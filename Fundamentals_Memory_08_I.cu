```cuda
/*
Use `hipMemcpyAsync` on a stream and record events around it to time the asynchronous transfer.
In this program we allocate pinned host memory and device memory for a large array of integers.
We then create a CUDA stream and two CUDA events (startEvent and stopEvent). 
Before issuing the asynchronous memory copy we record startEvent, and immediately after we record stopEvent. 
We then synchronize on the stream (or the stopEvent) to ensure the copy has finished.
Finally, we query the elapsed time between the two events using hipEventElapsedTime, which gives the time in milliseconds that the asynchronous transfer took. 
The program also demonstrates a simple error checking macro and reports the measured transfer time and the transfer throughput in GB/s. 
The logic is straightforward: allocate memory, initialize host data, create stream and events, time the async memcpy, synchronize, compute elapsed time, clean up.
*/

#include <cstdio>
#include <cstdlib>

#define CHECK_CUDA(call)                                                   \
    do {                                                                   \
        hipError_t err = (call);                                          \
        if (err != hipSuccess) {                                         \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                   \
                    __FILE__, __LINE__, hipGetErrorString(err));         \
            exit(EXIT_FAILURE);                                           \
        }                                                                  \
    } while (0)

int main() {
    const size_t N = 10 * 1024 * 1024; // 10 million integers
    const size_t size = N * sizeof(int);

    // Allocate pinned host memory for source and destination
    int *h_src, *h_dst;
    CHECK_CUDA(hipHostAlloc((void**)&h_src, size, hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc((void**)&h_dst, size, hipHostMallocDefault));

    // Initialize source data
    for (size_t i = 0; i < N; ++i) {
        h_src[i] = static_cast<int>(i);
    }

    // Allocate device memory
    int *d_buf;
    CHECK_CUDA(hipMalloc((void**)&d_buf, size));

    // Create a stream
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    // Create events for timing
    hipEvent_t startEvent, stopEvent;
    CHECK_CUDA(hipEventCreate(&startEvent));
    CHECK_CUDA(hipEventCreate(&stopEvent));

    // Record start event
    CHECK_CUDA(hipEventRecord(startEvent, stream));

    // Asynchronously copy from host to device
    CHECK_CUDA(hipMemcpyAsync(d_buf, h_src, size, hipMemcpyHostToDevice, stream));

    // Record stop event
    CHECK_CUDA(hipEventRecord(stopEvent, stream));

    // Wait for the stream to complete (ensures copy finished)
    CHECK_CUDA(hipStreamSynchronize(stream));

    // Compute elapsed time
    float ms = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&ms, startEvent, stopEvent));

    // Calculate throughput in GB/s
    float gbTransferred = (float)size / (1024.0f * 1024.0f * 1024.0f);
    float throughput = gbTransferred / (ms / 1000.0f);

    printf("Asynchronous memcpy transferred %.2f GB in %.3f ms (throughput: %.2f GB/s)\n",
           gbTransferred, ms, throughput);

    // Clean up
    CHECK_CUDA(hipEventDestroy(startEvent));
    CHECK_CUDA(hipEventDestroy(stopEvent));
    CHECK_CUDA(hipStreamDestroy(stream));
    CHECK_CUDA(hipFree(d_buf));
    CHECK_CUDA(hipHostFree(h_src));
    CHECK_CUDA(hipHostFree(h_dst));

    return 0;
}
```