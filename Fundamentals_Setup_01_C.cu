```cpp
/* 
Write a program to explicitly set the active CUDA device to 0 using `hipSetDevice`. Verify the call was successful by then calling `hipGetDevice` and printing the result.

Thinking process:
1. Include necessary headers: <hip/hip_runtime.h> for CUDA runtime API and <cstdio> for printing.
2. In main, call hipSetDevice(0). Check the return value; if not hipSuccess, print an error and exit.
3. Call hipGetDevice to retrieve the currently active device. Again check the return value for errors.
4. Print the device number obtained from hipGetDevice, which should be 0 if everything worked.
5. Return 0 from main to indicate successful execution.
6. Ensure the code is self-contained in a single .cu file so it can be compiled with nvcc.
*/

#include <hip/hip_runtime.h>
#include <cstdio>

int main(void) {
    int device = 0;
    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipSetDevice(%d) failed: %s\n", device, hipGetErrorString(err));
        return 1;
    }

    int currentDevice = -1;
    err = hipGetDevice(&currentDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipGetDevice failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Successfully set active device to %d. Current device is %d.\n", device, currentDevice);

    return 0;
}
```