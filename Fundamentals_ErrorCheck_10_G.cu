#include "hip/hip_runtime.h"
```cpp
/*
What happens if you call `elapsed_ms()` before `stop()` has been called or before the event has completed? The result is undefined or an error.

Thinking:
- The CUDA API provides hipEventElapsedTime to compute elapsed time between two events.
- It requires that both events have been recorded with hipEventRecord before the call.
- If the stop event has not been recorded yet, or the events have not been properly synchronized, hipEventElapsedTime may return hipErrorNotReady or other error, indicating the operation cannot be performed.
- Calling hipEventElapsedTime before stop is recorded could lead to undefined behavior if the API does not guard against it, but in practice it returns an error.
- The demonstration will record a start event, launch a kernel, call hipEventElapsedTime immediately (before recording stop), print the error, then record stop, and call again to get a valid elapsed time.
- This shows that the result is undefined or error until the stop event has completed.
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                            \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",                      \
                    __FILE__, __LINE__, hipGetErrorString(err));            \
            exit(EXIT_FAILURE);                                              \
        }                                                                     \
    } while (0)

// A simple kernel that performs a small amount of work
__global__ void dummyKernel(int *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Simple computation to keep the GPU busy
        for (int i = 0; i < 1000; ++i) {
            data[idx] = data[idx] * (i % 3 + 1);
        }
    }
}

int main(void) {
    const int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(int);

    // Allocate host memory
    int *h_data = (int *)malloc(size);
    for (int i = 0; i < N; ++i) h_data[i] = i;

    // Allocate device memory
    int *d_data;
    CHECK_CUDA(hipMalloc((void **)&d_data, size));
    CHECK_CUDA(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));

    // Create CUDA events
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Record the start event
    CHECK_CUDA(hipEventRecord(start, 0));

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    dummyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, N);

    // **Important:** We do NOT record the stop event yet.
    // Call hipEventElapsedTime before stop has been recorded.
    float elapsed_ms = 0.0f;
    hipError_t err = hipEventElapsedTime(&elapsed_ms, start, stop);
    if (err != hipSuccess) {
        printf("First hipEventElapsedTime call (before stop recorded) returned error: %s\n",
               hipGetErrorString(err));
    } else {
        printf("First hipEventElapsedTime call (before stop recorded) succeeded unexpectedly. Elapsed: %f ms\n",
               elapsed_ms);
    }

    // Now record the stop event
    CHECK_CUDA(hipEventRecord(stop, 0));

    // Wait for the stop event to complete
    CHECK_CUDA(hipEventSynchronize(stop));

    // Now call hipEventElapsedTime again, should succeed
    err = hipEventElapsedTime(&elapsed_ms, start, stop);
    if (err != hipSuccess) {
        printf("Second hipEventElapsedTime call failed: %s\n",
               hipGetErrorString(err));
    } else {
        printf("Second hipEventElapsedTime call succeeded. Elapsed: %f ms\n",
               elapsed_ms);
    }

    // Clean up
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(d_data));
    free(h_data);

    return 0;
}
```