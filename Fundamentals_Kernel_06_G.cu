#include "hip/hip_runtime.h"
```cuda
/* 
Launch the kernel with 10 threads, but have all 10 threads try to write to `data[0]`. 
What value do you expect to see when you copy it back? (This is a race condition). 

Thinking: 
- The kernel is launched with 10 threads in a single block. 
- All threads write to the same memory location `data[0]`. 
- Because the writes are not synchronized and no atomic operation is used, the writes 
  will race with each other. 
- The final value of `data[0]` after the kernel completes will be whatever value was 
  written by the last thread that the hardware happens to execute last. 
- Since thread scheduling on the GPU is non-deterministic for these operations, 
  the result is unpredictable and could be any of the values written by the threads 
  (e.g., 0 through 9). This demonstrates a classic race condition.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel where each thread writes its thread index to data[0]
__global__ void writeToData(int *data) {
    // All threads write to the same memory location
    data[0] = threadIdx.x;
}

int main(void) {
    const int ARRAY_SIZE = 1;
    const int NUM_THREADS = 10;

    // Host array
    int h_data[ARRAY_SIZE] = {0};

    // Device pointer
    int *d_data = NULL;
    hipError_t err;

    // Allocate device memory
    err = hipMalloc((void**)&d_data, ARRAY_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy host to device (not really needed here but kept for completeness)
    err = hipMemcpy(d_data, h_data, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy H2D failed: %s\n", hipGetErrorString(err));
        hipFree(d_data);
        return 1;
    }

    // Launch kernel with 10 threads
    writeToData<<<1, NUM_THREADS>>>(d_data);

    // Wait for GPU to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        hipFree(d_data);
        return 1;
    }

    // Copy result back to host
    err = hipMemcpy(h_data, d_data, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy D2H failed: %s\n", hipGetErrorString(err));
        hipFree(d_data);
        return 1;
    }

    // Print the result
    printf("Value of data[0] after kernel execution: %d\n", h_data[0]);

    // Free device memory
    hipFree(d_data);

    return 0;
}
```