```cuda
/* 
Aim of the program:
Query and print the `persistingL2CacheMaxSize` property.

Thinking:
- The property `persistingL2CacheMaxSize` is exposed via the CUDA Runtime API
  as the device attribute `cudaDevAttrPersistingL2CacheMaxSize`.  
- We will use `hipGetDeviceCount()` to determine the number of CUDA-capable
  devices present on the system.  
- For each device, we will call `hipDeviceGetAttribute()` with the attribute
  identifier to retrieve the size in bytes.  
- The returned value is an integer; if the attribute is not supported on a
  particular device the API will return `hipErrorInvalidValue`.  
- We will print the device index and the attribute value (or an error
  message) in a human-readable format.
- Error handling will be performed with a helper macro `CUDA_CHECK` that
  prints an error message and exits if any CUDA API call fails.
*/

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

// Helper macro for error checking
#define CUDA_CHECK(call)                                            \
    do {                                                            \
        hipError_t err = (call);                                   \
        if (err != hipSuccess) {                                   \
            fprintf(stderr, "CUDA error at %s:%d - %s\n",           \
                    __FILE__, __LINE__, hipGetErrorString(err));  \
            exit(EXIT_FAILURE);                                     \
        }                                                           \
    } while (0)

int main(void) {
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return EXIT_SUCCESS;
    }

    printf("Found %d CUDA device(s).\n", deviceCount);

    for (int dev = 0; dev < deviceCount; ++dev) {
        int persistingL2CacheMaxSize = 0;
        hipError_t err = hipDeviceGetAttribute(&persistingL2CacheMaxSize,
                                                 cudaDevAttrPersistingL2CacheMaxSize,
                                                 dev);
        if (err == hipSuccess) {
            printf("Device %d: persistingL2CacheMaxSize = %d bytes\n",
                   dev, persistingL2CacheMaxSize);
        } else if (err == hipErrorInvalidValue) {
            printf("Device %d: persistingL2CacheMaxSize attribute not supported.\n",
                   dev);
        } else {
            fprintf(stderr, "Failed to query persistingL2CacheMaxSize for device %d: %s\n",
                    dev, hipGetErrorString(err));
        }
    }

    return EXIT_SUCCESS;
}
```