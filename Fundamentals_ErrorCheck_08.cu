#include "hip/hip_runtime.h"
```cpp
/*
Aim of the program:
Run a program that has an out-of-bounds memory access inside the kernel (an asynchronous error) with and without proper post-launch error checking (`hipDeviceSynchronize`). Observe that without it, the program may complete without any visible error, but potentially with incorrect results.

Thinking:
- Allocate a small device array (N = 10).
- Launch a kernel that writes to indices up to N+5, thereby accessing memory beyond the allocated region.
- Perform the launch twice:
    1. Without calling `hipDeviceSynchronize()` after the kernel. Immediately check for errors with `hipGetLastError()` and perform a `hipMemcpy()` to bring data back to the host. The asynchronous out-of-bounds access will not be reported yet, so we expect no error and possibly corrupted data.
    2. With a `hipDeviceSynchronize()` after the kernel launch. The error from the out-of-bounds write will be reported at this point.
- Print host data to see that values may be corrupted in the first case.
- The program demonstrates the importance of synchronizing after kernel launches to catch asynchronous errors such as out-of-bounds memory accesses.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel that deliberately writes beyond the bounds of the allocated array.
__global__ void outOfBoundsKernel(int *d_arr, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Intentionally write up to N+5 elements.
    if (idx < N + 5)
    {
        d_arr[idx] = idx;  // Out-of-bounds write when idx >= N
    }
}

int main()
{
    const int N = 10;               // Size of the device array
    const int OUT_BOUNDS = 5;       // Extra elements to write beyond bounds
    const int TOTAL = N + OUT_BOUNDS; // Total elements we will attempt to write

    int *d_arr = nullptr;
    hipError_t err = hipMalloc((void **)&d_arr, N * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Allocate host memory large enough to hold the potentially corrupted data.
    int h_arr[TOTAL];
    // Initialize host memory for clarity.
    for (int i = 0; i < TOTAL; ++i)
        h_arr[i] = -1;

    // ---------- Case 1: Launch kernel without synchronization ----------
    printf("\nCase 1: Launch kernel without hipDeviceSynchronize()\n");
    outOfBoundsKernel<<<1, TOTAL>>>(d_arr, N);

    // Immediately check for errors (will not catch asynchronous errors).
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("hipGetLastError after kernel launch: %s\n", hipGetErrorString(err));
    else
        printf("hipGetLastError after kernel launch: hipSuccess\n");

    // Copy back the first N elements (the allocated region).
    err = hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        printf("hipMemcpy after kernel launch (no sync): %s\n", hipGetErrorString(err));
    else
        printf("hipMemcpy after kernel launch (no sync): hipSuccess\n");

    // Print the host data to observe any corruption.
    printf("Host array after no-sync launch (first %d elements):\n", N);
    for (int i = 0; i < N; ++i)
        printf("h_arr[%d] = %d\n", i, h_arr[i]);

    // ---------- Case 2: Launch kernel with synchronization ----------
    // Reinitialize device array to zero for clarity.
    err = hipMemset(d_arr, 0, N * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "hipMemset failed: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return 1;
    }

    // Reinitialize host memory.
    for (int i = 0; i < TOTAL; ++i)
        h_arr[i] = -1;

    printf("\nCase 2: Launch kernel with hipDeviceSynchronize()\n");
    outOfBoundsKernel<<<1, TOTAL>>>(d_arr, N);

    // Synchronize to catch asynchronous errors.
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        printf("hipDeviceSynchronize after kernel launch: %s\n", hipGetErrorString(err));
    else
        printf("hipDeviceSynchronize after kernel launch: hipSuccess\n");

    // After synchronization, check last error again.
    err = hipGetLastError();
    if (err != hipSuccess)
        printf("hipGetLastError after hipDeviceSynchronize: %s\n", hipGetErrorString(err));
    else
        printf("hipGetLastError after hipDeviceSynchronize: hipSuccess\n");

    // Copy back the first N elements.
    err = hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        printf("hipMemcpy after kernel launch (with sync): %s\n", hipGetErrorString(err));
    else
        printf("hipMemcpy after kernel launch (with sync): hipSuccess\n");

    // Print the host data to observe any corruption.
    printf("Host array after sync launch (first %d elements):\n", N);
    for (int i = 0; i < N; ++i)
        printf("h_arr[%d] = %d\n", i, h_arr[i]);

    // Clean up.
    hipFree(d_arr);
    return 0;
}
```