/*
Pass a valid device pointer but a `nullptr` host pointer to `cudaMemcpy` HtoD. What error does this cause?
The aim of this program is to demonstrate the error that occurs when attempting to copy data from a host
pointer that is null to a device memory location. In CUDA, `cudaMemcpy` expects both source and destination
pointers to be valid for the direction of transfer. If the source pointer is `nullptr` for a
`cudaMemcpyHostToDevice` operation, the API should return `cudaErrorInvalidValue`. 
The program allocates device memory, sets the host pointer to `nullptr`, performs the copy, and
prints the resulting error code and string. This confirms that the error thrown is indeed
`cudaErrorInvalidValue`.
*/

#include <cstdio>
#include <hip/hip_runtime.h>

int main(void) {
    const size_t N = 10;
    const size_t size = N * sizeof(int);

    // Allocate device memory
    int* d_ptr = nullptr;
    hipError_t err = hipMalloc((void**)&d_ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    // Host pointer is null
    int* h_ptr = nullptr;

    // Attempt to copy from null host pointer to device
    err = hipMemcpy(d_ptr, h_ptr, size, hipMemcpyHostToDevice);

    // Print the error
    printf("hipMemcpy returned error: %s (code %d)\n", hipGetErrorString(err), err);

    // Clean up
    hipFree(d_ptr);

    return (err == hipSuccess) ? EXIT_SUCCESS : EXIT_FAILURE;
}
