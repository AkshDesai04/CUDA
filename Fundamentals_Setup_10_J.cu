```cuda
/*
Aim: Query and print the maximum texture dimension for a
1D texture using hipDeviceGetAttribute with hipDeviceAttributeMaxTexture1DWidth.

Thinking:
The program is a simple CUDA host code that uses the CUDA runtime API to
retrieve the maximum supported width for 1D textures on each available GPU.
The relevant API is hipDeviceGetAttribute which takes an attribute enum,
a device ID, and returns the attribute value. The attribute enum
hipDeviceAttributeMaxTexture1DWidth specifies the maximum width of a 1D texture.

Steps:
1. Include necessary headers: <hip/hip_runtime.h> for CUDA runtime functions,
   <iostream> for printing.
2. In main():
   a. Query the number of CUDA devices with hipGetDeviceCount.
   b. If no devices are found, print a message and exit.
   c. Loop over each device ID from 0 to deviceCount-1.
   d. For each device, call hipDeviceGetAttribute with
      hipDeviceAttributeMaxTexture1DWidth.
   e. Handle any errors from hipDeviceGetAttribute and print a
      descriptive error message.
   f. If successful, print the device number and the maximum texture
      1D width.
3. Use proper error checking for each CUDA call. The helper macro
   CUDA_CHECK will wrap calls and abort on failure with a message.
4. Compile with nvcc and run. It will print the maximum 1D texture width
   for each GPU in the system.

Edge cases:
- Devices that do not support 1D textures? In practice all GPUs support
  1D textures; the attribute should be valid. If an attribute is not
  supported, CUDA will return hipErrorInvalidValue, which we handle.
- Very large values are represented as int; on recent GPUs the value
  can be >2^31, so we cast to size_t for printing to avoid overflow.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>

// Helper macro for CUDA error checking
#define CUDA_CHECK(call)                                                        \
    do {                                                                        \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__        \
                      << " : " << hipGetErrorString(err) << std::endl;          \
            std::exit(EXIT_FAILURE);                                            \
        }                                                                       \
    } while (0)

int main() {
    int deviceCount = 0;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        std::cout << "No CUDA devices found." << std::endl;
        return EXIT_SUCCESS;
    }

    std::cout << "Found " << deviceCount << " CUDA device(s)." << std::endl;

    for (int dev = 0; dev < deviceCount; ++dev) {
        size_t maxTexture1DWidth = 0;
        hipError_t err = hipDeviceGetAttribute(
            reinterpret_cast<int*>(&maxTexture1DWidth),
            hipDeviceAttributeMaxTexture1DWidth,
            dev);

        if (err != hipSuccess) {
            std::cerr << "Failed to get hipDeviceAttributeMaxTexture1DWidth for device "
                      << dev << " : " << hipGetErrorString(err) << std::endl;
            continue;
        }

        std::cout << "Device " << dev
                  << " : Maximum 1D texture width = "
                  << maxTexture1DWidth << std::endl;
    }

    return EXIT_SUCCESS;
}
```