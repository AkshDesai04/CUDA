#include "hip/hip_runtime.h"
```cpp
/*
Theoretical maximum speedup depends on the relative times of Copy, Exec, and Copy.
If T_copy_HtoD + T_copy_DtoH > T_exec, the program is memory-bound and overlap is very effective.

=== Thinking and Design Notes ===
1. The program demonstrates asynchronous data transfer and kernel execution using CUDA streams.
2. We measure three critical timing components:
   - T_copy_HtoD: time to copy input data from host to device.
   - T_exec: time to execute the kernel on the device.
   - T_copy_DtoH: time to copy output data from device back to host.
3. We launch the kernel in a separate CUDA stream from the memory copies to allow overlap.
4. Timing is measured with CUDA events (hipEventRecord, hipEventSynchronize, hipEventElapsedTime).
5. After obtaining the raw timings (in milliseconds), we calculate:
   - Sequential time: T_seq = T_copy_HtoD + T_exec + T_copy_DtoH.
   - Overlapped time: T_ovlp = max(T_copy_HtoD, T_exec) + T_copy_DtoH.
   - Theoretical speedup: Speedup = T_seq / T_ovlp.
6. The program prints the three timings, the overlapped time, the sequential time, and the speedup.
7. By varying array size or kernel workload, we can observe the memory-bound scenario
   where T_copy_HtoD + T_copy_DtoH > T_exec, leading to an effective overlap.
8. The kernel performs a simple vector addition (C = A + B) to keep the example focused
   on transfer overhead rather than computation.
9. The code is self-contained, written in CUDA C, and can be compiled with nvcc:
   nvcc -o async_overlap async_overlap.cu
10. No external libraries are required beyond the CUDA runtime.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Simple vector addition kernel: C = A + B
__global__ void vecAdd(const float *A, const float *B, float *C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}

// Helper function to check CUDA errors
inline void checkCuda(hipError_t result, const char *msg)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[])
{
    // Parameters
    const int N = 1 << 24;               // 16M elements (~64 MB per array)
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    size_t bytes = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(bytes);
    float *h_B = (float *)malloc(bytes);
    float *h_C = (float *)malloc(bytes);
    if (!h_A || !h_B || !h_C)
    {
        fprintf(stderr, "Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    // Initialize host arrays
    for (int i = 0; i < N; ++i)
    {
        h_A[i] = (float)i;
        h_B[i] = (float)(N - i);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCuda(hipMalloc((void **)&d_A, bytes), "hipMalloc d_A");
    checkCuda(hipMalloc((void **)&d_B, bytes), "hipMalloc d_B");
    checkCuda(hipMalloc((void **)&d_C, bytes), "hipMalloc d_C");

    // Create CUDA streams
    hipStream_t streamCopy, streamKernel;
    checkCuda(hipStreamCreate(&streamCopy), "hipStreamCreate streamCopy");
    checkCuda(hipStreamCreate(&streamKernel), "hipStreamCreate streamKernel");

    // Create CUDA events for timing
    hipEvent_t startEvent, stopEvent;
    checkCuda(hipEventCreate(&startEvent), "hipEventCreate startEvent");
    checkCuda(hipEventCreate(&stopEvent), "hipEventCreate stopEvent");

    // --- Asynchronous data transfer and kernel launch with overlap ---

    // Record start event
    checkCuda(hipEventRecord(startEvent, 0), "hipEventRecord startEvent");

    // 1. Asynchronously copy A and B to device in streamCopy
    checkCuda(hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice, streamCopy), "hipMemcpyAsync HtoD d_A");
    checkCuda(hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice, streamCopy), "hipMemcpyAsync HtoD d_B");

    // 2. Launch kernel in streamKernel (will wait for device to be ready)
    checkCuda(hipMemcpyAsync(NULL, NULL, 0, hipMemcpyDefault, streamKernel), "hipStreamSynchronize dummy");
    checkCuda(hipEventRecord(startEvent, streamKernel), "hipEventRecord startEvent kernel");
    vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streamKernel>>>(d_A, d_B, d_C, N);
    checkCuda(hipGetLastError(), "Kernel launch");

    // 3. Asynchronously copy result back to host in streamCopy after kernel completes
    checkCuda(hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost, streamCopy), "hipMemcpyAsync DtoH d_C");

    // Record stop event
    checkCuda(hipEventRecord(stopEvent, streamCopy), "hipEventRecord stopEvent");

    // Synchronize to ensure all operations are finished
    checkCuda(hipEventSynchronize(stopEvent), "hipEventSynchronize stopEvent");

    // Measure elapsed time in milliseconds
    float ms_total;
    checkCuda(hipEventElapsedTime(&ms_total, startEvent, stopEvent), "hipEventElapsedTime");

    // --- Separate measurements for individual components ---

    // Measure HtoD copy time
    hipEvent_t h2d_start, h2d_stop;
    checkCuda(hipEventCreate(&h2d_start), "hipEventCreate h2d_start");
    checkCuda(hipEventCreate(&h2d_stop), "hipEventCreate h2d_stop");
    checkCuda(hipEventRecord(h2d_start, 0), "hipEventRecord h2d_start");
    checkCuda(hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice, streamCopy), "hipMemcpyAsync HtoD d_A");
    checkCuda(hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice, streamCopy), "hipMemcpyAsync HtoD d_B");
    checkCuda(hipEventRecord(h2d_stop, streamCopy), "hipEventRecord h2d_stop");
    checkCuda(hipEventSynchronize(h2d_stop), "hipEventSynchronize h2d_stop");
    float ms_h2d;
    checkCuda(hipEventElapsedTime(&ms_h2d, h2d_start, h2d_stop), "hipEventElapsedTime h2d");

    // Measure kernel execution time
    hipEvent_t k_start, k_stop;
    checkCuda(hipEventCreate(&k_start), "hipEventCreate k_start");
    checkCuda(hipEventCreate(&k_stop), "hipEventCreate k_stop");
    checkCuda(hipEventRecord(k_start, 0), "hipEventRecord k_start");
    vecAdd<<<blocksPerGrid, threadsPerBlock, 0, streamKernel>>>(d_A, d_B, d_C, N);
    checkCuda(hipEventRecord(k_stop, streamKernel), "hipEventRecord k_stop");
    checkCuda(hipEventSynchronize(k_stop), "hipEventSynchronize k_stop");
    float ms_k;
    checkCuda(hipEventElapsedTime(&ms_k, k_start, k_stop), "hipEventElapsedTime k");

    // Measure DtoH copy time
    hipEvent_t d2h_start, d2h_stop;
    checkCuda(hipEventCreate(&d2h_start), "hipEventCreate d2h_start");
    checkCuda(hipEventCreate(&d2h_stop), "hipEventCreate d2h_stop");
    checkCuda(hipEventRecord(d2h_start, 0), "hipEventRecord d2h_start");
    checkCuda(hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost, streamCopy), "hipMemcpyAsync DtoH d_C");
    checkCuda(hipEventRecord(d2h_stop, streamCopy), "hipEventRecord d2h_stop");
    checkCuda(hipEventSynchronize(d2h_stop), "hipEventSynchronize d2h_stop");
    float ms_d2h;
    checkCuda(hipEventElapsedTime(&ms_d2h, d2h_start, d2h_stop), "hipEventElapsedTime d2h");

    // Compute sequential and overlapped times
    float seq_time = ms_h2d + ms_k + ms_d2h;
    float ovlp_time = fmaxf(ms_h2d, ms_k) + ms_d2h;
    float speedup = seq_time / ovlp_time;

    // Output results
    printf("=== Timing Results (ms) ===\n");
    printf("Host-to-Device copy time:   %f\n", ms_h2d);
    printf("Kernel execution time:      %f\n", ms_k);
    printf("Device-to-Host copy time:   %f\n", ms_d2h);
    printf("\nSequential time:            %f\n", seq_time);
    printf("Overlapped time:            %f\n", ovlp_time);
    printf("Theoretical speedup:        %f\n", speedup);

    // Clean up
    checkCuda(hipEventDestroy(startEvent), "hipEventDestroy startEvent");
    checkCuda(hipEventDestroy(stopEvent), "hipEventDestroy stopEvent");
    checkCuda(hipEventDestroy(h2d_start), "hipEventDestroy h2d_start");
    checkCuda(hipEventDestroy(h2d_stop), "hipEventDestroy h2d_stop");
    checkCuda(hipEventDestroy(k_start), "hipEventDestroy k_start");
    checkCuda(hipEventDestroy(k_stop), "hipEventDestroy k_stop");
    checkCuda(hipEventDestroy(d2h_start), "hipEventDestroy d2h_start");
    checkCuda(hipEventDestroy(d2h_stop), "hipEventDestroy d2h_stop");

    checkCuda(hipStreamDestroy(streamCopy), "hipStreamDestroy streamCopy");
    checkCuda(hipStreamDestroy(streamKernel), "hipStreamDestroy streamKernel");

    checkCuda(hipFree(d_A), "hipFree d_A");
    checkCuda(hipFree(d_B), "hipFree d_B");
    checkCuda(hipFree(d_C), "hipFree d_C");

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
```