/*
Aim: Use your GpuTimer class to time a vector addition kernel.

Thinking:
1. The task requires a CUDA program that times a vector addition kernel using a GpuTimer class. Since GpuTimer is not part of the CUDA runtime, I'll implement a small utility class that uses CUDA events to measure elapsed GPU time. The class will have methods start(), stop(), elapsed(), and reset().

2. The vector addition kernel will add two float arrays element‑wise. It will be launched with a typical grid/block configuration.

3. In main:
   - Allocate host vectors A, B, and C with a chosen size (e.g., 1e7 elements).
   - Initialize A and B with some data.
   - Allocate corresponding device memory.
   - Copy A and B to the device.
   - Create a GpuTimer instance.
   - Start the timer, launch the kernel, synchronize, stop the timer.
   - Copy result back to host and optionally verify correctness.
   - Print the elapsed time.

4. The code will be self‑contained in a single .cu file and will compile with `nvcc`.

5. Edge cases: error checking for CUDA calls, ensuring the timer works correctly. We'll also ensure we clean up all resources.

Now the actual CUDA code follows.
*/

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

// GpuTimer class using CUDA events
class GpuTimer {
public:
    GpuTimer() {
        hipEventCreate(&startEvent);
        hipEventCreate(&stopEvent);
    }

    ~GpuTimer() {
        hipEventDestroy(startEvent);
        hipEventDestroy(stopEvent);
    }

    void reset() {
        hipEventRecord(startEvent, 0);
        hipEventRecord(stopEvent, 0);
    }

    void start() {
        hipEventRecord(startEvent, 0);
    }

    void stop() {
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
    }

    float elapsed() const { // milliseconds
        float ms = 0;
        hipEventElapsedTime(&ms, startEvent, stopEvent);
        return ms;
    }

private:
    hipEvent_t startEvent, stopEvent;
};

// Vector addition kernel
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 1 << 24; // about 16 million elements
    const size_t bytes = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(N - i);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    // Copy data to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Determine grid/block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Create and use GpuTimer
    GpuTimer timer;
    timer.start();

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Stop timer after kernel execution
    timer.stop();

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Verify a few elements
    bool correct = true;
    for (int i = 0; i < 10; ++i) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            correct = false;
            break;
        }
    }

    // Print results
    printf("Vector addition completed. Elapsed time: %f ms\n", timer.elapsed());
    printf("Verification: %s\n", correct ? "PASSED" : "FAILED");

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
