#include "hip/hip_runtime.h"
```cpp
/*
Aim of the program:
Use pinned host memory (`hipHostMalloc`) for your data buffers.

Thinking:
- The goal is to demonstrate how to allocate host memory that is pinned (page-locked) so that CUDA can perform faster memory transfers between host and device.
- We'll allocate a simple array of floats on the host using hipHostMalloc. This ensures the memory is pinned.
- We'll allocate device memory of the same size with hipMalloc.
- We'll copy data from the pinned host buffer to the device buffer using hipMemcpy.
- We'll launch a simple kernel that increments each element by 1.0f.
- We'll copy the result back from the device to the pinned host buffer.
- Finally, we'll verify the result on the host.
- Error checking is performed after each CUDA API call and kernel launch.
- Resources (pinned host memory, device memory) are freed before program exit.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(call)                                          \
    do {                                                          \
        hipError_t err = call;                                   \
        if (err != hipSuccess) {                                 \
            fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", \
                    __FILE__, __LINE__, err, hipGetErrorName(err), hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    } while (0)

// Simple kernel: add 1.0f to each element
__global__ void addOne(float *d_data, size_t N) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        d_data[idx] += 1.0f;
    }
}

int main(void) {
    const size_t N = 1 << 20; // 1M elements
    const size_t size = N * sizeof(float);

    // Allocate pinned host memory
    float *h_data = NULL;
    CHECK_CUDA(hipHostMalloc((void**)&h_data, size));

    // Initialize host data
    for (size_t i = 0; i < N; ++i) {
        h_data[i] = static_cast<float>(i);
    }

    // Allocate device memory
    float *d_data = NULL;
    CHECK_CUDA(hipMalloc((void**)&d_data, size));

    // Copy data from host to device
    CHECK_CUDA(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));

    // Launch kernel
    const size_t threadsPerBlock = 256;
    const size_t blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    addOne<<<blocksPerGrid, threadsPerBlock>>>(d_data, N);
    CHECK_CUDA(hipGetLastError()); // Check kernel launch errors
    CHECK_CUDA(hipDeviceSynchronize()); // Ensure kernel finished

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost));

    // Verify results
    bool success = true;
    for (size_t i = 0; i < N; ++i) {
        if (fabs(h_data[i] - (static_cast<float>(i) + 1.0f)) > 1e-5f) {
            fprintf(stderr, "Mismatch at index %zu: host %f, expected %f\n",
                    i, h_data[i], static_cast<float>(i) + 1.0f);
            success = false;
            break;
        }
    }
    if (success) {
        printf("Verification passed.\n");
    } else {
        printf("Verification failed.\n");
    }

    // Clean up
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipHostFree(h_data));

    return 0;
}
```