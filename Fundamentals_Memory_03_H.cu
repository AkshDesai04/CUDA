/*
What happens if the source and destination pointers for a `hipMemcpyDeviceToDevice` overlap? (Behavior is undefined, use `cudaMemmove` if needed).

The goal of this program is to illustrate the undefined behavior that can occur when performing a `hipMemcpyDeviceToDevice` where the source and destination memory regions overlap.  CUDA’s documentation explicitly states that such an operation is undefined; the result can be corrupted data or other unexpected behavior.  To correctly handle overlapping copies we should use `cudaMemmove`.

In this example, we allocate a device array of 10 integers and initialise it from the host.  We then create two device pointers that point into this array such that the memory regions we copy overlap.  We first perform a standard `hipMemcpyDeviceToDevice` on these overlapping ranges and copy the data back to the host to observe any corruption.  Next we perform the same copy using `cudaMemmove`, which is guaranteed to handle overlaps correctly, and copy the data back again to compare the results.

The output will show that the result of the `hipMemcpyDeviceToDevice` is unreliable, whereas the `cudaMemmove` version yields the expected data.  This demonstrates the importance of choosing the right API for overlapping memory copies in CUDA.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                         \
    do {                                                         \
        hipError_t err = call;                                  \
        if (err != hipSuccess) {                                \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n",    \
                    #call, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                  \
        }                                                        \
    } while (0)

int main(void)
{
    const int N = 10;
    const int bytes = N * sizeof(int);

    // Host memory allocation
    int h_host[N];
    for (int i = 0; i < N; ++i)
        h_host[i] = i + 1;  // 1, 2, 3, ...

    // Device memory allocation
    int *d_dev;
    CHECK_CUDA(hipMalloc((void **)&d_dev, bytes));

    // Copy data from host to device
    CHECK_CUDA(hipMemcpy(d_dev, h_host, bytes, hipMemcpyHostToDevice));

    // Create overlapping pointers
    // p_src points to elements 2,3,4,5 (indices 2-5)
    // p_dst points to elements 5,6,7,8 (indices 5-8)
    int *p_src = d_dev + 2;
    int *p_dst = d_dev + 5;

    // Number of ints to copy (4 ints = 16 bytes)
    size_t copy_bytes = 4 * sizeof(int);

    printf("=== Before copy ===\n");
    CHECK_CUDA(hipMemcpy(h_host, d_dev, bytes, hipMemcpyDeviceToHost));
    for (int i = 0; i < N; ++i)
        printf("%2d ", h_host[i]);
    printf("\n\n");

    // Perform undefined overlapping copy
    printf("Performing hipMemcpyDeviceToDevice with overlapping regions...\n");
    CHECK_CUDA(hipMemcpy(p_dst, p_src, copy_bytes, hipMemcpyDeviceToDevice));

    // Copy back to host to inspect result
    CHECK_CUDA(hipMemcpy(h_host, d_dev, bytes, hipMemcpyDeviceToHost));
    printf("=== After hipMemcpyDeviceToDevice (overlap) ===\n");
    for (int i = 0; i < N; ++i)
        printf("%2d ", h_host[i]);
    printf("\n\n");

    // Reinitialize device array to original values
    CHECK_CUDA(hipMemcpy(d_dev, h_host, bytes, hipMemcpyHostToDevice));
    for (int i = 0; i < N; ++i)
        h_host[i] = i + 1;
    CHECK_CUDA(hipMemcpy(d_dev, h_host, bytes, hipMemcpyHostToDevice));

    // Perform correct overlapping copy using cudaMemmove
    printf("Performing cudaMemmove with overlapping regions...\n");
    CHECK_CUDA(cudaMemmove(p_dst, p_src, copy_bytes));

    // Copy back to host to inspect result
    CHECK_CUDA(hipMemcpy(h_host, d_dev, bytes, hipMemcpyDeviceToHost));
    printf("=== After cudaMemmove (overlap) ===\n");
    for (int i = 0; i < N; ++i)
        printf("%2d ", h_host[i]);
    printf("\n");

    // Clean up
    CHECK_CUDA(hipFree(d_dev));

    return 0;
}
